#include "demo_utils.cuh"
#include <cmath>
#include <fstream>
#include <iomanip>
#include <iostream>

using namespace quad;

int
main()
{
  double epsrel = 1.0e-3; // starting error tolerance.
  double const epsrel_min = 1.024e-10;
  double true_value = 1.286889807581113e+13;
  GENZ_2_6D integrand;
  constexpr int ndim = 6;
  
  PrintHeader();
  while (cu_time_and_call<GENZ_2_6D, ndim>("GENZ_2_6D",
                       integrand,
                       epsrel,
                       true_value,
                       std::cout) == true &&
         epsrel > epsrel_min) {
    epsrel /= 5.0;
  }
}
