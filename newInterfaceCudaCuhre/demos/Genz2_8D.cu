#include "demo_utils.cuh"
#include <cmath>
#include <fstream>
#include <iomanip>
#include <iostream>

using namespace quad;

int
main()
{
  double epsrel = 1.0e-3; // starting error tolerance.
  double const epsrel_min = 1.0e-13;
  double true_value = 3.015702399795044e+17;
  GENZ_2_8D integrand;
  constexpr int ndim = 8;
  PrintHeader();

  while (cu_time_and_call<GENZ_2_8D, ndim>("GENZ_2_8D",
                           integrand,
                           epsrel,
                           true_value,
                           std::cout) == true &&
             epsrel > epsrel_min) {
    epsrel /= 5.0;
   }
}
