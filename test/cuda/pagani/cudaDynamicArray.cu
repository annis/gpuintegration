#include "hip/hip_runtime.h"
#define CATCH_CONFIG_MAIN

#include "catch2/catch.hpp"
#include "common/cuda/hipArray.cuh"

template <typename arrayType>
__global__ void
set_vals_at_indices(arrayType array, arrayType indices, arrayType vals)
{
  for(int i=0; i < indices.size(); ++i){
	const size_t index_to_change = indices[i];
	array[index_to_change] = vals[i];
  }
}

template <typename arrayType, typename T>
__global__ void
set_vals_at_indices(T* array, arrayType indices, arrayType vals)
{
  for(int i=0; i < indices.size(); ++i){
	const size_t index_to_change = indices[i];
	array[index_to_change] = vals[i];
  }
}

TEST_CASE("Data can be set on the device and accessed on host")
{
	using int_array = gpu::cudaDynamicArray<int>;
	int_array array;
	array.Reserve(5);

	for (int i = 0; i < array.size(); ++i)
		array[i] = i;
  
	SECTION("Data can be set and accessed on host"){
		CHECK(array[0] == 0);
		CHECK(array[4] == 4);
	}
  
 
	constexpr int vals_to_edit = 3;
	std::array<int, vals_to_edit> indices = {1, 3, 4};
	std::array<int, vals_to_edit> vals = {11, 33, 44};
	
	int_array d_indices(indices.data(), indices.size());
	int_array d_vals(vals.data(), vals.size());

	SECTION("c-style array constructor works"){
		CHECK(d_indices[0] == 1);
		CHECK(d_indices[1] == 3);
		CHECK(d_indices[2] == 4);
		
		CHECK(d_vals[0] == 11);
		CHECK(d_vals[1] == 33);
		CHECK(d_vals[2] == 44);
	}
	
	set_vals_at_indices<int_array><<<1,1>>>(array, d_indices, d_vals);
	hipDeviceSynchronize();
	
	SECTION("Copy constructor makes deep-copy"){
		//passing by value to kernel invokes copy-constructor, which does deep, not shallow copy
		//thus values don't update when accessing the array on host
		CHECK(array[1] != 11);
		CHECK(array[3] != 33);
		CHECK(array[4] != 44);
	}
	
	set_vals_at_indices<int_array, int><<<1,1>>>(array.data(), d_indices, d_vals);
	hipDeviceSynchronize();
	
	SECTION("Can still access data on host after editing on device"){
		//if we pass pointer to that data (which is allocated in unified memory)
		//we can get update on the device properly
		CHECK(array[1] == 11);
		CHECK(array[3] == 33);
		CHECK(array[4] == 44);
	}
	
	int_array copy(array);
	SECTION("copy-constructor works"){
		for(int i=0; i < array.size(); ++i)
			CHECK(copy[i] == array[i]);
	}
}
