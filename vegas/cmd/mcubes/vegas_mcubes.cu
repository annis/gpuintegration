#include "hip/hip_runtime.h"
/*

code works for gaussian and sin using switch statement. device pointerr/template slow
down the code by 2x

chunksize needs to be tuned based on the ncalls. For now hardwired using a switch statement


nvcc -O2 -DCUSTOM -o vegas vegas_mcubes.cu -arch=sm_70
OR
nvcc -O2 -DCURAND -o vegas vegas_mcubes.cu -arch=sm_70

example run command

nvprof ./vegas 0 6 0.0  10.0  1.0E+09  10, 0, 0

nvprof  ./vegas 1 9 -1.0  1.0  1.0E+07 15 10 10

nvprof ./vegas 2 2 -1.0 1.0  1.0E+09 1 0 0

Last three arguments are: total iterations, iteration

*/
#include <chrono>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hiprand/hiprand_kernel.h>
#include <stdint.h>
#include <ctime>
#include <iostream>

#define WARP_SIZE 32
#define BLOCK_DIM_X 128
#define ALPH 1.5 //commented out by Ioannis in order to match python vegas default of .5
//#define ALPH 0.5
#define NDMX  500
#define MXDIM 20

#define NDMX1 NDMX+1
#define MXDIM1 MXDIM+1
#define PI 3.14159265358979323846
#include "xorshift.cu"

#define IMAX(a,b) \
    ({ __typeof__ (a) _a = (a); \
      __typeof__ (b) _b = (b); \
      _a > _b ? _a : _b; })

#define IMIN(a,b) \
    ({ __typeof__ (a) _a = (a); \
      __typeof__ (b) _b = (b); \
      _a < _b ? _a : _b; })


//Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError() {                                          \
        hipError_t e=hipGetLastError();                                 \
        if(e!=hipSuccess) {                                              \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
            exit(EXIT_FAILURE);                                           \
        }                                                                 \
}


#include "func.cuh"

int verbosity =0;

 using MilliSeconds =
    std::chrono::duration<double, std::chrono::milliseconds::period>;

template<typename T>
void PrintArray(T* array, int size, std::string label){
    printf("Will try to print v:%i\n", verbosity);
    if(verbosity == 0)
        return;
    for(int i=0; i< size; i++)
        std::cout<<label<<"["<<i<<"]:"<<array[i]<<"\n";
}

__inline__ __device__
double warpReduceSum(double val) {
	val += __shfl_down_sync(0xffffffff, val, 16, WARP_SIZE);
	val += __shfl_down_sync(0xffffffff, val, 8, WARP_SIZE);
	val += __shfl_down_sync(0xffffffff, val, 4, WARP_SIZE);
	val += __shfl_down_sync(0xffffffff, val, 2, WARP_SIZE);
	val += __shfl_down_sync(0xffffffff, val, 1, WARP_SIZE);
	return val;
}

__inline__ __device__
double blockReduceSum(double val) {

	static __shared__ double shared[32]; // Shared mem for 32 partial sums
	int lane = threadIdx.x % warpSize;
	int wid = threadIdx.x / warpSize;

	val = warpReduceSum(val);     // Each warp performs partial reduction

	if (lane == 0) 
        shared[wid] = val; // Write reduced value to shared memory

	__syncthreads();              // Wait for all partial reductions

	//read from shared memory only if that warp existed
	val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;

	if (wid == 0) val = warpReduceSum(val); //Final reduce within first warp

	return val;
}

__inline__ __device__  void get_indx(int ms, int *da, int ND, int NINTV) {
	int dp[MXDIM];
	int j, t0, t1;
	int m = ms;
	dp[0] = 1;
	dp[1] = NINTV;


	for (j = 0; j < ND - 2; j++) {
		dp[j + 2] = dp[j + 1] * NINTV;
	}
	//
	for (j = 0; j < ND; j++) {
		t0 = dp[ND - j - 1];
		t1 = m / t0;
		da[j] = 1 + t1;
		m = m - t1 * t0;

	}
}

__inline__ __device__  void get_indxN(int mc, int *da, int nd, int ng, double scc, double scic, double ing) {
	int kgt;
	for (int j = 0; j < nd - 1 ; j++) {
		kgt  = mc * scic ;
		mc = mc - kgt * scc ;
		scic = scic * ng;
		scc = scc * ing ;
		da[j] =  kgt + 1;
	}
	da[nd - 1] = mc + 1;

}

__inline__ __device__  void get_indxT(int mc, int *da, int nd, int ng, double scc, double scic, double ing) {
	int kgt;
	for (int j = 0; j < nd - 1 ; j++) {
		kgt  = mc * scic ;
		mc = mc - kgt * scc ;
		scic = scic * ng;
		scc = scc * ing ;
		da[j] =  kgt + 1;
	}
	da[nd - 1] = mc + 1;

}

__global__ void vegas_kernel(int ng, int ndim, int npg, double xjac, double dxg,
                             double *result_dev, double xnd, double *xi,
                             double *d, double *dx, double *regn, int ncubes,
                             int iter, double sc, double sci, double ing,
                             int chunkSize, uint32_t totalNumThreads,
                             int LastChunk, int fcode) {


#ifdef CUSTOM
	uint64_t temp;
	uint32_t a = 1103515245;
	uint32_t c = 12345;
	uint32_t one, expi;
	one = 1;
	expi = 31;
	uint32_t p = one << expi;
#endif


	uint32_t seed, seed_init;
	seed_init = (iter) * ncubes;



	int m = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = threadIdx.x;


	double fb, f2b, wgt, xn, xo, rc, f, f2, ran00;
	int kg[MXDIM + 1];
	int ia[MXDIM + 1];
	double x[MXDIM + 1];
	int k, j;
	double fbg, f2bg;
	//if(tx == 30 && blockIdx.x == 6771) printf("here m is %d\n", m);

	if (m < totalNumThreads) {
		if (m == totalNumThreads - 1) 
            chunkSize = LastChunk + 1;
		seed = seed_init + m * chunkSize;
#ifdef CURAND
		hiprandState localState;
		hiprand_init(seed, 0, 0, &localState);
#endif
		fbg = f2bg = 0.0;
		get_indx(m * chunkSize, &kg[1], ndim, ng);
		for (int t = 0; t < chunkSize; t++) {
			fb = f2b = 0.0;

			for ( k = 1; k <= npg; k++) {
				wgt = xjac;

				for ( j = 1; j <= ndim; j++) {
#ifdef CUSTOM
					temp =  a * seed + c;
					seed = temp & (p - 1);
					ran00 = (double) seed / (double) p ;
#endif
#ifdef CURAND
					ran00 = hiprand_uniform(&localState);
#endif

					xn = (kg[j] - ran00) * dxg + 1.0;
					ia[j] = IMAX(IMIN((int)(xn), NDMX), 1);

					if (ia[j] > 1) {
						xo = xi[j * NDMX1 + ia[j]] - xi[j * NDMX1 + ia[j] - 1];
						rc = xi[j * NDMX1 + ia[j] - 1] + (xn - ia[j]) * xo;
					} else {
						xo = xi[j * NDMX1 + ia[j]];
						rc = (xn - ia[j]) * xo;
					}

					x[j] = regn[j] + rc * dx[j];
                    
					wgt *= xo * xnd;
				}
				double tmp;

				switch (fcode) {
				case 0:
					tmp = (*func1)(x, ndim);
					break;
				case 1:
					tmp = (*func2)(x, ndim);
					break;
				case 2:
					tmp = (*func3)(x, ndim);
					break;			
                case 3:
                    tmp = (*BoxIntegral8_22)(x, ndim);
                    break;
                case 4:
                    tmp = (*GENZ1_8D)(x, ndim);
                    break;
                case 5:
                    tmp = (*GENZ2_2D)(x, ndim);
                    break;
                case 6:
                    tmp = (*GENZ2_6D)(x, ndim);
                    break;
                case 7:
                    tmp = (*GENZ3_3D)(x, ndim);
                    break;
                case 8:
                    tmp = (*GENZ4_5D)(x, ndim);
                    break;
                case 9:
                    tmp = (*GENZ5_8D)(x, ndim);
                    break;
                case 10:
                    tmp = (*BoxIntegral8_15)(x, ndim);
                    break;
                case 11:
                    tmp = (*GENZ6_6D)(x, ndim);
                    break;
                case 12:
                    tmp = (*GENZ4_8D)(x, ndim);
                    break;
                case 13:
                    tmp = (*GENZ3_8D)(x, ndim);
                    break;
                case 14:
                    tmp = sqsum(x, ndim);
                    break;
                case 15:
                    tmp = sumsqroot(x, ndim);
                    break;
                case 16:
                    tmp = prodones(x, ndim);
                    break;
                case 17:
                    tmp = prodexp(x, ndim);
                    break;
                case 18:
                    tmp = prodcub(x, ndim);
                    break;
                case 19:
                    tmp = prodx(x, ndim);
                    break;
                case 20:
                    tmp = sumfifj(x, ndim);
                    break;
                case 21:
                    tmp = sumfonefj(x, ndim);
                    break;
                case 22:
                    tmp = hellekalek(x, ndim);
                    break;
                case 23:
                    tmp = roosarnoldone(x, ndim);
                    break;
                case 24:
                    tmp = roosarnoldtwo(x, ndim);
                    break;
                case 25:
                    tmp = roosarnoldthree(x, ndim);
                    break;
                case 26:
                    tmp = rst(x, ndim);
                    break;
                case 27:
                    tmp = sobolprod(x, ndim);
                    break;
                case 28:
                    tmp = oscill(x, ndim);
                    break;
                case 29:
                    tmp = prpeak(x, ndim);
                    break;
                case 30:
                    tmp = sum(x, ndim);
                    break;
				default:
					tmp = (*func2)(x, ndim);
					break;
				}
                
				f = wgt * tmp;
				f2 = f * f;

				fb += f;
				f2b += f2;
#pragma unroll 2
				for ( j = 1; j <= ndim; j++) {
					atomicAdd(&d[ia[j]*MXDIM1 + j], fabs(f));
					//if(j == 1 && ia[j] == 1)
                    //    printf("For bin %i adding to index %i the value of %.8f x:%f, %f, %f\n", ia[j], ia[j]*MXDIM1 + j, fabs(f), x[1], x[2], x[3]);
				}

			}  // end of npg loop

			f2b = sqrt(f2b * npg);
			f2b = (f2b - fb) * (f2b + fb);

			fbg += fb;
			f2bg += f2b;

			for (int k = ndim; k >= 1; k--) {
				kg[k] %= ng;
				if (++kg[k] != 1) break;
			}

		} //end of chunk for loop

		fbg  = blockReduceSum(fbg);
		f2bg = blockReduceSum(f2bg);

		if (tx == 0) {
			atomicAdd(&result_dev[0], fbg);
			atomicAdd(&result_dev[1], f2bg);
		}
	} // end of subcube if
}

__global__ void vegas_kernelF(int ng, int ndim, int npg, double xjac, double dxg,
                              double *result_dev, double xnd, double *xi,
                              double *d, double *dx, double *regn, int ncubes,
                              int iter, double sc, double sci, double ing,
                              int chunkSize, uint32_t totalNumThreads,
                              int LastChunk, int fcode) {


#ifdef CUSTOM
	uint64_t temp;
	uint32_t a = 1103515245;
	uint32_t c = 12345;
	uint32_t one, expi;
	one = 1;
	expi = 31;
	uint32_t p = one << expi;
#endif


	uint32_t seed, seed_init;
	seed_init = (iter) * ncubes;



	int m = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = threadIdx.x;


	double fb, f2b, wgt, xn, xo, rc, f, f2, ran00;
	int kg[MXDIM + 1];
	//int ia[MXDIM + 1];
	int iaj;
	double x[MXDIM + 1];
	int k, j;
	double fbg, f2bg;

	if (m < totalNumThreads) {
		if (m == totalNumThreads - 1) chunkSize = LastChunk + 1;
		seed = seed_init + m * chunkSize;
#ifdef CURAND
		hiprandState localState;
		hiprand_init(seed, 0, 0, &localState);
#endif
		fbg = f2bg = 0.0;
		get_indx(m * chunkSize, &kg[1], ndim, ng);
		for (int t = 0; t < chunkSize; t++) {
			fb = f2b = 0.0;
			//get_indx(m * chunkSize + t, &kg[1], ndim, ng);

			for ( k = 1; k <= npg; k++) {
				wgt = xjac;

				for ( j = 1; j <= ndim; j++) {
#ifdef CUSTOM
					temp =  a * seed + c;
					seed = temp & (p - 1);
					ran00 = (double) seed / (double) p ;
#endif
#ifdef CURAND
					ran00 = hiprand_uniform(&localState);
#endif

					xn = (kg[j] - ran00) * dxg + 1.0;
					iaj = IMAX(IMIN((int)(xn), NDMX), 1);

					if (iaj > 1) {
						xo = xi[j * NDMX1 + iaj] - xi[j * NDMX1 + iaj - 1];
						rc = xi[j * NDMX1 + iaj - 1] + (xn - iaj) * xo;
					} else {
						xo = xi[j * NDMX1 + iaj];
						rc = (xn - iaj) * xo;
					}

					//x[j] = regn[1] + rc * dx[1]; //not sure why it was like this
					x[j] = regn[j] + rc * dx[j];

					wgt *= xo * xnd;
				}

				double tmp;

				switch (fcode) {
				case 0:
					tmp = (*func1)(x, ndim);
					break;
				case 1:
					tmp = (*func2)(x, ndim);
					break;
				case 2:
					tmp = (*func3)(x, ndim);
					break;			
                case 3:
                    
                    tmp = (*BoxIntegral8_22)(x, ndim);
                    break;
                case 4:
                    tmp = (*GENZ1_8D)(x, ndim);
                    break;
                case 5:
                    tmp = (*GENZ2_2D)(x, ndim);
                    break;
                case 6:
                    tmp = (*GENZ2_6D)(x, ndim);
                    break;
                case 7:
                    tmp = (*GENZ3_3D)(x, ndim);
                    break;
                case 8:
                    tmp = (*GENZ4_5D)(x, ndim);
                    break;
                case 9:
                    tmp = (*GENZ5_8D)(x, ndim);
                    break;
                case 10:
                    tmp = (*BoxIntegral8_15)(x, ndim);
                    break;
                case 11:
                    tmp = (*GENZ6_6D)(x, ndim);
                    break;
                case 12:
                    tmp = (*GENZ4_8D)(x, ndim);
                    break;
                case 13:
                    tmp = (*GENZ3_8D)(x, ndim);
                    break;
                case 14:
                    tmp = wgt * sqsum(x, ndim);
                    break;
                case 15:
                    tmp = wgt * sumsqroot(x, ndim);
                    break;
                case 16:
                    tmp = wgt * prodones(x, ndim);
                    break;
                case 17:
                    tmp = wgt * prodexp(x, ndim);
                    break;
                case 18:
                    tmp = wgt * prodcub(x, ndim);
                    break;
                case 19:
                    tmp = wgt * prodx(x, ndim);
                    break;
                case 20:
                    tmp = wgt * sumfifj(x, ndim);
                    break;
                case 21:
                    tmp = wgt * sumfonefj(x, ndim);
                    break;
                case 22:
                    tmp = wgt * hellekalek(x, ndim);
                    break;
                case 23:
                    tmp = wgt * roosarnoldone(x, ndim);
                    break;
                case 24:
                    tmp = wgt * roosarnoldtwo(x, ndim);
                    break;
                case 25:
                    tmp = wgt * roosarnoldthree(x, ndim);
                    break;
                case 26:
                    tmp = wgt * rst(x, ndim);
                    break;
                case 27:
                    tmp = wgt * sobolprod(x, ndim);
                    break;
                case 28:
                    tmp = wgt * oscill(x, ndim);
                    break;
                case 29:
                    tmp = wgt * prpeak(x, ndim);
                    break;
                case 30:
                    tmp = wgt * sum(x, ndim);
                    break;
				default:
					tmp = (*func2)(x, ndim);
					break;
				}
            
                
				f = wgt * tmp; //is this f(x)/p(x)?
				f2 = f * f; //this is (f(x)/p(x))^2 in equation 2.

				fb += f;
				f2b += f2;


			}  // end of npg loop

			f2b = sqrt(f2b * npg);
            f2b = (f2b - fb) * (f2b + fb); //this is equivalent to s^(2) - (s^(1))^2  
            
			fbg += fb;
			f2bg += f2b;

			for (int k = ndim; k >= 1; k--) {
				kg[k] %= ng;
				if (++kg[k] != 1) break;
			}

		} //end of chunk for loop

		fbg  = blockReduceSum(fbg);
		f2bg = blockReduceSum(f2bg);


		if (tx == 0) {
			atomicAdd(&result_dev[0], fbg);
			atomicAdd(&result_dev[1], f2bg);
		}
        
        


	} // end of subcube if

}

void rebin(double rc, int nd, double r[], double xin[], double xi[]){
    
    
    //--------------------------------
    //Assumptions
    //dr is the remaining distance to cover in the axis that still needs to be assigned to bins
    //xin is the length we have already assigned
    //what is r?
    //--------------------------------
    
	int i, k = 0;
	double dr = 0.0, xn = 0.0, xo = 0.0;
    
	for (i = 1; i < nd; i++) {
        
        //printf("BIN:%i\n", i);
        //printf("\tEvaluating rc>dr :%i\n", rc > dr);
        
		while (rc > dr){
			dr += r[++k];
            //printf("\tsetting dr:%f r[%i]:%f\n", dr, k, r[k]);
        }
        
		if (k > 1) 
            xo = xi[k - 1];
        
        //printf("\tSetting xn to %f\n", xi[k]);
        //printf("\tSetting dr to %f\n", dr);
        //printf("\txo:%.8f\n", xo);
        //printf("\tr[%i]:%.8f\n", k, r[k]);
        
		xn = xi[k];
		dr -= rc;
        
        //printf("\tSetting xin[%i] to %f\n", i, xn - (xn - xo) * dr / r[k]);
        
		xin[i] = xn - (xn - xo) * dr / r[k];
        
	}

	for (i = 1; i < nd; i++) 
        xi[i] = xin[i];
	xi[nd] = 1.0;

}

void vegas(double regn[], int ndim, int fcode,
           double ncall, double *tgral, double *sd,
           double *chi2a, int titer, int itmax, int skip)
{
	int i, it, j, k, nd, ndo, ng, npg, ncubes;
	double calls, dv2g, dxg, rc, ti, tsi, wgt, xjac, xn, xnd, xo;

	double schi, si, swgt;
	double result[2];
	double *d, *dt, *dx, *r, *x, *xi, *xin;
	int *ia;

	d = (double*)malloc(sizeof(double) * (NDMX + 1) * (MXDIM + 1)) ;
	dt = (double*)malloc(sizeof(double) * (MXDIM + 1)) ;
	dx = (double*)malloc(sizeof(double) * (MXDIM + 1)) ;
	r = (double*)malloc(sizeof(double) * (NDMX + 1)) ;
	x = (double*)malloc(sizeof(double) * (MXDIM + 1)) ;
	xi = (double*)malloc(sizeof(double) * (MXDIM + 1) * (NDMX + 1)) ;
	xin = (double*)malloc(sizeof(double) * (NDMX + 1)) ;
	ia = (int*)malloc(sizeof(int) * (MXDIM + 1)) ;


// code works only  for (2 * ng - NDMX) >= 0)
	
	ndo = 1;
	for (j = 1; j <= ndim; j++) 
        xi[j * NDMX1 + 1] = 1.0;
	si = swgt = schi = 0.0;
	nd = NDMX;
	ng = 1;
	ng = (int)pow(ncall / 2.0 /*+ 0.25*/, 1.0 / ndim); //why do we add .25?
	for (k = 1, i = 1; i < ndim; i++) 
        k *= ng;
	double sci = 1.0 / k;
	double sc = k;
	k *= ng;
	ncubes = k;
	npg = IMAX(ncall / k, 2);
	calls = (double)npg * (double)k;
    //printf("actual number of calls:%f\n", calls);
	dxg = 1.0 / ng;
	double ing = dxg;
	for (dv2g = 1, i = 1; i <= ndim; i++) 
        dv2g *= dxg;
	dv2g = (calls * dv2g * calls * dv2g) / npg / npg / (npg - 1.0);
	xnd = nd;
	dxg *= xnd;
	xjac = 1.0 / calls;
	for (j = 1; j <= ndim; j++) {
		dx[j] = regn[j + ndim] - regn[j];
		xjac *= dx[j];
	}

	for (i = 1; i <= IMAX(nd, ndo); i++) 
        r[i] = 1.0;
	for (j = 1; j <= ndim; j++) 
        rebin(ndo / xnd, nd, r, xin, &xi[j * NDMX1]);
	ndo = nd;



	double *d_dev, *dx_dev, *x_dev, *xi_dev, *regn_dev,  *result_dev;
	int *ia_dev;

	hipMalloc((void**)&result_dev, sizeof(double) * 2); cudaCheckError();
	hipMalloc((void**)&d_dev, sizeof(double) * (NDMX + 1) * (MXDIM + 1)); cudaCheckError();
	hipMalloc((void**)&dx_dev, sizeof(double) * (MXDIM + 1)); cudaCheckError();
	hipMalloc((void**)&x_dev, sizeof(double) * (MXDIM + 1)); cudaCheckError();
	hipMalloc((void**)&xi_dev, sizeof(double) * (MXDIM + 1) * (NDMX + 1)); cudaCheckError();
	hipMalloc((void**)&regn_dev, sizeof(double) * ((ndim * 2) + 1)); cudaCheckError();
	hipMalloc((void**)&ia_dev, sizeof(int) * (MXDIM + 1)); cudaCheckError();

	hipMemcpy( dx_dev, dx, sizeof(double) * (MXDIM + 1), hipMemcpyHostToDevice) ; cudaCheckError();
	hipMemcpy( x_dev, x, sizeof(double) * (MXDIM + 1), hipMemcpyHostToDevice) ; cudaCheckError();
	hipMemcpy( regn_dev, regn, sizeof(double) * ((ndim * 2) + 1), hipMemcpyHostToDevice) ; cudaCheckError();

	hipMemset(ia_dev, 0, sizeof(int) * (MXDIM + 1));

	int chunkSize;

	switch (fcode) {
	case 0:
		chunkSize = 2048;
		break;
	case 1:
		chunkSize = 32;
		break;
	case 2:
		chunkSize = 2048;
		break;		
	default:
		//chunkSize = 2048;
        chunkSize = 32;
        //chunkSize = 1;
		break;
	}

	uint32_t totalNumThreads = (uint32_t) ((ncubes + chunkSize - 1) / chunkSize);
	uint32_t totalCubes = totalNumThreads * chunkSize;
	int extra = totalCubes - ncubes;
	int LastChunk = chunkSize - extra;
	uint32_t nBlocks = ((uint32_t) (((ncubes + BLOCK_DIM_X - 1) / BLOCK_DIM_X)) / chunkSize) + 1;
	uint32_t nThreads = BLOCK_DIM_X;
    
    std::cout<<"ncubes:"<<ncubes<<"\n";
    std::cout<<"npg:"<<npg<<"\n";
    std::cout<<"npg*ncubes*chunkSize:"<<npg*ncubes*chunkSize<<"\n";
    std::cout<<"totalNumThreads:"<<totalNumThreads<<"\n";
	for (it = 1; it <= itmax; it++) {

		ti = tsi = 0.0;
		for (j = 1; j <= ndim; j++) {
			for (i = 1; i <= nd; i++) d[i * MXDIM1 + j] = 0.0;
		}
        
		hipMemcpy( xi_dev, xi, sizeof(double) * (MXDIM + 1) * (NDMX + 1), hipMemcpyHostToDevice) ; cudaCheckError();	//bin bounds
		hipMemset(d_dev, 0, sizeof(double) * (NDMX + 1) * (MXDIM + 1));	//bin contributions
		hipMemset(result_dev, 0, 2 * sizeof(double));
        //std::cout<<"Launchign with "<<nBlocks<<","<<nThreads<<std::endl;
        
        std::cout<<"---------------------------------------\n";
        //PrintArray<double>(xi, (MXDIM + 1) * (NDMX + 1), "xi");
        
		vegas_kernel <<< nBlocks, nThreads>>>(ng, ndim, npg, xjac, dxg, result_dev, xnd,
		                                      xi_dev, d_dev, dx_dev, regn_dev, ncubes, it, sc,
		                                      sci,  ing, chunkSize, totalNumThreads,
		                                      LastChunk, fcode);


		hipMemcpy(xi, xi_dev, sizeof(double) * (MXDIM + 1) * (NDMX + 1), hipMemcpyDeviceToHost); cudaCheckError();	//is this necessary? the kernel doesn't change xi_dev
		hipMemcpy( d, d_dev,  sizeof(double) * (NDMX + 1) * (MXDIM + 1), hipMemcpyDeviceToHost) ; cudaCheckError();	//we do need to the contributions for the rebinning

		hipMemcpy(result, result_dev, sizeof(double) * 2, hipMemcpyDeviceToHost);
        
        //PrintArray<double> (d, (MXDIM + 1) * (NDMX + 1), "d");

		//printf("ti is %f", ti);
		ti  = result[0];
		tsi = result[1];
   
		tsi *= dv2g;
		//printf("iter = %d  integ = %e   std = %e\n", it, ti, sqrt(tsi));

		if (it > skip) {
			wgt = 1.0 / tsi;
			si += wgt * ti;
			schi += wgt * ti * ti;
			swgt += wgt;
			*tgral = si / swgt;
			*chi2a = (schi - si * (*tgral)) / (it - 0.9999);
			if (*chi2a < 0.0) *chi2a = 0.0;
			*sd = sqrt(1.0 / swgt);
			tsi = sqrt(tsi);
			printf("%5d,   %14.7g, -%9.2g,  %9.2g\n", it, *tgral, *sd, *chi2a);
		}
        
        std::cout<<"Rebining Process\n";
        
		for (j = 1; j <= ndim; j++) {
            
			xo = d[1 * MXDIM1 + j]; //bin 1 of dim j, and bin 2 just below           
			xn = d[2 * MXDIM1 + j];                                     
            
            //printf("Contribution of bin 1:%.8f\n", xo);
            //printf("Contribution of bin 2:%.8f\n", xn);
            
			d[1 * MXDIM1 + j] = (xo + xn) / 2.0;                        
            //printf("Storing their average in the spot of contribution for bin 1\n");
            
			dt[j] = d[1 * MXDIM1 + j];       //set dt sum to contribution of bin 1                           
            
            
            //printf("Going through %i bins starting at the second one (i:2)\n", nd);
			for (i = 2; i < nd; i++) {
                //rc is the contribution of the first and last bin? why?
				rc = xo + xn;                                           
                
				xo = xn;                                                
                
				xn = d[(i + 1) * MXDIM1 + j];                           
                
                //printf("Contribution of bin A:%.8f\n", xo);
                //printf("contribution of bin B:%.8f\n", xn);
                
				d[i * MXDIM1 + j] = (rc + xn) / 3.0;                    
                //printf("updating with new three way average the contribution of bin %i\n", i);
                
                
				dt[j] += d[i * MXDIM1 + j];                                
                
                
			}
            
            //do bin nd last
			d[nd * MXDIM1 + j] = (xo + xn) / 2.0;                      
            
			dt[j] += d[nd * MXDIM1 + j];                                
            
		}
        
        //printf("DIM: after summation\n");
        //for(int j = 0; j < (MXDIM + 1) * (NDMX + 1); j++)
        //    printf("d[%i]:%.8f\n", j, d[j]);
        
		for (j = 1; j <= ndim; j++) {
			//printf("Checking if dt[%i] is greater than 0:%.8f\n", j, dt[j]);
            if (dt[j] > 0.0) {  //enter if there is any contribution only
				rc = 0.0;
                //printf("Setting rc to zero\n");
				for (i = 1; i <= nd; i++) {
					//if (d[i * MXDIM1 + j] < TINY) d[i * MXDIM1 + j] = TINY;
                    //printf("Setting r[%i] to %f\n", i, pow((1.0 - d[i * MXDIM1 + j] / dt[j]) /(log(dt[j]) - log(d[i * MXDIM1 + j])), ALPH));
                               
					r[i] = pow((1.0 - d[i * MXDIM1 + j] / dt[j]) /(log(dt[j]) - log(d[i * MXDIM1 + j])), ALPH);
                    //r[i] = pow((d[i * MXDIM1 + j] / dt[j] - 1.) /(log(d[i * MXDIM1 + j])-log(dt[j])), ALPH);
                    //printf("Incrementing rc by r[%i]:%f -> rc:%f\n", i, r[i], rc+r[i]);           
					rc += r[i]; //rc is it the total number of sub-increments
                    //printf("r[%i]:%.8f\n", i, r[i]);        //is r[i] the new weight of each bin (instead of the number of sub-increments?
				}
                
                
                //printf("Calling rebin rc/xnd:%.8f xnd:%.8f rc:%.8f\n", rc/xnd, xnd, rc);
				rebin(rc / xnd, nd, r, xin, &xi[j * NDMX1]);
			}

		}

	}  // end of iterations

	//  Start of iterations without adjustment

	hipMemcpy( xi_dev, xi, sizeof(double) * (MXDIM + 1) * (NDMX + 1), hipMemcpyHostToDevice) ; cudaCheckError();

	for (it = itmax+1; it <= titer; it++) {

		ti = tsi = 0.0;

		hipMemset(result_dev, 0, 2 * sizeof(double));
        
		vegas_kernelF <<< nBlocks, nThreads>>>(ng, ndim, npg, xjac, dxg, result_dev, xnd,
		                                       xi_dev, d_dev, dx_dev, regn_dev, ncubes, it, sc,
		                                       sci,  ing, chunkSize, totalNumThreads,
		                                       LastChunk, fcode);


		hipMemcpy(result, result_dev, sizeof(double) * 2, hipMemcpyDeviceToHost);

		//printf("ti is %f", ti);
		ti  = result[0];
		tsi = result[1];
		tsi *= dv2g; //is dv2g 1/(M-1)?
		//printf("iter %d  integ = %.15e   std = %.15e\n", it, ti, sqrt(tsi));

		wgt = 1.0 / tsi;
		si += wgt * ti;
		schi += wgt * ti * ti;
		swgt += wgt;
		*tgral = si / swgt;
		*chi2a = (schi - si * (*tgral)) / (it - 0.9999);
		if (*chi2a < 0.0) *chi2a = 0.0;
		*sd = sqrt(1.0 / swgt);
		tsi = sqrt(tsi);
		//printf("it %d\n", it);
		printf("%5d   %14.7g+/-%9.4g  %9.2g\n", it, *tgral, *sd, *chi2a);
		//printf("%3d   %e  %e\n", it, ti, tsi);

	}  // end of iterations



	free(d);
	free(dt);
	free(dx);
	free(ia);
	free(x);
	free(xi);

	hipFree(d_dev);
	hipFree(dx_dev);
	hipFree(ia_dev);
	hipFree(x_dev);
	hipFree(xi_dev);
	hipFree(regn_dev);



}

int main(int argc, char **argv)
{

	if (argc < 9) {
		printf( "****************************************\n"
		        "Usage (6 arguments):\n"
		        "./vegas_mcubes FCODE  DIM LL  UL  NCALLS  SKIP\n"
		        "FCODE = 0 to MAX_NUMBER_OF_FUNCTIONS-1\n"
		        "NCALLS in scientific notation, e.g. 1.0E+07 \n"
		        "****************************************\n");
		exit(-1);
	}
    
	int  j;
	double avgi, chi2a, sd;
	double regn[2 * MXDIM + 1];

	int fcode = atoi(argv[1]);
	int ndim = atoi(argv[2]);
	float LL = atof(argv[3]);
	float UL = atof(argv[4]);
	double ncall = atof(argv[5]);
	int titer = atoi(argv[6]);
	int itmax = atoi(argv[7]);
	int skip = atoi(argv[8]);
    verbosity = atoi(argv[9]);
    
    std::cout<<"Ncall:"<<ncall<<"\n";
    std::cout<<"verbosity:"<<verbosity<<"\n";
    auto t0 = std::chrono::high_resolution_clock::now();
	avgi = sd = chi2a = 0.0;
    
	for (j = 1; j <= ndim; j++) {
		regn[j] = LL;
		regn[j + ndim] = UL;
	}

    
	vegas(regn, ndim, fcode, ncall, &avgi, &sd, &chi2a, titer, itmax, skip);
    MilliSeconds dt = std::chrono::high_resolution_clock::now() - t0;

	//printf("Number of iterations performed: %d\n", itmax);

	//printf("Integral, Standard Dev., Chi-sq. = %.18f %.20f% 12.6f\n",avgi, sd, chi2a);
    std::cout.precision(15);
    std::cout << fcode << ","
            << std::scientific << avgi << "," 
             << std::scientific << sd << "," 
             << titer << "," 
             << itmax << "," 
             << skip << "," 
             << ncall << ","
             << chi2a << ","
             << dt.count() << "\n";
    
    printf("Absolute error %.15e\n", abs(1.084656084656085e-02 - avgi));
	return 0;

}


