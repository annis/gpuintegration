#include "function.cuh"
#include "demo_utils.cuh"
#include <chrono>
#include <cmath>
#include <fstream>
#include <iomanip>
#include <iostream>

using namespace quad;

int
main()
{
  double epsrel = 1.0e-3; // starting error tolerance.
  double const epsrel_min = 1.0e-12;
  double true_value = 0.039462780237263662026;
  GENZ_5_2D integrand;
  
  constexpr int ndim = 2;
  Config configuration;
  configuration.outfileVerbosity = 0;
  configuration.heuristicID = 4;
  
  PrintHeader();
  while (cu_time_and_call<GENZ_5_2D, ndim>("pdc_f1_latest",
                       integrand,
                       epsrel,
                       true_value,
                       "gpucuhre",
                       std::cout,
                       configuration) == true &&
         epsrel >= epsrel_min) {
    epsrel /= 5.0;
  }
}
