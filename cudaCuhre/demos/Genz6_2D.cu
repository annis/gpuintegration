#include "function.cuh"
#include "demo_utils.cuh"
#include <chrono>
#include <cmath>
#include <fstream>
#include <iomanip>
#include <iostream>

using namespace quad;

int
main()
{
  double epsrel = 1.0e-3; // starting error tolerance.
  double const epsrel_min = 1.0e-12;
  double true_value = 120489.75982636053604;
  GENZ_6_2D integrand;
  std::cout << "id, value, epsrel, epsabs, estimate, errorest, regions, "
             "converge, final, total_time\n";
  constexpr int ndim = 2;
  Config configuration;
  configuration.outfileVerbosity = 0;
  configuration.heuristicID = 4;
  
  while (cu_time_and_call<GENZ_6_2D, ndim>("pdc_f1_latest",
                       integrand,
                       epsrel,
                       true_value,
                       "gpucuhre",
                       std::cout,
                       configuration) == true &&
         epsrel >= epsrel_min) {
    epsrel /= 5.0;
  }


}
