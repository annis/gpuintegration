#include "hip/hip_runtime.h"
#include "demos/function.cuh"
#include "cudaCuhre/demos/demo_utils.cuh"
#include <chrono>
#include <cmath>
#include <fstream>
#include <iomanip>
#include <iostream>

using namespace quad;

namespace detail{
    class BoxIntegral8_22 {
        public:
          __device__ __host__ double
          operator()(double x,
                     double y,
                     double z,
                     double k,
                     double l,
                     double m,
                     double n,
                     double o)
          {
            double s = 22;
            double sum = 0;
            sum = pow(x, 2) + pow(y, 2) + pow(z, 2) + pow(k, 2) + pow(l, 2) +
                  pow(m, 2) + pow(n, 2) + pow(o, 2);
            return pow(sum, s / 2)/1495369.283757217694;
          }
    };
}

int
main()
{
  double epsrel = 3.2e-7;//1.0e-3; // starting error tolerance.
  double const epsrel_min = 1.0240000000000002e-10;
  double true_value = 1.0;
  constexpr int ndim = 8;
  detail::BoxIntegral8_22 integrand;
    
  Config configuration;
  configuration.outfileVerbosity = 0;
  int heuristics[3] = {0, 2,4};
  
  PrintHeader();
  for(int i=2; i>=0; i--){
      epsrel = 1.0e-3;
      configuration.heuristicID = heuristics[i];
      while (cu_time_and_call<detail::BoxIntegral8_22, ndim>("B8_22",
                           integrand,
                           epsrel,
                           true_value,
                           "gpucuhre",
                           std::cout,
                           configuration) == true &&
             epsrel > epsrel_min) {
        epsrel /= 5.0;
      }
  }
}
