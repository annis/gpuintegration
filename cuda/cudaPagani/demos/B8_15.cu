#include "hip/hip_runtime.h"
#include "cuda/cudaPagani/demos/demo_utils.cuh"
#include "cuda/cudaPagani/demos/function.cuh"
#include <chrono>
#include <cmath>
#include <fstream>
#include <iomanip>
#include <iostream>

using namespace quad;

namespace detail {
  class BoxIntegral8_15 {
  public:
    __device__ __host__ double
    operator()(double x,
               double y,
               double z,
               double k,
               double l,
               double m,
               double n,
               double o)
    {

      double s = 15;
      double sum = 0;
      sum = pow(x, 2) + pow(y, 2) + pow(z, 2) + pow(k, 2) + pow(l, 2) +
            pow(m, 2) + pow(n, 2) + pow(o, 2);
      return pow(sum, s / 2);
    }
  };
}

int
main()
{
  double epsrel = 1.0e-3; // starting error tolerance.
  double true_value = 8879.851175413485;
  double const epsrel_min = 1.0240000000000002e-10;
  detail::BoxIntegral8_15 integrand;
  constexpr int ndim = 8;

  Config configuration;
  configuration.outfileVerbosity = 0;
  // configuration.heuristicID = 0;
  // configuration.phase_2 = true;

  PrintHeader();
  while (cu_time_and_call<detail::BoxIntegral8_15, ndim>("8D f8",
                                                         integrand,
                                                         epsrel,
                                                         true_value,
                                                         "gpucuhre",
                                                         std::cout,
                                                         configuration) ==
           true &&
         epsrel > epsrel_min) {
    epsrel /= 5.0;
  }
}
