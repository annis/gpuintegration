#include "hip/hip_runtime.h"
#include "cudaCuhre/demos/function.cuh"
#include "cudaCuhre/demos/demo_utils.cuh"
#include <chrono>
#include <cmath>
#include <fstream>
#include <iomanip>
#include <iostream>

using namespace quad;

namespace detail{
    class GENZ_5_8D {
    public:
      __device__ __host__ double
      operator()(double x, double y, double z, double k, double m, double n, double p, double q)
      {
        double beta = .5;
        double t1 = -10.*fabs(x - beta) - 10.* fabs(y - beta) - 10.* fabs(z - beta) - 10.* fabs(k - beta) - 10.* fabs(m - beta) - 10.* fabs(n - beta) - 10.* fabs(p - beta) - 10.* fabs(q - beta);
        return exp(t1);
      }
    };
}

int
main()
{
  double epsrel = 1.0e-3; // starting error tolerance.
  double const epsrel_min = 1.024e-10;
  double true_value = 2.425217625641885e-06;
  detail::GENZ_5_8D integrand;
  
  constexpr int ndim = 8;
  Config configuration;
  configuration.outfileVerbosity = 0;  
  //configuration.heuristicID = 0;
  //configuration.phase_2 = true;
  PrintHeader();

  while(cu_time_and_call<detail::GENZ_5_8D, ndim>("GENZ5_8D",
                                                integrand,
                                                epsrel,
                                                true_value,
                                                "gpucuhre",
                                                std::cout,
                                                configuration) == true && epsrel > epsrel_min){
        epsrel /= 5;    
  }
}
