#include "hip/hip_runtime.h"
#include "demo_utils.cuh"
#include "function.cuh"
#include <chrono>
#include <cmath>
#include <fstream>
#include <iomanip>
#include <iostream>

using namespace quad;

namespace detail {
  class GENZ_4_5D {
  public:
    __device__ __host__ float
    operator()(float x, float y, float z, float w, float v)
    {
      float beta = .5;
      return exp(
        -1.0 * (pow(25, 2) * pow(x - beta, 2) + pow(25, 2) * pow(y - beta, 2) +
                pow(25, 2) * pow(z - beta, 2) + pow(25, 2) * pow(w - beta, 2) +
                pow(25, 2) * pow(v - beta, 2)));
    }
  };
}

int
main()
{
  float epsrel = 1.e-3;
  float const epsrel_min = 1.0240000000000002e-10;
  float true_value = 1.79132603674879e-06;
  detail::GENZ_4_5D integrand;
  PrintHeader();
  constexpr int ndim = 5;
  Config configuration;
  configuration.outfileVerbosity = 0;
  // configuration.heuristicID = 0;
  // configuration.phase_2 = false;
  while (floatIntegrands::cu_time_and_call<detail::GENZ_4_5D, ndim>(
           "5D f4",
           integrand,
           epsrel,
           true_value,
           "gpucuhre",
           std::cout,
           configuration) == true &&
         epsrel > epsrel_min) {
    epsrel /= 5.0;
    break;
  }
}
