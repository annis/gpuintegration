#include "hip/hip_runtime.h"
#include "catch2/catch.hpp"
#include "modules/sigma_miscent_y1_scalarintegrand.hh"

#include <iostream>					//to overload >> in quad::Interp2D
#include "utils/str_to_doubles.hh"  //to utilize inside overloaded quad::Interp2D >> operator
#include <vector> 					

#include <fstream>
#include <stdexcept>
#include <string>

//using namespace y3_cluster;

namespace quad {
	
  class Managed 
{
public:
  void *operator new(size_t len) {
    void *ptr;
    hipMallocManaged(&ptr, len);
    hipDeviceSynchronize();
    return ptr;
  }

  void operator delete(void *ptr) {
    hipDeviceSynchronize();
    hipFree(ptr);
  }
};	
	
  class Interp2D : public Managed{
  
  public:
    __device__ __host__
    Interp2D(){};
	
    double* interpT;
    double* interpR;
    double* interpC;
    size_t _rows;
    size_t _cols;
	
    __host__ __device__
	Interp2D(double* xs, double* ys, double* zs, size_t cols, size_t rows){
		printf("Constructor called\n");
		//hipMalloc((void**)&interpR, sizeof(double)*rows);
		//hipMalloc((void**)&interpC, sizeof(double)*cols);
		//hipMalloc((void**)&interpT, sizeof(double)*rows*cols);
		
		//hipMemcpy(interpR, ys, sizeof(double)*rows, hipMemcpyHostToDevice);
		//hipMemcpy(interpC, xs, sizeof(double)*cols, hipMemcpyHostToDevice);
		//hipMemcpy(interpT, zs, sizeof(double)*rows*cols, hipMemcpyHostToDevice);
		memcpy(interpR, ys, sizeof(double)*rows);
		memcpy(interpC, xs, sizeof(double)*cols);
		memcpy(interpT, zs, sizeof(double)*rows*cols);
		
		_rows = rows;
		_cols = cols;
    }
	
	__device__ __host__
	bool AreNeighbors(const double val, double* arr, const size_t leftIndex, const size_t RightIndex) const{
		if(arr[leftIndex] < val && arr[RightIndex] > val)
			return true;
		return false;
	}
	
	friend std::istream&
    operator>>(std::istream& is, Interp2D& interp)
    {
      assert(is.good());
      std::string buffer;
      std::getline(is, buffer);
      std::vector<double> xs = cosmosis::str_to_doubles(buffer);
      std::getline(is, buffer);
      std::vector<double> ys = cosmosis::str_to_doubles(buffer);
      std::getline(is, buffer);
      std::vector<double> zs  = cosmosis::str_to_doubles(buffer);
	  
	  interp._cols = xs.size();
	  interp._rows = ys.size();
	  hipMallocManaged((void**)&(*&interp), sizeof(Interp2D));

	  hipMallocManaged((void**)&interp.interpR, sizeof(double)*ys.size());
	  hipDeviceSynchronize();
	  hipMallocManaged((void**)&interp.interpC, sizeof(double)*xs.size());
	  hipDeviceSynchronize();
	  hipMallocManaged((void**)&interp.interpT, sizeof(double)*zs.size());
	  hipDeviceSynchronize();
	  
	  memcpy(interp.interpR, ys.data(), sizeof(double)*ys.size());
	  memcpy(interp.interpC, xs.data(), sizeof(double)*xs.size());
	  memcpy(interp.interpT, zs.data(), sizeof(double)*zs.size());
	  
	  /*for(int i=0; i< interp._rows; i++)
		  printf("ys[%i]:%f\n", i, interp.interpR[i]);
	  for(int i=0; i< interp._cols; i++)
		  printf("xs[%i]:%f\n", i, interp.interpC[i]);
	  for(int i=0; i< zs.size(); i++)
		  printf("zs[%i]:%f\n", i, interp.interpT[i]);*/
	  
      return is;
    }
	
	__host__ __device__
	Interp2D(const Interp2D &source) {
		printf("Copy constructor called\n");
		interpT = source.interpT;
		interpC = source.interpC;
		interpR = source.interpR;
		_cols = source._cols;
		_rows = source._rows;
	} 
	
	//what to do if extrapolation is attempted?
	__device__ __host__
	void FindNeighbourIndices(const double val, double* arr, const size_t size, size_t& leftI, size_t& rightI) const{
		//assert for improper sizes?
		size_t currentIndex = size/2;
		size_t lastIndex = size - 1;
		leftI = 0;
		rightI = size - 1;
		
		//for(size_t i=0; i<size; ++i)
		//	printf("arr[%lu]:%f\n", i, arr[i]);
		
		while(currentIndex != 0 && currentIndex != lastIndex){
			currentIndex = leftI + (rightI - leftI)/2;
			//printf("currentIndex:%lu looking for %f within %lu range\n", currentIndex, val, size);
			if(AreNeighbors(val, arr, currentIndex-1, currentIndex)){
				leftI = currentIndex -1;
				rightI = currentIndex;
				return;
			}
			
			//printf("%f vs %f\n", arr[currentIndex], val);
			
			if(arr[currentIndex] > val){
				//printf("changing rightI from %lu to %lu\n", rightI, currentIndex);
				rightI = currentIndex;
			}
			else{
				//printf("changing leftI from %lu to %lu\n", leftI, currentIndex);
				leftI = currentIndex;
			}
			//currentIndex = arr[currentIndex] > val ? currentIndex /= 2 : currentIndex + (size-currentIndex)/2;
			//currentIndex = arr[currentIndex] > val ? (currentIndex-leftI) / 2 : currentIndex + (size-currentIndex)/2;
			
		}
		
		//values can't be found, how to handle?
		leftI  = 0;
		rightI = 0;
	}
	
    __device__ __host__ double
    operator()(double x, double y) const
    {
	  //y1, y2, x1, x2, are the indices of where to find the four neighbouring points in the z-table
	  size_t y1 = 0, y2 = 0;
	  size_t x1 = 0, x2 = 0;
		
	  FindNeighbourIndices(y, interpR, _rows, y1, y2);
	  FindNeighbourIndices(x, interpC, _cols, x1, x2);
	  
	  //printf("coordinates: %lu, %lu, %lu, %lu\n", y1, y2, x1, x2);
	  //this is how  zij is accessed by gsl2.6 Interp2D i.e. zij = z[j*xsize+i], where i=0,...,xsize-1, j=0, ..., ysize-1
	  const double q11 = interpT[y1*_cols + x1];
	  const double q12 = interpT[y2*_cols + x1];
	  const double q21 = interpT[y1*_cols + x2];
	  const double q22 = interpT[y2*_cols + x2];
	  //printf("values at coordinats:%.8f, %.8f, %.8f, %.8f\n", q11, q12, q21, q22);
	  
	  const double x1_val = interpC[x1];
	  const double x2_val = interpC[x2];
	  const double y1_val = interpR[y1];
	  const double y2_val = interpR[y2];
	  
	  const double f_x_y1 = q11*(x2_val-x)/(x2_val-x1) + q21*(x-x1_val)/(x2_val-x1_val);
	  const double f_x_y2 = q12*(x2_val-x)/(x2_val-x1_val) + q22*(x-x1_val)/(x2_val-x1_val);
	  
	  double f_x_y = f_x_y1*(y2_val-y)/(y2_val-y1_val) + f_x_y2*(y-y1_val)/(y2_val-y1_val); 
	  return f_x_y;
    }
	
	__device__ __host__ double
    min_x() const{ 
	return interpC[0]; }
	
	__device__ __host__ double
    max_x() const{ 
	return interpC[_cols-1];  }
	
    __device__  __host__  double
    min_y() const{ 
	return interpR[0]; }
	
    __device__ __host__ double
    max_y() const{ 
	return interpC[_rows-1]; }
	
	__device__  __host__ double
	do_clamp(double v, double lo, double hi) const
    {
		assert(!(hi < lo));
		return (v < lo) ? lo : (hi < v) ? hi : v;
    }
	
	__device__ __host__ double
    eval(double x, double y) const
    {
      return this->operator()(x, y);
    };
	
	__device__  __host__
    double
    clamp(double x, double y) const
    {
      return eval(do_clamp(x, min_x(), max_x()), do_clamp(y, min_y(), max_y()));
    }
  };
}

template <class T>
class hmf_t {
	  public:
	  
		__device__ __host__ 
		hmf_t() = default;
		__device__ __host__
		hmf_t(typename T::Interp2D* nmz, double s, double q)
		  : _nmz(nmz), _s(s), _q(q)
		{}
		
		using doubles = std::vector<double>;
		
		__device__ __host__
		double
		operator()(double lnM, double zt) const{
		  //printf("Inside operator ");
		  //printf("interpolation result:%f\n", _nmz->clamp(lnM, zt));
		  return _nmz->clamp(lnM, zt) *
				 (_s * (lnM * 0.4342944819 - 13.8124426028) + _q);
		}
		
		friend std::ostream&
		operator<<(std::ostream& os, hmf_t const& m){
		  auto const old_flags = os.flags();
		  os << std::hexfloat;
		  os << *(m._nmz) << '\n' << m._s << ' ' << m._q;
		  os.flags(old_flags);
		  return os;
		}

		friend std::istream&
		operator>>(std::istream& is, hmf_t& m){
		  assert(is.good());
		  //auto table = std::make_shared<typename T::Interp2D>();
		  //needs to be deleted
		  typename T::Interp2D *table = new typename T::Interp2D;
		  is >> *table;
		  
		  std::string buffer;
		  std::getline(is, buffer);
		  std::vector<double> const vals_read = cosmosis::str_to_doubles(buffer);
		  if (vals_read.size() == 2)
		  {
			m = hmf_t(table, vals_read[0], vals_read[1]);
		  }
		  else
		  {
			is.setstate(std::ios_base::failbit);
		  };
		  return is;
		}
		
	  private:
		typename T::Interp2D* _nmz;
		double _s = 0.0;
		double _q = 0.0;
};

struct GPU {
  typedef quad::Interp2D Interp2D;
};

struct CPU {
  typedef y3_cluster::Interp2D Interp2D;
};

template<typename T>
__global__ 
void
testKernel(T* model, double x, double y){
	//printf("Entered kernel\n");
	printf("quad:cudac gpu model:%.8f\n", model->operator()(x, y));
}

template <class M>
M
make_from_file(char const* filename)
{
  static_assert(std::is_default_constructible<M>::value, "Type must be default constructable");
  char const* basedir = std::getenv("Y3_CLUSTER_CPP_DIR");
  if (basedir == nullptr) throw std::runtime_error("Y3_CLUSTER_CPP_DIR was not defined\n");
  std::string fname(basedir);
  fname += '/';
  fname += filename;
  std::ifstream in(fname);
  if (!in) {
    std::string msg("Failed to open file: ");
    msg += fname;
    throw std::runtime_error(msg);
  }
  M result;
  in >> result;
  return result;
}

struct DataElement //: public Managed
{
  char *name;
  int value;
};

__global__ 
void Kernel(DataElement *elem) {
  printf("On device: name=%s, value=%d\n", elem->name, elem->value);

  elem->name[0] = 'd';
  elem->value++;
}

void launch(DataElement *elem) {
  Kernel<<< 1, 1 >>>(elem);
  hipDeviceSynchronize();
}

class example{
	public:
		__host__ __device__
		example(){
			x = 1;
			y = 2;
			hipMallocManaged((void**)&data, sizeof(int)*10);
			for(int i=0; i<10; i++){
				data[i] = 10+i;
				printf("%i\n", data[i]);
			}
		}
		
		int call(int i){
			printf("inside call\n");
			return data[0];}
		int x;
		int y; 
		int *data;
	
};

  template<class T>
  class X{
	public:
		hmf_t<T> model1;
		hmf_t<T> model2;
	    
		__device__ __host__
		double operator()(double x, double y){
			printf("model 1:%.8f\n", model1(x, y));
			printf("model 2:%.8f\n", model2(x, y));
		}
  };

int main(){
  hmf_t<CPU> hmf  = make_from_file<hmf_t<CPU>>("data/HMF_t.dump");
  hmf_t<GPU> hmf2 = make_from_file<hmf_t<GPU>>("data/HMF_t.dump");
  hmf_t<GPU> *dhmf2;
  hipMallocManaged((void**)&dhmf2, sizeof(hmf_t<GPU>));
  hipDeviceSynchronize();
  memcpy(dhmf2, &hmf2, sizeof(hmf_t<GPU>));
  X<GPU> x;
  x.model1 = make_from_file<hmf_t<GPU>>("data/HMF_t.dump");
  x.model2 = make_from_file<hmf_t<GPU>>("data/HMF_t.dump");
  double const zt = 0x1.cccccccccccccp-2;
  double const lnM = 0x1.0cp+5;
  x(lnM, zt);
  
  printf("y3_cluster cpu model:%.8f\n", hmf(lnM, zt));
  printf("quad:cudac cpu model:%.8f\n", dhmf2->operator()(lnM, zt));
  
  testKernel<hmf_t<GPU>><<<1,1>>>(dhmf2, lnM, zt);
  hipDeviceSynchronize();
  
  return 0;
}
