#include "hip/hip_runtime.h"
#include "function.cuh"
#include "demo_utils.cuh"
#include <chrono>
#include <cmath>
#include <fstream>
#include <iomanip>
#include <iostream>

using namespace quad;

namespace detail{
    class GENZ_5_5D {
    public:
      __device__ __host__ double
      operator()(double x, double y, double z, double k, double m)
      {
        double beta = .5;
        double t1 = -10.*fabs(x - beta) - 10.* fabs(y - beta) - 10.* fabs(z - beta) - 10.* fabs(k - beta) - 10.* fabs(m - beta);
        return exp(t1);
      }
    };
}

int
main()
{
  double epsrel = 1.0e-3; // starting error tolerance.
  double const epsrel_min = 1.024e-10;
  double true_value = 0.0003093636;
  detail::GENZ_5_5D integrand;
  
  constexpr int ndim = 5;
  Config configuration;
  configuration.outfileVerbosity = 0;
  //configuration.heuristicID = 0;
  //configuration.phase_2 = false;
  
  PrintHeader();
  while (cu_time_and_call<detail::GENZ_5_5D, ndim>("GENZ5_5D",
                       integrand,
                       epsrel,
                       true_value,
                       "gpucuhre",
                       std::cout,
                       configuration) == true &&
         epsrel > epsrel_min) {
    epsrel /= 5.0;
  }
}
