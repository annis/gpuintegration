#include "hip/hip_runtime.h"
#include <chrono>
#include <cmath>
#include <iomanip>

#include "cudaPagani/quad/quad.h"
#include "cudaPagani/quad/util/cudaUtil.h"
#include "function.cuh"

#include "cudaPagani/quad/GPUquad/Interp2D.cuh"
#include "cudaPagani/quad/GPUquad/Pagani.cuh"
#include "cudaPagani/quad/util/Volume.cuh"

using namespace quad;
using std::chrono::duration;
using std::chrono::high_resolution_clock;

constexpr double EPSABS = 1.0e-40;

int
main(int argc, char** argv)
{
  TYPE epsrel = 2.560e-09;
  constexpr int ndim = 8;

  Pagani<TYPE, ndim> pagani;
  BoxIntegral8_22 integrand;
  int _final = 1;
  int outfileVerbosity = 0;
  int phase_I_type = 0; // alternative phase 1

  double highs[ndim] = {1., 1., 1., 1., 1., 1., 1., 1.};
  double lows[ndim] = {0., 0., 0., 0., 0., 0., 0., 0.};
  Volume<double, ndim> vol(lows, highs);
  double true_value = 1495369.283757217694;

  using MilliSeconds =
    std::chrono::duration<double, std::chrono::milliseconds::period>;
  auto t0 = std::chrono::high_resolution_clock::now();
  cuhreResult result = pagani.integrate<BoxIntegral8_22>(
    integrand, epsrel, EPSABS, &vol, outfileVerbosity, _final, phase_I_type);
  MilliSeconds dt = std::chrono::high_resolution_clock::now() - t0;

  printf("%.20f +- %.20f epsrel:%e, nregions:%lu flag:%i time:%f error:%.17f, "
         "ratio:%.17f failed phase2 blocks:%i\n",
         result.estimate,
         result.errorest,
         epsrel,
         result.nregions,
         result.status,
         dt.count(),
         abs(true_value - result.estimate),
         result.errorest / MaxErr(result.estimate, epsrel, EPSABS),
         result.phase2_failedblocks);
  return 0;
}
