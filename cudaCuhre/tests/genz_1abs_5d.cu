#include "hip/hip_runtime.h"
#include "genz_1abs_5d.cuh"

#include "quad/quad.h"
#include "quad/util/cudaUtil.h"

#include "quad/GPUquad/Cuhre.cuh"

#include <chrono>
#include <cmath>
#include <iomanip>
#include <iostream>


// Integrate the provided integrand using quad::Cuhre, to the given relative
// error tolerance, writing out timing information to std::cout.
// Return true if the algorithm converged, and false otherwise.

template <typename F>
bool
time_and_call(F integrand, double epsrel, double correct_answer, char const* algname)
{
  using MilliSeconds = std::chrono::duration<double, std::chrono::milliseconds::period>;
  double constexpr epsabs = 1.0e-40;

  double lows[] =  {0., 0., 0., 0., 0.};
  double highs[] = {1., 1., 1., 1., 1.};
  constexpr int ndim = 5;
  quad::Volume<double, ndim> vol(lows, highs);

  // Why does the integration algorithm need ndim as a template parameter?
  quad::Cuhre<double, ndim> alg(0, nullptr, 0, 0, 1);
 
  auto const t0 = std::chrono::high_resolution_clock::now();
  auto const res = alg.integrate(integrand, epsrel, epsabs, &vol);
  MilliSeconds dt = std::chrono::high_resolution_clock::now() - t0;
  double const absolute_error = std::abs(res.estimate - correct_answer);
  bool const good = (res.status == 0);
  std::cout << std::scientific
    << algname << '\t'
    << epsrel << '\t';
  if (good) {
    std::cout << res.estimate << '\t'
      << res.errorest << '\t'
      << absolute_error << '\t';
  } else {
    std::cout << "NA\tNA\tNA\t";
  }
  std::cout << res.neval << '\t'
    << res.nregions << '\t'
    << dt.count()
    << std::endl;
  return good;
}

int main()
{
  Genz_1abs_5d integrand;
  double epsrel = 1.0e-3;

  std::cout<< "alg\tepsrel\tvalue\terrorest\terror\tneval\tnregions\ttime\n";

  while (time_and_call(integrand, epsrel, 1.0, "gpucuhre")) {
      epsrel /= 5.0;
  }
}

