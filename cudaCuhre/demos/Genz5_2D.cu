#include "hip/hip_runtime.h"
#include "function.cuh"
#include "quad/GPUquad/Cuhre.cuh"
#include "quad/quad.h"
#include "quad/util/Volume.cuh"
#include "quad/util/cudaUtil.h"
#include <chrono>
#include <cmath>
#include <fstream>
#include <iomanip>
#include <iostream>

using namespace quad;

template <typename F>
bool
time_and_call(std::string id,
              F integrand,
              double epsrel,
              double true_value,
              char const* algname,
              std::ostream& outfile,
              int _final = 0)
{
  using MilliSeconds =
    std::chrono::duration<double, std::chrono::milliseconds::period>;
  double constexpr epsabs = 1.0e-40;

  double lows[] = {0., 0};
  double highs[] = {1., 1.};

  constexpr int ndim = 2;
  quad::Volume<double, ndim> vol(lows, highs);
  int const key = 0;
  int const verbose = 0;
  int const numdevices = 1;
  quad::Cuhre<double, ndim> alg(0, nullptr, key, verbose, numdevices);

  int outfileVerbosity = 0;
  constexpr int phase_I_type = 0; // alternative phase 1

  auto const t0 = std::chrono::high_resolution_clock::now();
  cuhreResult const result = alg.integrate<F>(
    integrand, epsrel, epsabs, &vol, outfileVerbosity, _final, phase_I_type);
  MilliSeconds dt = std::chrono::high_resolution_clock::now() - t0;
  double const absolute_error = std::abs(result.estimate - true_value);
  bool good = false;

  if (result.status == 0 || result.status == 2) {
    good = true;
  }

  outfile << std::fixed << id << ",\t" << std::fixed << true_value << ",\t"
          << std::scientific << epsrel << ",\t\t\t" << std::scientific
          << epsabs << ",\t" << std::fixed << result.estimate << ",\t"
          << std::fixed << result.errorest << ",\t" << std::fixed
          << result.nregions << ",\t" << std::fixed << result.status << ",\t"
          << _final << ",\t" << result.lastPhase << ",\t" << dt.count() << std::endl;
  return good;
}

int
main()
{
  double epsrel = 1.0e-3; // starting error tolerance.
  double const epsrel_min = 1.0e-12;
  double true_value = 0.039462780237263662026;
  GENZ_5_2D integrand;
  std::cout << "id, value, epsrel, epsabs, estimate, errorest, regions, "
             "converge, final, total_time\n";
  int _final = 1;
  while (time_and_call("pdc_f1_latest",
                       integrand,
                       epsrel,
                       true_value,
                       "gpucuhre",
                       std::cout,
                       _final) == true &&
         epsrel >= epsrel_min) {
    epsrel /= 5.0;
  }

  _final = 0;
  epsrel = 1.0e-3;

  while (time_and_call("pdc_f0_latest",
                       integrand,
                       epsrel,
                       true_value,
                       "gpucuhre",
                       std::cout,
                       _final) == true &&
         epsrel >= epsrel_min) {
    epsrel = epsrel >= 1e-6 ? epsrel / 5.0 : epsrel / 2.0;
  }
}
