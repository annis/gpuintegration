#include <chrono>
#include <iostream>
#include <vector>

#include <array>
#include <fstream>
#include <stdexcept>
#include <string>
#include <chrono>
#include "vegas/vegas_mcubes_1DT.cuh"

using std::cout;
using std::chrono::duration;
using std::chrono::high_resolution_clock;

int main(){
    
    using MilliSeconds =
    std::chrono::duration<double, std::chrono::milliseconds::period>;
    int functionID = 0;
    int NDIM = 6;
    double lbound[6] = {0., 0., 0., 0., 0., 0.};
    double rbound[6] = {10., 10., 10., 10., 10., 10.};
    double _ncall = 2.0E+09;
    int minIters = 10;
    int maxIters = 5; 
    int _skip = 0;
    int chunkSize = 2048;
    MilliSeconds dt;
    
    std::cout <<"ID,\t"
              <<"Integral,\t"
              <<"std,\t"
              <<"chisq,\t"
              <<"ncalls,\t"
              <<"chunkSize,\t"
              <<"iters,\t"
              <<"time\n";
    
    while(chunkSize >= 32){
        
        while(_ncall >= 1e3){
            cout<<"0,\t";
            auto t0 = std::chrono::high_resolution_clock::now();
            vegas_mcubes_1D(functionID, NDIM, lbound, rbound, _ncall, minIters, maxIters, _skip, chunkSize);
            dt = std::chrono::high_resolution_clock::now() - t0;
            hipDeviceReset();
            
            std::cout<< _ncall <<",\t"
                     << chunkSize <<",\t"
                     << maxIters <<",\t"
                     << dt.count()
                     <<"\n";
            _ncall /= 4;
        }
        
        _ncall = 2.0e09;
        chunkSize = chunkSize/2;
    }
    
    return 0;
}