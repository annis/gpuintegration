#include "hip/hip_runtime.h"
#include "vegas/demos/demo_utils.cuh"
#include "vegas/vegasT.cuh"

class GENZ_2_6D {
public:
  __device__ __host__ double
  operator()(double x, double y, double z, double k, double l, double m)
  {
    double a = 50.;
    double b = .5;

    double term_1 = 1. / ((1. / pow(a, 2)) + pow(x - b, 2));
    double term_2 = 1. / ((1. / pow(a, 2)) + pow(y - b, 2));
    double term_3 = 1. / ((1. / pow(a, 2)) + pow(z - b, 2));
    double term_4 = 1. / ((1. / pow(a, 2)) + pow(k - b, 2));
    double term_5 = 1. / ((1. / pow(a, 2)) + pow(l - b, 2));
    double term_6 = 1. / ((1. / pow(a, 2)) + pow(m - b, 2));

    double val = term_1 * term_2 * term_3 * term_4 * term_5 * term_6;
    return val;
  }
};

int
main(int argc, char** argv)
{
  double epsrel = 1e-3;
  double epsrel_min = 1e-9;
  constexpr int ndim = 6;
  double ncall = 1.e8;
  int titer = 100;
  int itmax = 40;
  int skip = 10;
  VegasParams params(ncall, titer, itmax, skip);

  double true_value = 1.286889807581113e+13;

  double lows[] = {0., 0., 0., 0., 0., 0.};
  double highs[] = {1., 1., 1., 1., 1., 1.};
  quad::Volume<double, ndim> volume(lows, highs);
  GENZ_2_6D integrand;

  PrintHeader();
  bool success = false;
  size_t expID = 0;
  do {
    params.ncall = ncall;
    for (int run = 0; run < 100; run++) {
      success = mcubes_time_and_call<GENZ_2_6D, ndim>(
        integrand, epsrel, true_value, "f2 6D", params, &volume);
      if (!success)
        break;
    }
    epsrel /= 5.;
  } while (epsrel >= epsrel_min && success == true);

  return 0;
}
