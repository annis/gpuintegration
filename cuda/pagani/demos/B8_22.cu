#include "hip/hip_runtime.h"
#include "cuda/pagani/demos/demo_utils.cuh"
#include "cuda/pagani/demos/function.cuh"
#include <chrono>
#include <cmath>
#include <fstream>
#include <iomanip>
#include <iostream>

using namespace quad;

namespace detail {
  class BoxIntegral8_22 {
  public:
    __device__ __host__ double
    operator()(double x,
               double y,
               double z,
               double k,
               double l,
               double m,
               double n,
               double o)
    {
      double s = 22;
      double sum = 0;
      sum = pow(x, 2) + pow(y, 2) + pow(z, 2) + pow(k, 2) + pow(l, 2) +
            pow(m, 2) + pow(n, 2) + pow(o, 2);
      return pow(sum, s / 2);
    }
  };
}

int
main()
{
  double epsrel = 1.e-3; // starting error tolerance.
  double const epsrel_min = 1.0240000000000002e-10;
  double true_value = 1495369.283757217694;
  constexpr int ndim = 8;
  detail::BoxIntegral8_22 integrand;

  Config configuration;
  configuration.outfileVerbosity = 0;
  // configuration.heuristicID = 0;
  // configuration.phase_2 = true;
  PrintHeader();
  while (cu_time_and_call<detail::BoxIntegral8_22, ndim>("8D f7",
                                                         integrand,
                                                         epsrel,
                                                         true_value,
                                                         "gpucuhre",
                                                         std::cout,
                                                         configuration) ==
           true &&
         epsrel > epsrel_min) {
    epsrel /= 5.0;
  }
}
