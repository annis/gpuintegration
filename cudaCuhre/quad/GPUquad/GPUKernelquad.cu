#include "hip/hip_runtime.h"
#ifndef CUDACUHRE_QUAD_GPUQUAD_GPUKERNELQUAD_CUH
#define CUDACUHRE_QUAD_GPUQUAD_GPUKERNELQUAD_CUH

#include "GPUQuadPhases.cu"
#include "GPUQuadRule.cu"

namespace quad {
  using namespace cooperative_groups;

  //===========
  // FOR DEBUGGINGG

  bool
  cudaMemoryTest()
  {
    const unsigned int N = 1048576;
    const unsigned int bytes = N * sizeof(int);
    int* h_a = (int*)malloc(bytes);
    int* d_a;
    hipMalloc((int**)&d_a, bytes);

    memset(h_a, 0, bytes);
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost);
    return true;
  }

  //==========

  __constant__ size_t dFEvalPerRegion;

  template <typename T>
  __global__ void
  PrintcuArray(T* array, int size)
  {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
      for (int i = 0; i < size; i++) {
        // if(i<10)
        printf("array[%i]:%.12f\n", i, array[i]);
        printf("array[%i]:%.12f\n", i, array[i]);
      }
    }
  }

  template <typename T>
  __global__ void
  PrintcuArray(T* array, T* array2, int size)
  {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
      for (int i = 0; i < size; i++)
        // if(i<10)
        printf("array[%i]:%.12f - %.12f\n", i, array[i], array[i] + array2[i]);
    }
  }

  template <typename T>
  __global__ void
  generateInitialRegions(T* dRegions,
                         T* dRegionsLength,
                         size_t numRegions,
                         T* newRegions,
                         T* newRegionsLength,
                         size_t newNumOfRegions,
                         int numOfDivisionsPerRegionPerDimension,
                         int NDIM)
  {

    extern __shared__ T slength[];
    size_t threadId = blockIdx.x * blockDim.x + threadIdx.x;


    if (threadIdx.x < NDIM) {
      slength[threadIdx.x] =
        dRegionsLength[threadIdx.x] / numOfDivisionsPerRegionPerDimension;
    }
    __syncthreads();

    if (threadId < newNumOfRegions) {
      size_t interval_index =
        threadId / pow((T)numOfDivisionsPerRegionPerDimension, (T)NDIM);
      size_t local_id =
        threadId % (size_t)pow((T)numOfDivisionsPerRegionPerDimension, (T)NDIM);
      for (int dim = 0; dim < NDIM; ++dim) {
        size_t id =
          (size_t)(local_id /
                   pow((T)numOfDivisionsPerRegionPerDimension, (T)dim)) %
          numOfDivisionsPerRegionPerDimension;
        newRegions[newNumOfRegions * dim + threadId] =
          dRegions[numRegions * dim + interval_index] + id * slength[dim];
        newRegionsLength[newNumOfRegions * dim + threadId] = slength[dim];
      }
    }
  }

  template <typename T>
  __global__ void
  alignRegions(T* dRegions,
                T* dRegionsLength,
                int* activeRegions,
                T* dRegionsIntegral,
                T* dRegionsError,
                T* dRegionsParentIntegral,
                T* dRegionsParentError,
                int* subDividingDimension,
                int* scannedArray,
                T* newActiveRegions,
                T* newActiveRegionsLength,
                int* newActiveRegionsBisectDim,
                size_t numRegions,
                size_t newNumRegions,
                int numOfDivisionOnDimension)
  {

    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < numRegions && activeRegions[tid] == 1) {
      size_t interval_index = scannedArray[tid];

      for (int i = 0; i < DIM; ++i) {
        newActiveRegions[i * newNumRegions + interval_index] =
          dRegions[i * numRegions + tid];
        newActiveRegionsLength[i * newNumRegions + interval_index] =
          dRegionsLength[i * numRegions + tid];
      }

      dRegionsParentIntegral[interval_index] =
        dRegionsIntegral[tid + numRegions];
      dRegionsParentError[interval_index] = dRegionsError[tid + numRegions];

      dRegionsParentIntegral[interval_index + newNumRegions] =
        dRegionsIntegral[tid + numRegions];
      dRegionsParentError[interval_index + newNumRegions] =
        dRegionsError[tid + numRegions];

      for (int i = 0; i < numOfDivisionOnDimension; ++i) {
        newActiveRegionsBisectDim[i * newNumRegions + interval_index] =
          subDividingDimension[tid];
      }
    }
  }

  template <typename T>
  __global__ void
  divideIntervalsGPU(T* genRegions,
                     T* genRegionsLength,
                     T* activeRegions,
                     T* activeRegionsLength,
                     int* activeRegionsBisectDim,
                     size_t numActiveRegions,
                     int numOfDivisionOnDimension)
  {

    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < numActiveRegions) {

      int bisectdim = activeRegionsBisectDim[tid];
      size_t data_size = numActiveRegions * numOfDivisionOnDimension;

      for (int i = 0; i < numOfDivisionOnDimension; ++i) {
        for (int dim = 0; dim < DIM; ++dim) {
          genRegions[i * numActiveRegions + dim * data_size + tid] =
            activeRegions[dim * numActiveRegions + tid];
          genRegionsLength[i * numActiveRegions + dim * data_size + tid] =
            activeRegionsLength[dim * numActiveRegions + tid];    
        }
      }

      for (int i = 0; i < numOfDivisionOnDimension; ++i) {

        T interval_length =
          activeRegionsLength[bisectdim * numActiveRegions + tid] /
          numOfDivisionOnDimension;
        genRegions[bisectdim * data_size + i * numActiveRegions + tid] =
          activeRegions[bisectdim * numActiveRegions + tid] +
          i * interval_length;
        genRegionsLength[i * numActiveRegions + bisectdim * data_size + tid] =
          interval_length;
      }
    }
  }

  template <typename T>
  class GPUKernelCuhre {
    T* dRegions;
    T* dRegionsLength;
    T* hRegions;
    T* hRegionsLength;
    int NDIM, KEY, VERBOSE;
    size_t numRegions, numFunctionEvaluations;
    size_t fEvalPerRegion;
    HostMemory<T> Host;
    DeviceMemory<T> Device;
    QuadRule<T> Rule;
    Structures<T> constMem;
    int NUM_DEVICES;
    // Debug Msg
    char msg[256];

    std::ostream& log;

  public:
    void
    ExpandcuArray(T*& array, int currentSize, int newSize)
    {
      T* temp = 0;
      QuadDebug(Device.AllocateMemory((void**)&temp, sizeof(T) * newSize));
      QuadDebug(hipMemcpy(
        temp, array, sizeof(T) * currentSize, hipMemcpyDeviceToDevice));
      QuadDebug(Device.ReleaseMemory(array));
      array = temp;
    }

    GPUKernelCuhre(std::ostream& logerr = std::cout) : log(logerr)
    {
      numRegions = 0;
      numFunctionEvaluations = 0;
      NDIM = 0;
      KEY = 0;
    }

    ~GPUKernelCuhre()
    {

      if (VERBOSE) {
        sprintf(msg, "GPUKerneCuhre Destructur");
        Println(log, msg);
      }

      QuadDebug(Device.ReleaseMemory(dRegions));
      QuadDebug(Device.ReleaseMemory(dRegionsLength));
      Host.ReleaseMemory(hRegions);
      Host.ReleaseMemory(hRegionsLength);
      QuadDebug(hipDeviceReset());
      // commented out by Ioannis, needs to be addressed
      // if(DIM > 8)
      // QuadDebug(Device.ReleaseMemory(gpuGenPos));
    }

    size_t
    getNumActiveRegions()
    {
      return numRegions;
    }

    void
    setRegionsData(T* data, size_t size)
    {
      hRegions = &data[0];
      hRegionsLength = &data[size * NDIM];
      numRegions = size;
    }

    T*
    getRegions(size_t size, int startIndex)
    {
      T* newhRegionsAndLength = 0;
      newhRegionsAndLength = (T*)Host.AllocateMemory(
        &newhRegionsAndLength, 2 * sizeof(T) * size * NDIM);
      T *newhRegions = &newhRegionsAndLength[0],
        *newhRegionsLength = &newhRegionsAndLength[size * NDIM];
      // NOTE:Copy order is important
      for (int dim = 0; dim < NDIM; ++dim) {
        QuadDebug(hipMemcpy(newhRegions + dim * size,
                             dRegions + dim * numRegions + startIndex,
                             sizeof(T) * size,
                             hipMemcpyDeviceToHost));
        QuadDebug(hipMemcpy(newhRegionsLength + dim * size,
                             dRegionsLength + dim * numRegions + startIndex,
                             sizeof(T) * size,
                             hipMemcpyDeviceToHost));
      }
      return newhRegionsAndLength;
    }

    void
    InitGPUKernelCuhre(int dim, int key, int verbose, int numDevices = 1)
    {
      QuadDebug(hipDeviceReset());
      NDIM = dim;
      KEY = key;
      VERBOSE = verbose;
      NUM_DEVICES = numDevices;
      fEvalPerRegion = (1 + 2 * NDIM + 2 * NDIM + 2 * NDIM + 2 * NDIM +
                        2 * NDIM * (NDIM - 1) + 4 * NDIM * (NDIM - 1) +
                        4 * NDIM * (NDIM - 1) * (NDIM - 2) / 3 + (1 << NDIM));
      QuadDebug(hipMemcpyToSymbol(HIP_SYMBOL(dFEvalPerRegion),
                                   &fEvalPerRegion,
                                   sizeof(size_t),
                                   0,
                                   hipMemcpyHostToDevice));
      Rule.Init(NDIM, fEvalPerRegion, KEY, VERBOSE, &constMem);
      QuadDebug(Device.SetHeapSize());
    }

    //@brief Template function to display GPU device array variables
    template <class K>
    void
    display(K* array, size_t size)
    {
      K* tmp = (K*)malloc(sizeof(K) * size);
      hipMemcpy(tmp, array, sizeof(K) * size, hipMemcpyDeviceToHost);
      for (int i = 0; i < size; ++i) {
        printf("%.20lf \n", (T)tmp[i]);
      }
    }

    void
    GenerateInitialRegions()
    {
      hRegions = (T*)Host.AllocateMemory(&hRegions, sizeof(T) * NDIM);
      hRegionsLength =
        (T*)Host.AllocateMemory(&hRegionsLength, sizeof(T) * NDIM);

      for (int dim = 0; dim < NDIM; ++dim) {
        hRegions[dim] = 0;
#if GENZ_TEST == 1
        hRegionsLength[dim] = b[dim];
#else
        hRegionsLength[dim] = 1;
#endif
      }

      QuadDebug(Device.AllocateMemory((void**)&dRegions, sizeof(T) * NDIM));
      QuadDebug(
        Device.AllocateMemory((void**)&dRegionsLength, sizeof(T) * NDIM));

      QuadDebug(hipMemcpy(
        dRegions, hRegions, sizeof(T) * NDIM, hipMemcpyHostToDevice));
      QuadDebug(hipMemcpy(dRegionsLength,
                           hRegionsLength,
                           sizeof(T) * NDIM,
                           hipMemcpyHostToDevice));

      size_t numThreads = 512;
	  //this has been changed temporarily, do not remove
      /*size_t numOfDivisionPerRegionPerDimension = 4;
      if(NDIM == 5 )numOfDivisionPerRegionPerDimension = 2;
      if(NDIM == 6 )numOfDivisionPerRegionPerDimension = 2;
      if(NDIM == 7 )numOfDivisionPerRegionPerDimension = 2;
      if(NDIM > 7 )numOfDivisionPerRegionPerDimension = 2;
      if(NDIM > 10 )numOfDivisionPerRegionPerDimension = 1;*/

      size_t numOfDivisionPerRegionPerDimension = 1;

      size_t numBlocks = (size_t)ceil(
        pow((T)numOfDivisionPerRegionPerDimension, (T)NDIM) / numThreads);
      numRegions = (size_t)pow((T)numOfDivisionPerRegionPerDimension, (T)NDIM);

      T* newRegions = 0;
      T* newRegionsLength = 0;
      QuadDebug(Device.AllocateMemory((void**)&newRegions,
                                      sizeof(T) * numRegions * NDIM));
      QuadDebug(Device.AllocateMemory((void**)&newRegionsLength,
                                      sizeof(T) * numRegions * NDIM));

      generateInitialRegions<T><<<numBlocks, numThreads, NDIM * sizeof(T)>>>(
        dRegions,
        dRegionsLength,
        1,
        newRegions,
        newRegionsLength,
        numRegions,
        numOfDivisionPerRegionPerDimension,
        NDIM);

      QuadDebug(Device.ReleaseMemory((void*)dRegions));
      QuadDebug(Device.ReleaseMemory((void*)dRegionsLength));

      dRegions = newRegions;
      dRegionsLength = newRegionsLength;
      QuadDebug(hipMemcpy(dRegions,
                           newRegions,
                           sizeof(T) * numRegions * NDIM,
                           hipMemcpyDeviceToDevice));
      QuadDebug(hipMemcpy(dRegionsLength,
                           newRegionsLength,
                           sizeof(T) * numRegions * NDIM,
                           hipMemcpyDeviceToDevice));
    }

    void
    GenerateActiveIntervals(int* activeRegions,
                             int* subDividingDimension,
                             T* dRegionsIntegral,
                             T* dRegionsError,
                             T*& dParentsIntegral,
                             T*& dParentsError)
    {

      int* scannedArray = 0;
      QuadDebug(
        Device.AllocateMemory((void**)&scannedArray, sizeof(int) * numRegions));

      thrust::device_ptr<int> d_ptr =
        thrust::device_pointer_cast(activeRegions);
      thrust::device_ptr<int> scan_ptr =
        thrust::device_pointer_cast(scannedArray);
      thrust::exclusive_scan(d_ptr, d_ptr + numRegions, scan_ptr);

      int last_element;
      size_t numActiveRegions = 0;

      QuadDebug(hipMemcpy(&last_element,
                           activeRegions + numRegions - 1,
                           sizeof(int),
                           hipMemcpyDeviceToHost));
      QuadDebug(hipMemcpy(&numActiveRegions,
                           scannedArray + numRegions - 1,
                           sizeof(int),
                           hipMemcpyDeviceToHost));

      if (last_element == 1)
        numActiveRegions++;

      if (VERBOSE) {
        printf("numRegions:%lu \t numActiveRegions (Bad Regions):%lu\n",
               numRegions,
               numActiveRegions);
      }

      if (numActiveRegions > 0) {

        int numOfDivisionOnDimension = 2;

        if (VERBOSE) {
          sprintf(msg,
                  "\nComputing NumOfDivisionsOnDimension\n\t#. of Active "
                  "Regions\t\t: %lu\n\tDivision on dimension\t\t: %i division",
                  numActiveRegions,
                  numOfDivisionOnDimension);
          Println(log, msg);
        }

        int* newActiveRegionsBisectDim = 0;
        T *newActiveRegions = 0, *newActiveRegionsLength = 0;

        hipMalloc((void**)&newActiveRegions,
                   sizeof(T) * numActiveRegions * NDIM);
        hipMalloc((void**)&newActiveRegionsLength,
                   sizeof(T) * numActiveRegions * NDIM);

        ExpandcuArray(dParentsIntegral, numRegions * 2, numActiveRegions * 4);
        ExpandcuArray(dParentsError, numRegions * 2, numActiveRegions * 4);

        hipMalloc((void**)&newActiveRegionsBisectDim,
                   sizeof(int) * numActiveRegions * numOfDivisionOnDimension);

        size_t numThreads = BLOCK_SIZE;
        size_t numBlocks =
          numRegions / numThreads + ((numRegions % numThreads) ? 1 : 0);

        if (VERBOSE) {
          Println(log, "\nCalling GPU Function align_intervals");
          sprintf(msg,
                  "\n\t# of input intervals\t\t: %ld\n\t#. of Active "
                  "Intervals\t\t: %ld\n\t#. of Thread Blocks\t\t: %ld\n\t#. of "
                  "Threads per Blocks\t: %ld\n",
                  numRegions,
                  numActiveRegions,
                  numBlocks,
                  numThreads);

          Println(log, msg);
        }

        hipDeviceSynchronize();

        alignRegions<T><<<numBlocks, numThreads>>>(dRegions,
                                                    dRegionsLength,
                                                    activeRegions,
                                                    dRegionsIntegral,
                                                    dRegionsError,
                                                    dParentsIntegral,
                                                    dParentsError,
                                                    subDividingDimension,
                                                    scannedArray,
                                                    newActiveRegions,
                                                    newActiveRegionsLength,
                                                    newActiveRegionsBisectDim,
                                                    numRegions,
                                                    numActiveRegions,
                                                    numOfDivisionOnDimension);

        if (VERBOSE) {
          Println(log, "\nCalling GPU Function divideIntervalsGPU");
          sprintf(msg,
                  "\n\t# of input intervals\t\t: %lu\n\t#. of division on "
                  "dimension\t: %i\n\t#. of Thread Blocks\t\t: %ld\n\t#. of "
                  "Threads per Blocks\t: %ld",
                  numActiveRegions,
                  numOfDivisionOnDimension,
                  numBlocks,
                  numThreads);
          Println(log, msg);
        }

        T *genRegions = 0, *genRegionsLength = 0;
        numBlocks = numActiveRegions / numThreads +
                    ((numActiveRegions % numThreads) ? 1 : 0);

        QuadDebug(hipMalloc((void**)&genRegions,
                             sizeof(T) * numActiveRegions * NDIM *
                               numOfDivisionOnDimension));
        QuadDebug(hipMalloc((void**)&genRegionsLength,
                             sizeof(T) * numActiveRegions * NDIM *
                               numOfDivisionOnDimension));

        divideIntervalsGPU<T>
          <<<numBlocks, numThreads>>>(genRegions,
                                      genRegionsLength,
                                      newActiveRegions,
                                      newActiveRegionsLength,
                                      newActiveRegionsBisectDim,
                                      numActiveRegions,
                                      numOfDivisionOnDimension);

        QuadDebug(Device.ReleaseMemory(newActiveRegions));
        QuadDebug(Device.ReleaseMemory(newActiveRegionsLength));
        QuadDebug(Device.ReleaseMemory(newActiveRegionsBisectDim));

        numRegions = numActiveRegions * numOfDivisionOnDimension;

        QuadDebug(Device.ReleaseMemory((void*)dRegions));
        QuadDebug(Device.ReleaseMemory((void*)dRegionsLength));
        QuadDebug(Device.ReleaseMemory((void*)scannedArray));

        dRegions = genRegions;
        dRegionsLength = genRegionsLength;
        hipDeviceSynchronize();
       
        hipDeviceSynchronize();
        // TODO: throws error
        // QuadDebug(hipMemcpy(dRegions, 		genRegions, sizeof(T) *
        // numRegions * NDIM, hipMemcpyDeviceToDevice));
        // QuadDebug(hipMemcpy(dRegionsLength, 	genRegionsLength,
        // sizeof(T)
        // * numRegions * NDIM, hipMemcpyDeviceToDevice));
      } else {
        numRegions = 0;
      }
    }

    void
    FirstPhaseIteration(T epsrel,
                         T epsabs,
                         T& integral,
                         T& error,
                         size_t& nregions,
                         size_t& neval,
                         T*& dParentsIntegral,
                         T*& dParentsError)
    {

      if (VERBOSE) {
        printf("===================================\n");
      }

      size_t numThreads = BLOCK_SIZE;
      size_t numBlocks = numRegions;

      T *dRegionsError = 0, *dRegionsIntegral = 0;
      T* newErrs = 0;

      if (VERBOSE) {
        printf(
          "Beginning of FirstPhaseIteration:: Allocating for %lu bad regions\n",
          numRegions * 2);
      }

      QuadDebug(Device.AllocateMemory((void**)&dRegionsIntegral,
                                      sizeof(T) * numRegions * 2));
      QuadDebug(Device.AllocateMemory((void**)&dRegionsError,
                                      sizeof(T) * numRegions * 2));

      if (numRegions == 1 && error == 0) {
        QuadDebug(Device.AllocateMemory((void**)&dParentsIntegral,
                                        sizeof(T) * numRegions * 2));
        QuadDebug(Device.AllocateMemory((void**)&dParentsError,
                                        sizeof(T) * numRegions * 2));
      }

      int *activeRegions = 0, *subDividingDimension = 0;

      if (VERBOSE) {
        printf("FirstPhaseIteration:: Currently have %lu bad regions\n",
               numRegions);
      }

      QuadDebug(Device.AllocateMemory((void**)&activeRegions,
                                      sizeof(int) * numRegions));
      QuadDebug(Device.AllocateMemory((void**)&subDividingDimension,
                                      sizeof(int) * numRegions));

      if (VERBOSE) {
        Println(log, "\nEntering function IntegrateFirstPhase \n");
        sprintf(msg,
                "\t# of input intervals\t\t: %ld\n\t#. of Thread Blocks\t\t: "
                "%ld\n\t#. of Threads per Blocks\t: %ld\n",
                numRegions,
                numBlocks,
                numThreads);
        Println(log, msg);
      }

      INTEGRATE_GPU_PHASE12<T><<<numBlocks, numThreads>>>(dRegions,
                                                          dRegionsLength,
                                                          numRegions,
                                                          dRegionsIntegral,
                                                          dRegionsError,
                                                          dParentsIntegral,
                                                          dParentsError,
                                                          activeRegions,
                                                          subDividingDimension,
                                                          epsrel,
                                                          epsabs,
                                                          constMem,
                                                          Rule.GET_FEVAL(),
                                                          Rule.GET_NSETS());

      QuadDebug(
        Device.AllocateMemory((void**)&newErrs, sizeof(T) * numRegions * 2));
      hipDeviceSynchronize();

      if (numRegions != 1) {
        RefineError<T><<<numBlocks, numThreads>>>(dRegionsIntegral,
                                                  dRegionsError,
                                                  dParentsIntegral,
                                                  dParentsError,
                                                  newErrs,
                                                  activeRegions,
                                                  numRegions,
                                                  epsrel,
                                                  epsabs);
        hipDeviceSynchronize();
        QuadDebug(hipMemcpy(dRegionsError,
                             newErrs,
                             sizeof(T) * numRegions * 2,
                             hipMemcpyDeviceToDevice));
        hipDeviceSynchronize();
      }

      nregions += numRegions;
      neval += numRegions * fEvalPerRegion;

      if (VERBOSE) {
        printf("computing the integral/error for %lu regions\n", numRegions);
      }

      // integral && error are the accumalated ones
      // we temporarily add the leaves to see what's happening

      thrust::device_ptr<T> wrapped_ptr;

      wrapped_ptr = thrust::device_pointer_cast(dRegionsIntegral + numRegions);
      T rG = integral + thrust::reduce(wrapped_ptr, wrapped_ptr + numRegions);

      wrapped_ptr = thrust::device_pointer_cast(dRegionsError + numRegions);
      T errG = error + thrust::reduce(wrapped_ptr, wrapped_ptr + numRegions);

      wrapped_ptr = thrust::device_pointer_cast(dRegionsIntegral);
      integral =
        integral + thrust::reduce(wrapped_ptr, wrapped_ptr + numRegions);

      wrapped_ptr = thrust::device_pointer_cast(dRegionsError);
      error = error + thrust::reduce(wrapped_ptr, wrapped_ptr + numRegions);

      if (VERBOSE) {
        printf("rG:%f\t errG:%f\t | global results: integral:%f\t error:%f\n",
               rG,
               errG,
               integral,
               error);
      }

      if ((errG <= MaxErr(rG, epsrel, epsabs)) && GLOBAL_ERROR) {

        if (VERBOSE) {
          sprintf(msg,
                  "Global Error Check -\t%ld integrand evaluations so far\n%lf "
                  "+- %lf ",
                  neval,
                  rG,
                  errG);
          Println(log, msg);
        }

        integral = rG;
        error = errG;
        numRegions = 0;
        return;
      }

      GenerateActiveIntervals(activeRegions,
                               subDividingDimension,
                               dRegionsIntegral,
                               dRegionsError,
                               dParentsIntegral,
                               dParentsError);

      if (VERBOSE) {
        printf("rG:%f\t errG:%f\t | global results: integral:%f\t error:%f\n",
               rG,
               errG,
               integral,
               error);
      }

      QuadDebug(hipFree(subDividingDimension));

      QuadDebug(hipFree(newErrs));
      QuadDebug(hipFree(activeRegions));
      QuadDebug(hipFree(dRegionsError));
      QuadDebug(hipFree(dRegionsIntegral));
    }

    void
    IntegrateFirstPhase(T epsrel,
                        T epsabs,
                        T& integral,
                        T& error,
                        size_t& nregions,
                        size_t& neval)
    {

      T *dParentsError = 0, *dParentsIntegral = 0;

      for (int i = 0; i < 100; i++) {

        FirstPhaseIteration(epsrel,
                             epsabs,
                             integral,
                             error,
                             nregions,
                             neval,
                             dParentsIntegral,
                             dParentsError);
       
        if (numRegions < 1) {
          printf("NO BAD SUBREGIONS LEFT\n");
          return;
        }
        // printf("FIRST_PHASE_MAXREGIONS:%i\n", FIRST_PHASE_MAXREGIONS);
        if (numRegions >= FIRST_PHASE_MAXREGIONS) {
          printf("Reached the limit on Phase 1 regions supported (%i)\n",
                 FIRST_PHASE_MAXREGIONS);
          break;
        }
      }

      QuadDebug(hipFree(dParentsIntegral));
      QuadDebug(hipFree(dParentsError));

      hRegions =
        (T*)Host.AllocateMemory(&hRegions, sizeof(T) * numRegions * NDIM);
      hRegionsLength =
        (T*)Host.AllocateMemory(&hRegionsLength, sizeof(T) * numRegions * NDIM);
      QuadDebug(hipMemcpy(hRegions,
                           dRegions,
                           sizeof(T) * numRegions * NDIM,
                           hipMemcpyDeviceToHost));
      QuadDebug(hipMemcpy(hRegionsLength,
                           dRegionsLength,
                           sizeof(T) * numRegions * NDIM,
                           hipMemcpyDeviceToHost));
    }

    int
    IntegrateSecondPhase(T epsrel,
                         T epsabs,
                         T& integral,
                         T& error,
                         size_t& nregions,
                         size_t& neval,
                         T* optionalInfo = 0)
    {

      int numFailedRegions = 0;
      int num_gpus = 0; // number of CUDA GPUs

      if (optionalInfo != 0) {
        optionalInfo[0] = -INFTY;
      }

      /////////////////////////////////////////////////////////////////
      // determine the number of CUDA capable GPUs
      //
      hipGetDeviceCount(&num_gpus);
      if (num_gpus < 1) {
        fprintf(stderr, "no CUDA capable devices were detected\n");
        exit(1);
      }
      int num_cpu_procs = omp_get_num_procs();

      /*
    Why did you have this section?
      for(int i = 1; i < num_gpus; i++){
    int gpu_id;
    QuadDebug(hipSetDevice(i));	// "% num_gpus" allows more CPU threads
    than GPU devices QuadDebug(hipGetDevice(&gpu_id));
    QuadDebug(hipDeviceReset());
      }
      */

      if (VERBOSE) {
        /////////////////////////////////////////////////////////////////
        // display CPU and GPU configuration
        sprintf(msg, "number of host CPUs:\t%d\n", omp_get_num_procs());
        printf("number of host CPUs:\t%d\n", omp_get_num_procs());
        Println(log, msg);
        sprintf(msg, "number of CUDA devices:\t%d\n", num_gpus);
        printf("number of CUDA devices:\t%d\n", num_gpus);
        Println(log, msg);
        for (int i = 0; i < num_gpus; i++) {
          hipDeviceProp_t dprop;
          hipGetDeviceProperties(&dprop, i);
          sprintf(msg, "   %d: %s\n", i, dprop.name);
          Println(log, msg);
        }
        Println(log, "---------------------------\n");
      }

      // this works ok, check command line arg with actual devices available
      if (NUM_DEVICES > num_gpus)
        NUM_DEVICES = num_gpus;

      omp_set_num_threads(NUM_DEVICES);
      hipStream_t stream[NUM_DEVICES];
      hipEvent_t event[NUM_DEVICES];

#pragma omp parallel

      {
        unsigned int cpu_thread_id = omp_get_thread_num();
        unsigned int num_cpu_threads = omp_get_num_threads();

        // set and check the CUDA device for this CPU thread
        int gpu_id = -1;

        QuadDebug(hipSetDevice(
          cpu_thread_id %
          num_gpus)); // "% num_gpus" allows more CPU threads than GPU devices
        QuadDebug(hipGetDevice(&gpu_id));
        warmUpKernel<<<FIRST_PHASE_MAXREGIONS, BLOCK_SIZE>>>();

        if (VERBOSE) {
          sprintf(msg,
                  "CPU thread %d (of %d) uses CUDA device %d\n",
                  cpu_thread_id,
                  num_cpu_threads,
                  gpu_id);
          Println(log, msg);
        }

        if (cpu_thread_id < num_cpu_threads) {

          size_t numRegionsThread = numRegions / num_cpu_threads;
          int startIndex = cpu_thread_id * numRegionsThread;
          int endIndex = (cpu_thread_id + 1) * numRegionsThread;
          if (cpu_thread_id == (num_cpu_threads - 1))
            endIndex = numRegions;

          numRegionsThread = endIndex - startIndex;

          if (VERBOSE) {
            printf(
              "Num Regions in Phase 2:%lu (%u)\n", numRegions, cpu_thread_id);
            printf("Number of Regions going to each GPU:%lu (%u)\n",
                   numRegionsThread,
                   cpu_thread_id);
            printf("startIndex:%i (%i)\n", startIndex, cpu_thread_id);
            printf("endIndex:%i (%i)\n", endIndex, cpu_thread_id);
          }
          // QuadDebug(Device.SetHeapSize());
          CudaCheckError();

          Rule.loadDeviceConstantMemory(&constMem, cpu_thread_id);
          size_t numThreads = BLOCK_SIZE;
          size_t numBlocks = numRegionsThread;

          T *dRegionsError = 0, *dRegionsIntegral = 0;
          T *dRegionsThread = 0, *dRegionsLengthThread = 0;

          QuadDebug(Device.AllocateMemory((void**)&dRegionsIntegral,
                                          sizeof(T) * numRegionsThread));
          QuadDebug(Device.AllocateMemory((void**)&dRegionsError,
                                          sizeof(T) * numRegionsThread));

          int *activeRegions = 0, *subDividingDimension = 0,
              *dRegionsNumRegion = 0;

          QuadDebug(Device.AllocateMemory((void**)&activeRegions,
                                          sizeof(int) * numRegionsThread));
          QuadDebug(Device.AllocateMemory((void**)&subDividingDimension,
                                          sizeof(int) * numRegionsThread));
          QuadDebug(Device.AllocateMemory((void**)&dRegionsNumRegion,
                                          sizeof(int) * numRegionsThread));
          QuadDebug(Device.AllocateMemory((void**)&dRegionsThread,
                                          sizeof(T) * numRegionsThread * NDIM));
          QuadDebug(Device.AllocateMemory((void**)&dRegionsLengthThread,
                                          sizeof(T) * numRegionsThread * NDIM));

          CudaCheckError();
          // NOTE:Copy order is important

          for (int dim = 0; dim < NDIM; ++dim) {
            /*printf("copying from hRegions(%i-%i) -> dRegions(%i-%i) (%i)
               |numRegionsThread:%i numRegions:%i\n", dim * numRegions +
               startIndex , dim * numRegions + startIndex+numRegionsThread, dim
               * numRegionsThread, dim * numRegionsThread+numRegionsThread,
                                                                                                                                                                    cpu_thread_id, numRegionsThread, numRegions);*/
            QuadDebug(hipMemcpy(dRegionsThread + dim * numRegionsThread,
                                 hRegions + dim * numRegions + startIndex,
                                 sizeof(T) * numRegionsThread,
                                 hipMemcpyHostToDevice));

            QuadDebug(hipMemcpy(dRegionsLengthThread + dim * numRegionsThread,
                                 hRegionsLength + dim * numRegions + startIndex,
                                 sizeof(T) * numRegionsThread,
                                 hipMemcpyHostToDevice));
          }

          CudaCheckError();

          hipEvent_t start;
          QuadDebug(hipStreamCreate(&stream[gpu_id]));
          QuadDebug(hipEventCreate(&start));
          QuadDebug(hipEventCreate(&event[gpu_id]));
          QuadDebug(hipEventRecord(start, stream[gpu_id]));
          CudaCheckError();

          if (VERBOSE) {
            Println(log, "\n GPU Function PHASE2");
            sprintf(msg,
                    "\t# of input intervals\t\t: %ld\n\t#. of Thread "
                    "Blocks\t\t: %ld\n\t#. of Threads per Blocks\t: %ld\n",
                    numRegionsThread,
                    numBlocks,
                    numThreads);
            // printf(msg, "\t# of input intervals\t\t: %ld\n\t#. of Thread
            // Blocks\t\t: %ld\n\t#. of Threads per Blocks\t:
            // %ld\n",numRegionsThread, numBlocks, numThreads);
            Println(log, msg);
          }
          CudaCheckError();

          // std::cout << " phase2 : 	blocks:" << numBlocks << " threads:" <<
          // numThreads << std::endl; printf("Status before entering phase 2
          // %.12f +- %.12f\n", integral, error);
          hipDeviceSetLimit(hipLimitMallocHeapSize, 2 * 128 * 1024 * 1024);

          double* exitCondition = nullptr;
          // QuadDebug(Device.AllocateMemory((void **)&exitCondition,
          // sizeof(double)*2)); hipMemcpy(&exitCondition[0], &integral,
          // sizeof(T),	hipMemcpyHostToDevice); hipMemcpy(&exitCondition[1],
          // &error, 		sizeof(T),	hipMemcpyHostToDevice);

          BLOCK_INTEGRATE_GPU_PHASE2<T>
            <<<numBlocks, numThreads, 0, stream[gpu_id]>>>(dRegionsThread,
                                                           dRegionsLengthThread,
                                                           numRegionsThread,
                                                           dRegionsIntegral,
                                                           dRegionsError,
                                                           dRegionsNumRegion,
                                                           activeRegions,
                                                           subDividingDimension,
                                                           epsrel,
                                                           epsabs,
                                                           gpu_id,
                                                           constMem,
                                                           Rule.GET_FEVAL(),
                                                           Rule.GET_NSETS(),
                                                           exitCondition);

          hipDeviceSynchronize();
          // printf("BLOCK INTEGRATE_GPU done %d gpu:%i\n", cpu_thread_id,
          // gpu_id);
          CudaCheckError();
          // printf("After error checking and sync %d\n", cpu_thread_id);
          hipDeviceSynchronize();
          hipEventRecord(event[gpu_id], stream[gpu_id]);
          hipEventSynchronize(event[gpu_id]);

          float elapsed_time;
          hipEventElapsedTime(&elapsed_time, start, event[gpu_id]);

          if (optionalInfo != 0 && elapsed_time > optionalInfo[0]) {
            optionalInfo[0] = elapsed_time;
          }

          if (VERBOSE) {
            sprintf(msg,
                    "\nSecond Phase Kernel by thread %d (of %d) using CUDA "
                    "device %d took %.1f ms ",
                    cpu_thread_id,
                    num_cpu_threads,
                    gpu_id,
                    elapsed_time);
            Println(log, msg);
          }

          hipEventDestroy(start);
          hipEventDestroy(event[gpu_id]);

          thrust::device_ptr<T> wrapped_ptr;
          wrapped_ptr = thrust::device_pointer_cast(dRegionsIntegral);
          T integResult =
            thrust::reduce(wrapped_ptr, wrapped_ptr + numRegionsThread);
          // printf("integral %.12f + result %.12f\n", integral, integResult);
          integral += integResult;

          wrapped_ptr = thrust::device_pointer_cast(dRegionsError);
          error =
            error + thrust::reduce(wrapped_ptr, wrapped_ptr + numRegionsThread);

          thrust::device_ptr<int> int_ptr =
            thrust::device_pointer_cast(dRegionsNumRegion);
          int regionCnt = thrust::reduce(int_ptr, int_ptr + numRegionsThread);
          nregions += regionCnt;
          // std::cout << "Num regions : " << regionCnt << std::endl;

          neval += (regionCnt - numRegionsThread) * fEvalPerRegion * 2 +
                   numRegionsThread * fEvalPerRegion;

          int_ptr = thrust::device_pointer_cast(activeRegions);
          numFailedRegions +=
            thrust::reduce(int_ptr, int_ptr + numRegionsThread);

          //std::cout << "--" << numFailedRegions << std::endl;
          // QuadDebug(hipDeviceReset());

          QuadDebug(Device.ReleaseMemory(dRegionsError));
          QuadDebug(Device.ReleaseMemory(dRegionsIntegral));
          QuadDebug(Device.ReleaseMemory(dRegionsThread));
          QuadDebug(Device.ReleaseMemory(dRegionsLengthThread));
          QuadDebug(Device.ReleaseMemory(activeRegions));
          QuadDebug(Device.ReleaseMemory(subDividingDimension));
          QuadDebug(Device.ReleaseMemory(dRegionsNumRegion));
          QuadDebug(hipDeviceSynchronize());
        } else
          printf("Rogue cpu thread\n");
      }

      // sprintf(msg, "Execution time : %.2lf", optionalInfo[0]);
      // Print(msg);
      return numFailedRegions;
    }
  };

}
#endif
