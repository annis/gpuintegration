#include "hip/hip_runtime.h"
#include "vegas/vegasT.cuh"
#include "vegas/demos/demo_utils.cuh"


class GENZ_2_6D {
public:
  __device__ __host__ double
  operator()(double x, double y, double z, double k, double l, double m)
  {
    double a = 50.;
    double b = .5;

    double term_1 = 1. / ((1. / pow(a, 2)) + pow(x - b, 2));
    double term_2 = 1. / ((1. / pow(a, 2)) + pow(y - b, 2));
    double term_3 = 1. / ((1. / pow(a, 2)) + pow(z - b, 2));
    double term_4 = 1. / ((1. / pow(a, 2)) + pow(k - b, 2));
    double term_5 = 1. / ((1. / pow(a, 2)) + pow(l - b, 2));
    double term_6 = 1. / ((1. / pow(a, 2)) + pow(m - b, 2));

    double val = term_1 * term_2 * term_3 * term_4 * term_5 * term_6;
    return val;
  }
};

int
main(int argc, char** argv)
{
  double epsrel = 1e-3;
  double epsrel_min = 1e-9;
  constexpr int ndim = 6;
  double ncall = 1.e8;
  int titer = 100;
  int itmax = 40;
  int skip = 10;
  VegasParams params(ncall, titer, itmax, skip);
  
  double true_value = 1.286889807581113e+13;

  double lows[] = {0., 0., 0., 0., 0., 0.};
  double highs[] = {1., 1., 1., 1., 1., 1.};
  quad::Volume<double, ndim> volume(lows, highs);
  GENZ_2_6D integrand;
  
  PrintHeader();
  //std::array<double, 10> required_ncall = {1.e8, 1.e8, 1.e8, 1.e8, 1.e8, 2.e9, 2.e9, 2.e9, 2.e9, 2.e9};
  std::array<double, 10> required_ncall = {1.e6, 1.e7, 1.e8, 1.e9, 2.e9, 3.e9, 4.e9, 5.e9, 2.e9, 2.e9};
  std::array<double, 10> adjust_iters = {40,40, 40, 40, 40, 40, 40, 40, 40, 40};
  bool success = false;
  size_t expID = 0;
  do{
        //params.ncall = required_ncall[expID];
        //params.num_adjust_iters = adjust_iters[expID];
        for(int run = 0; run < 100; run++){
            success = mcubes_time_and_call<GENZ_2_6D, ndim>
            (integrand, epsrel, true_value, "f2 6D", params, &volume);
            if(!success)
                break;
            }
        epsrel /= 5.;
       // expID++;
       // params.num_adjust_iters += 5;
  }while(epsrel >= epsrel_min && success == true); 
  
  return 0;
}



