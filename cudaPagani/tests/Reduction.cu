#include "hip/hip_runtime.h"
#define CATCH_CONFIG_MAIN
#include "catch2/catch.hpp"
#include "demos/function.cuh"
#include "quad/GPUquad/Sample.cuh"
#include "quad/quad.h"
#include "quad/util/Volume.cuh"
#include "quad/util/cudaUtil.h"
#include <chrono>
#include <cmath>
#include <fstream>
#include <iomanip>
#include <iostream>

using namespace quad;

class PTest {
public:   
    __device__ __host__ 
    double operator()(double x, double y){
        double res = 15.37;
        return res;
    }
};

class NTest {
public:   
    __device__ __host__ 
    double operator()(double x, double y){
        double res = -15.37;
        return res;
    }
};

class ZTest {
public:   
    __device__ __host__ 
    double operator()(double x, double y){
        return 0.;
    }
};

TEST_CASE("Constant Positive Value Function")
{
   constexpr int ndim = 2;
    size_t numRegions = 16;
    PTest integrand;
    size_t maxIters = 1;
	int heuristicID = 0; 
    double epsrel = 1.0e-3;
    double epsabs = 1.0e-12;
    int key = 0;
    int verbose = 0;
    int numDevices = 1;
    Cuhre<double, 2> cuhre(0, nullptr, key, verbose, numDevices);
    cuhreResult res = cuhre.integrate<PTest>(integrand, epsrel, epsabs);
        
    double integral = res.estimate;
    double error = res.errorest;
    
    //returns are never precisely equal to 0. and 15.37
	printf("ttotalEstimate:%.15f\n", integral);
    CHECK(abs(integral - 15.37) <= .00000000000001);
   
}