#include "hip/hip_runtime.h"
#include <iostream>
#include "cuda/pagani/demos/new_time_and_call.cuh"

class F_5_8D {
public:
  __device__ __host__ double
  operator()(double x,
             double y,
             double z,
             double k,
             double m,
             double n,
             double p,
             double q)
  {
	double beta = .5;
    double t1 = -10. * fabs(x - beta) - 10. * fabs(y - beta) -
                10. * fabs(z - beta) - 10. * fabs(k - beta) -
                10. * fabs(m - beta) - 10. * fabs(n - beta) -
                10. * fabs(p - beta) - 10. * fabs(q - beta);
    return exp(t1);
  }
};

int
main(int argc, char** argv)
{
  int num_repeats = argc > 1 ? std::stoi(argv[1]) : 11;
  constexpr int ndim = 8;
  F_5_8D integrand;
  quad::Volume<double, ndim> vol;
  call_cubature_rules<F_5_8D, ndim>(integrand, vol, num_repeats);
  return 0;
}
