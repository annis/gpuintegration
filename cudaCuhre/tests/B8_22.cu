#include "hip/hip_runtime.h"
#include <chrono>
#include <cmath>
#include <iostream>
#include <fstream>
#include "function.cuh"
#include "quad/quad.h"
#include "quad/util/cudaUtil.h"
#include <iomanip>
#include "quad/GPUquad/Cuhre.cuh"
#include "quad/util/Volume.cuh"

using namespace quad;

template <typename F>
bool
time_and_call(std::string id, F integrand, double epsrel, double true_value, char const* algname, std::stringstream& outfile,	int _final= 0)
{
  using MilliSeconds = std::chrono::duration<double, std::chrono::milliseconds::period>;
  double constexpr epsabs = 1.0e-40;
	
  double lows[] =  {0., 0., 0., 0., 0., 0., 0., 0.};
  double highs[] = {1., 1., 1., 1., 1., 1., 1., 1.};
  
  constexpr int ndim = 8;
  quad::Volume<double, ndim> vol(lows, highs);
  quad::Cuhre<double, ndim> alg(0, nullptr, 0, 0, 1);
	
  //std::string id 			= "BoxIntegral8_22";
  int outfileVerbosity  	= 0;
  constexpr  int phase_I_type 			= 0; // alternative phase 1

  auto const t0 = std::chrono::high_resolution_clock::now();
  cuhreResult const result = alg.integrate<BoxIntegral8_22>(integrand, epsrel, epsabs, &vol, outfileVerbosity, _final, phase_I_type);
  MilliSeconds dt = std::chrono::high_resolution_clock::now() - t0;
  double const absolute_error = std::abs(result.estimate - true_value);
  bool good = false;
  
  if(result.status == 0 || result.status == 2){
	  good = true;
  }
  
  std::cout <<std::fixed<<id<<",\t"
		    <<std::fixed<<true_value<<",\t"
			<<std::scientific<<epsrel<<",\t\t\t"
			<<std::scientific<<epsabs<<",\t"
			<<std::fixed<<result.estimate<<",\t"
			<<std::fixed<<result.errorest<<",\t"
			<<std::fixed<<result.nregions<<",\t"
			<<std::fixed<<result.status<<",\t"
			<<_final<<",\t"
			<<dt.count()<<std::endl;
  //printf("%.15f +- %.15f epsrel:%e final:%i nregions:%lu flag:%i time:%f\n", result.value, result.error, epsrel, _final, result.nregions, result.status, dt.count());
  return good;
}

int main(){
	double epsrel  = 1.0e-3;  // starting error tolerance.	
	int _final 			= 0;
	double true_value 	= 1495369.283757217694;
	std::stringstream outfile;
	BoxIntegral8_22 integrand;
	outfile<<"id, value, epsrel, epsabs, estimate, errorest, regions, converge, final, total_time" << std::endl; 
	_final = 1;
	while (time_and_call("pdcuhre_f1", integrand, epsrel, true_value, "gpucuhre", outfile, _final) == true && epsrel>=1e-8) {
		epsrel /= 5.0;
	}
	
	_final = 0;
	epsrel = 1.0e-3;
	
	
	while (time_and_call("pdcuhre_f0",integrand, epsrel, true_value, "gpucuhre", outfile, _final) == true && epsrel >= 2.56e-09) {
      epsrel = epsrel>=1e-6 ? epsrel / 5.0 : epsrel / 2.0;
	}
}