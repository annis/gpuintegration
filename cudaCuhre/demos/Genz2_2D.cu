#include "function.cuh"
#include "demo_utils.cuh"
#include <chrono>
#include <cmath>
#include <fstream>
#include <iomanip>
#include <iostream>

using namespace quad;

int
main()
{
  double epsrel = 1.0e-3; // starting error tolerance.
  double const epsrel_min = 1.0e-13;
  double true_value = 23434.04;
  GENZ_2_2D integrand;
  constexpr int ndim = 2;
  Config configuration;
  configuration.outfileVerbosity = 0;
  configuration.heuristicID = 4;
  
  PrintHeader();
  while (cu_time_and_call<GENZ_2_2D, ndim>("GENZ_2_2D",
                       integrand,
                       epsrel,
                       true_value,
                       "gpucuhre",
                       std::cout,
                       configuration) == true &&
         epsrel >= epsrel_min) {
    epsrel /= 5.0;
  }
}
