#include "hip/hip_runtime.h"
#include "function.cuh"
#include "demo_utils.cuh"
#include <chrono>
#include <cmath>
#include <fstream>
#include <iomanip>
#include <iostream>

using namespace quad;


namespace detail{
    class GENZ_4_5D {
    public:
        __device__ __host__ double
        operator()(double x, double y, double z, double w, double v){
            //double alpha = 25.;
            double beta = .5;
            return exp(-1.0*(pow(25,2)*pow(x-beta, 2) + 
                             pow(25,2)*pow(y-beta, 2) +
                             pow(25,2)*pow(z-beta, 2) +
                             pow(25,2)*pow(w-beta, 2) +
                             pow(25,2)*pow(v-beta, 2))
                      );
        }
    };
}

int
main()
{
  double epsrel = 1e-3;
  double const epsrel_min = 1.0240000000000002e-10;
  double true_value = 1.79132603674879e-06;
  detail::GENZ_4_5D integrand;
  PrintHeader();
  constexpr int ndim = 5;
  Config configuration;
  configuration.outfileVerbosity = 0;
  //configuration.heuristicID = 0;
  //configuration.phase_2 = false;
  while (cu_time_and_call<detail::GENZ_4_5D, ndim>("GENZ4_5D",
                                                integrand,
                                                epsrel,
                                                true_value,
                                                "gpucuhre",
                                                std::cout,
                                                configuration) == true &&
                                                epsrel > epsrel_min) {
    epsrel /= 5.0;
    //break;
  }

}
