#include "hip/hip_runtime.h"
#include <chrono>
#include <cmath>

#include "function.cuh"
#include "quad/quad.h"
#include "quad/util/cudaUtil.h"
#include <iomanip>

#include "quad/GPUquad/Cuhre.cuh"
#include "quad/util/Volume.cuh"

using namespace quad;
using std::chrono::duration;
using std::chrono::high_resolution_clock;

constexpr double EPSABS = 1e-12;

__global__ void
testKernel(){
	FUNC2 test_integrand;
	printf("GPU RESULT %a\n",    test_integrand(0x1.f4b65783633c5p-1, 0x1.f4b65783633c5p-1, 0x1p-1, 0x1p-1, 0x1p-1, 0x1p-1, 0x1p-1, 0x1.69350f939876p-6));
	printf("GPU RESULT %.17f\n", test_integrand(0x1.f4b65783633c5p-1, 0x1.f4b65783633c5p-1, 0x1p-1, 0x1p-1, 0x1p-1, 0x1p-1, 0x1p-1, 0x1.69350f939876p-6));
}

int
main(int argc, char** argv)
{
  // Initialize command line
  CommandLineArgs args(argc, argv);
  bool g_verbose = args.CheckCmdLineFlag("v");

  // Print usage
  if (args.CheckCmdLineFlag("help")) {
    printf("%s "
           "[--e=<relative-error>] "
           "[--verbose=<0/1>] "
           "\n",
           argv[0]);
    exit(0);
  }

  TYPE epsrel = 1.0e-4;
  if (args.CheckCmdLineFlag("e")) {
    args.GetCmdLineArgument("e", epsrel);
  }
  // Verbose output
  int verbose = 0;
  if (args.CheckCmdLineFlag("verbose")) {
    args.GetCmdLineArgument("verbose", verbose);
  }

  // Num Devices
  int numDevices = 1;
  if (args.CheckCmdLineFlag("N")) {
    args.GetCmdLineArgument("N", numDevices);
  }

  // Initialize device
  QuadDebugExit(args.DeviceInit());

  constexpr int ndim = 5;

  Cuhre<TYPE, ndim> cuhre(argc, argv, 0, verbose, numDevices);
  absCosSum5DWithoutKPlus1 integrand;
  int _final = 0;
  int outfileVerbosity = 0;
  int phase_I_type = 0; // alternative phase 1
	
  double highs[ndim] = {1., 1., 1., 1., 1.};
  double lows[ndim]  = {0., 0., 0., 0., 0.};
  Volume<double, ndim> vol(lows, highs);
  double true_value = 0.999926247661939;
  
  using MilliSeconds =
    std::chrono::duration<double, std::chrono::milliseconds::period>;
  auto t0 = std::chrono::high_resolution_clock::now();
  cuhreResult result = cuhre.integrate<absCosSum5DWithoutKPlus1>(
    integrand, epsrel, EPSABS, &vol, outfileVerbosity, _final, phase_I_type);
  MilliSeconds dt = std::chrono::high_resolution_clock::now() - t0;
	
  std::cout.precision(17);
  std::cout<<true_value<<",\t"
			<<epsrel<<",\t"
			<<EPSABS<<",\t"
			<<result.value<<",\t"
			<<result.error<<",\t"
			<<result.nregions<<",\t"
			<<result.status<<",\t"
			<<_final<<",\t"
			<<dt.count()<<std::endl;	
  printf("%.15f +- %.15f epsrel:%f, nregions:%lu flag:%i time:%f\n",
         result.value,
         result.error,
         epsrel,
         result.nregions,
         result.status,
         dt.count());
  return 0;
}
