#include "hip/hip_runtime.h"
#include "demo_utils.cuh"
#include <cmath>
#include <fstream>
#include <iomanip>
#include <iostream>

using namespace quad;


namespace detail{
    class GENZ_3_6D {
        public:
          __device__ __host__ double
          operator()(double x, double y, double z, double w, double v, double u)
          {
            return pow(1+6*u+5*v+4*w+3*x+2*y+z, -7);
          }
    };   
  
}

int
main()
{
  double epsrel = 1.0e-3; // starting error tolerance.
  double const epsrel_min = 1.024e-10;
  double true_value = 7.1790160638199853886e-7;
  detail::GENZ_3_6D integrand;         
  constexpr int ndim = 6;
  PrintHeader();
  
  while (cu_time_and_call<detail::GENZ_3_6D, ndim>("GENZ_3_6D",
                       integrand,
                       epsrel,
                       true_value,
                       std::cout) == true &&
         epsrel > epsrel_min) {
    epsrel /= 5.0;
  }
}
