#include "hip/hip_runtime.h"
#define CATCH_CONFIG_MAIN
#include "catch2/catch.hpp"
#include "cuda/pagani/demos/function.cuh"
#include "cuda/pagani/quad/GPUquad/Pagani.cuh"
#include "cuda/pagani/quad/quad.h"
#include "cuda/pagani/quad/util/Volume.cuh"
#include "cuda/pagani/quad/util/cudaUtil.h"
#include "cuda/pagani/quad/util/mem_util.cuh"
#include "cuda/pagani/quad/util/cudaMemoryUtil.h"
#include "cuda/pagani/quad/GPUquad/Sub_region_filter.cuh"

#include <chrono>
#include <cmath>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <numeric>
#include <array>

#ifdef ZERO_BANK_CONFLICTS
#define CONFLICT_FREE_OFFSET(n) \
((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))
#else
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_NUM_BANKS)
#endif

#define MAX_BLOCK_SZ 1024
#define NUM_BANKS 32
#define LOG_NUM_BANKS 5

template<typename T>
void
thrust_exclusive_scan(T* arr, size_t size, T* out){
	thrust::device_ptr<T> d_ptr = thrust::device_pointer_cast(arr);
    thrust::device_ptr<T> scan_ptr = thrust::device_pointer_cast(out);
    thrust::exclusive_scan(d_ptr, d_ptr + size, scan_ptr);
}

__global__
void gpu_sum_scan_blelloch(int* const d_out,
	const int* const d_in,
	int* const d_block_sums,
	const size_t numElems)
{
	extern __shared__ int s_out[];

	unsigned int glbl_tid = blockDim.x * blockIdx.x + threadIdx.x;

	// Zero out shared memory
	// Especially important when padding shmem for
	//  non-power of 2 sized input
	//s_out[2 * threadIdx.x] = 0;
	//s_out[2 * threadIdx.x + 1] = 0;
	s_out[threadIdx.x] = 0;
	s_out[threadIdx.x + blockDim.x] = 0;

	__syncthreads();

	// Copy d_in to shared memory per block
	//if (2 * glbl_tid < numElems)
	//{
	//	s_out[2 * threadIdx.x] = d_in[2 * glbl_tid];
	//	if (2 * glbl_tid + 1 < numElems)
	//		s_out[2 * threadIdx.x + 1] = d_in[2 * glbl_tid + 1];
	//}
	unsigned int cpy_idx = 2 * blockIdx.x * blockDim.x + threadIdx.x;
	if (cpy_idx < numElems)
	{
		s_out[threadIdx.x] = d_in[cpy_idx];
		if (cpy_idx + blockDim.x < numElems)
			s_out[threadIdx.x + blockDim.x] = d_in[cpy_idx + blockDim.x];
	}

	__syncthreads();

	// Reduce/Upsweep step

	// 2^11 = 2048, the max amount of data a block can blelloch scan
	unsigned int max_steps = 11; 

	unsigned int r_idx = 0;
	unsigned int l_idx = 0;
	int sum = 0; // global sum can be passed to host if needed
	unsigned int t_active = 0;
	for (int s = 0; s < max_steps; ++s)
	{
		t_active = 0;

		// calculate necessary indexes
		// right index must be (t+1) * 2^(s+1)) - 1
		r_idx = ((threadIdx.x + 1) * (1 << (s + 1))) - 1;
		if (r_idx >= 0 && r_idx < 2048)
			t_active = 1;

		if (t_active)
		{
			// left index must be r_idx - 2^s
			l_idx = r_idx - (1 << s);

			// do the actual add operation
			sum = s_out[l_idx] + s_out[r_idx];
		}
		__syncthreads();

		if (t_active)
			s_out[r_idx] = sum;
		__syncthreads();
	}

	// Copy last element (total sum of block) to block sums array
	// Then, reset last element to operation's identity (sum, 0)
	if (threadIdx.x == 0)
	{
		d_block_sums[blockIdx.x] = s_out[r_idx];
		s_out[r_idx] = 0;
	}

	__syncthreads();

	// Downsweep step

	for (int s = max_steps - 1; s >= 0; --s)
	{
		// calculate necessary indexes
		// right index must be (t+1) * 2^(s+1)) - 1
		r_idx = ((threadIdx.x + 1) * (1 << (s + 1))) - 1;
		if (r_idx >= 0 && r_idx < 2048)
		{
			t_active = 1;
		}

		unsigned int r_cpy = 0;
		unsigned int lr_sum = 0;
		if (t_active)
		{
			// left index must be r_idx - 2^s
			l_idx = r_idx - (1 << s);

			// do the downsweep operation
			r_cpy = s_out[r_idx];
			lr_sum = s_out[l_idx] + s_out[r_idx];
		}
		__syncthreads();

		if (t_active)
		{
			s_out[l_idx] = r_cpy;
			s_out[r_idx] = lr_sum;
		}
		__syncthreads();
	}

	// Copy the results to global memory
	//if (2 * glbl_tid < numElems)
	//{
	//	d_out[2 * glbl_tid] = s_out[2 * threadIdx.x];
	//	if (2 * glbl_tid + 1 < numElems)
	//		d_out[2 * glbl_tid + 1] = s_out[2 * threadIdx.x + 1];
	//}
	if (cpy_idx < numElems)
	{
		d_out[cpy_idx] = s_out[threadIdx.x];
		if (cpy_idx + blockDim.x < numElems)
			d_out[cpy_idx + blockDim.x] = s_out[threadIdx.x + blockDim.x];
	}
}

__global__
void gpu_add_block_sums(int* const d_out,
	const int* const d_in,
	int* const d_block_sums,
	const size_t numElems)
{
	//unsigned int glbl_t_idx = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int d_block_sum_val = d_block_sums[blockIdx.x];

	//unsigned int d_in_val_0 = 0;
	//unsigned int d_in_val_1 = 0;

	// Simple implementation's performance is not significantly (if at all)
	//  better than previous verbose implementation
	unsigned int cpy_idx = 2 * blockIdx.x * blockDim.x + threadIdx.x;
	if (cpy_idx < numElems)
	{
		d_out[cpy_idx] = d_in[cpy_idx] + d_block_sum_val;
		if (cpy_idx + blockDim.x < numElems)
			d_out[cpy_idx + blockDim.x] = d_in[cpy_idx + blockDim.x] + d_block_sum_val;
	}

	//if (2 * glbl_t_idx < numElems)
	//{
	//	d_out[2 * glbl_t_idx] = d_in[2 * glbl_t_idx] + d_block_sum_val;
	//	if (2 * glbl_t_idx + 1 < numElems)
	//		d_out[2 * glbl_t_idx + 1] = d_in[2 * glbl_t_idx + 1] + d_block_sum_val;
	//}

	//if (2 * glbl_t_idx < numElems)
	//{
	//	d_in_val_0 = d_in[2 * glbl_t_idx];
	//	if (2 * glbl_t_idx + 1 < numElems)
	//		d_in_val_1 = d_in[2 * glbl_t_idx + 1];
	//}
	//else
	//	return;
	//__syncthreads();

	//d_out[2 * glbl_t_idx] = d_in_val_0 + d_block_sum_val;
	//if (2 * glbl_t_idx + 1 < numElems)
	//	d_out[2 * glbl_t_idx + 1] = d_in_val_1 + d_block_sum_val;
}

// Modified version of Mark Harris' implementation of the Blelloch scan
//  according to https://www.mimuw.edu.pl/~ps209291/kgkp/slides/scan.pdf
__global__
void gpu_prescan(int* const d_out,
	const int* const d_in,
	int* const d_block_sums,
	const unsigned int len,
	const unsigned int shmem_sz,
	const unsigned int max_elems_per_block)
{
	// Allocated on invocation
	extern __shared__ int s_out[];

	int thid = threadIdx.x;
	int ai = thid;
	int bi = thid + blockDim.x;

	// Zero out the shared memory
	// Helpful especially when input size is not power of two
	s_out[thid] = 0;
	s_out[thid + blockDim.x] = 0;
	// If CONFLICT_FREE_OFFSET is used, shared memory
	//  must be a few more than 2 * blockDim.x
	if (thid + max_elems_per_block < shmem_sz)
		s_out[thid + max_elems_per_block] = 0;

	__syncthreads();
	
	// Copy d_in to shared memory
	// Note that d_in's elements are scattered into shared memory
	//  in light of avoiding bank conflicts
	unsigned int cpy_idx = max_elems_per_block * blockIdx.x + threadIdx.x;
	if (cpy_idx < len)
	{
		s_out[ai + CONFLICT_FREE_OFFSET(ai)] = d_in[cpy_idx];
		if (cpy_idx + blockDim.x < len)
			s_out[bi + CONFLICT_FREE_OFFSET(bi)] = d_in[cpy_idx + blockDim.x];
	}

	// For both upsweep and downsweep:
	// Sequential indices with conflict free padding
	//  Amount of padding = target index / num banks
	//  This "shifts" the target indices by one every multiple
	//   of the num banks
	// offset controls the stride and starting index of 
	//  target elems at every iteration
	// d just controls which threads are active
	// Sweeps are pivoted on the last element of shared memory

	// Upsweep/Reduce step
	int offset = 1;
	for (int d = max_elems_per_block >> 1; d > 0; d >>= 1)
	{
		__syncthreads();

		if (thid < d)
		{
			int ai = offset * ((thid << 1) + 1) - 1;
			int bi = offset * ((thid << 1) + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);

			s_out[bi] += s_out[ai];
		}
		offset <<= 1;
	}

	// Save the total sum on the global block sums array
	// Then clear the last element on the shared memory
	if (thid == 0) 
	{ 
		d_block_sums[blockIdx.x] = s_out[max_elems_per_block - 1 
			+ CONFLICT_FREE_OFFSET(max_elems_per_block - 1)];
		s_out[max_elems_per_block - 1 
			+ CONFLICT_FREE_OFFSET(max_elems_per_block - 1)] = 0;
	}

	// Downsweep step
	for (int d = 1; d < max_elems_per_block; d <<= 1)
	{
		offset >>= 1;
		__syncthreads();

		if (thid < d)
		{
			int ai = offset * ((thid << 1) + 1) - 1;
			int bi = offset * ((thid << 1) + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);

			int temp = s_out[ai];
			s_out[ai] = s_out[bi];
			s_out[bi] += temp;
		}
	}
	__syncthreads();

	// Copy contents of shared memory to global memory
	if (cpy_idx < len)
	{
		d_out[cpy_idx] = s_out[ai + CONFLICT_FREE_OFFSET(ai)];
		if (cpy_idx + blockDim.x < len)
			d_out[cpy_idx + blockDim.x] = s_out[bi + CONFLICT_FREE_OFFSET(bi)];
	}
}

void sum_scan_blelloch(int* const d_out,
	const int* const d_in,
	const size_t numElems)
{
	// Zero out d_out
	(hipMemset(d_out, 0, numElems * sizeof(unsigned int)));

	// Set up number of threads and blocks
	
	unsigned int block_sz = MAX_BLOCK_SZ / 2;
	unsigned int max_elems_per_block = 2 * block_sz; // due to binary tree nature of algorithm

	// If input size is not power of two, the remainder will still need a whole block
	// Thus, number of blocks must be the ceiling of input size / max elems that a block can handle
	//unsigned int grid_sz = (unsigned int) std::ceil((double) numElems / (double) max_elems_per_block);
	// UPDATE: Instead of using ceiling and risking miscalculation due to precision, just automatically  
	//  add 1 to the grid size when the input size cannot be divided cleanly by the block's capacity
	unsigned int grid_sz = numElems / max_elems_per_block;
	// Take advantage of the fact that integer division drops the decimals
	if (numElems % max_elems_per_block != 0) 
		grid_sz += 1;

	// Conflict free padding requires that shared memory be more than 2 * block_sz
	unsigned int shmem_sz = max_elems_per_block + ((max_elems_per_block - 1) >> LOG_NUM_BANKS);

	// Allocate memory for array of total sums produced by each block
	// Array length must be the same as number of blocks
	int* d_block_sums;
	(hipMalloc(&d_block_sums, sizeof(int) * grid_sz));
	(hipMemset(d_block_sums, 0, sizeof(int) * grid_sz));

	// Sum scan data allocated to each block
	//gpu_sum_scan_blelloch<<<grid_sz, block_sz, sizeof(unsigned int) * max_elems_per_block >>>(d_out, d_in, d_block_sums, numElems);
	gpu_prescan<<<grid_sz, block_sz, sizeof(int) * shmem_sz>>>(d_out, 
																	d_in, 
																	d_block_sums, 
																	numElems, 
																	shmem_sz,
																	max_elems_per_block);

	// Sum scan total sums produced by each block
	// Use basic implementation if number of total sums is <= 2 * block_sz
	//  (This requires only one block to do the scan)
	if (grid_sz <= max_elems_per_block)
	{
		int* d_dummy_blocks_sums;
		(hipMalloc(&d_dummy_blocks_sums, sizeof(int)));
		(hipMemset(d_dummy_blocks_sums, 0, sizeof(int)));
		//gpu_sum_scan_blelloch<<<1, block_sz, sizeof(unsigned int) * max_elems_per_block>>>(d_block_sums, d_block_sums, d_dummy_blocks_sums, grid_sz);
		gpu_prescan<<<1, block_sz, sizeof(int) * shmem_sz>>>(d_block_sums, 
																	d_block_sums, 
																	d_dummy_blocks_sums, 
																	grid_sz, 
																	shmem_sz,
																	max_elems_per_block);
		(hipFree(d_dummy_blocks_sums));
	}
	// Else, recurse on this same function as you'll need the full-blown scan
	//  for the block sums
	else
	{
		int* d_in_block_sums;
		(hipMalloc(&d_in_block_sums, sizeof(int) * grid_sz));
		(hipMemcpy(d_in_block_sums, d_block_sums, sizeof(int) * grid_sz, hipMemcpyDeviceToDevice));
		sum_scan_blelloch(d_block_sums, d_in_block_sums, grid_sz);
		(hipFree(d_in_block_sums));
	}
	
	//// Uncomment to examine block sums
	//unsigned int* h_block_sums = new unsigned int[grid_sz];
	//(hipMemcpy(h_block_sums, d_block_sums, sizeof(unsigned int) * grid_sz, hipMemcpyDeviceToHost));
	//std::cout << "Block sums: ";
	//for (int i = 0; i < grid_sz; ++i)
	//{
	//	std::cout << h_block_sums[i] << ", ";
	//}
	//std::cout << std::endl;
	//std::cout << "Block sums length: " << grid_sz << std::endl;
	//delete[] h_block_sums;

	// Add each block's total sum to its scan output
	// in order to get the final, global scanned array
	gpu_add_block_sums<<<grid_sz, block_sz>>>(d_out, d_out, d_block_sums, numElems);

	(hipFree(d_block_sums));
}

TEST_CASE("Exclusvie scan of array of size 8")
{
	constexpr size_t size = 8;
	std::array<int, size> arr = {3, 1, 7, 0, 4, 1, 6, 3};
	std::array<int, size> true_results = {0, 3, 4, 11, 11, 15, 16, 22};
	
	int *out = quad::cuda_malloc_managed<int>(size);
	int *d_arr = quad::cuda_malloc_managed<int>(size);
	cuda_memcpy_to_device<int>(d_arr, arr.data(), size);
	
	sum_scan_blelloch(out, d_arr, size);
	
	//for(int i = 0; i < size; ++i)
	//	std::cout<<out[i]<<std::endl;
	
	SECTION("Check results of custom function")
	{
		for(int i = 0; i < size; ++i)
			CHECK(true_results[i] == out[i]);
	}
	
	for(int i=0; i < size; ++i){
		out[i] = 0;	
	}
	
	thrust_exclusive_scan<int>(d_arr, size, out);

	
	SECTION("Thrust Gets the same results")
	{
		for(int i = 0; i < size; ++i)
			CHECK(true_results[i] == out[i]);
	}
	
	
	hipFree(d_arr);
	hipFree(out);
}

TEST_CASE("Exclusvie scan of array of non-power-two size")
{
	constexpr size_t size = 10000;
	std::array<int, size> arr;
	std::iota(arr.begin(), arr.end(), 1.);
	
	int *out_thrust = quad::cuda_malloc_managed<int>(size);
	int *out_custom = quad::cuda_malloc_managed<int>(size);
	int *d_arr = quad::cuda_malloc_managed<int>(size);
	cuda_memcpy_to_device<int>(d_arr, arr.data(), size);
	
	sum_scan_blelloch(out_custom, d_arr, size);
	thrust_exclusive_scan<int>(d_arr, size, out_thrust);
	
	SECTION("Check results of custom function")
	{
		for(int i = 0; i < size; ++i)
			CHECK(out_thrust[i] == out_custom[i]);
	}
	
	hipFree(d_arr);
	hipFree(out_thrust);
	hipFree(out_custom);
}


TEST_CASE("Exclusvie scan of array of odd size")
{
	constexpr size_t size = 10001;
	std::array<int, size> arr;
	std::iota(arr.begin(), arr.end(), 1.);
	
	int *out_thrust = quad::cuda_malloc_managed<int>(size);
	int *out_custom = quad::cuda_malloc_managed<int>(size);
	int *d_arr = quad::cuda_malloc_managed<int>(size);
	cuda_memcpy_to_device<int>(d_arr, arr.data(), size);
	
	sum_scan_blelloch(out_custom, d_arr, size);
	thrust_exclusive_scan<int>(d_arr, size, out_thrust);
	
	SECTION("Check results of custom function")
	{
		for(int i = 0; i < size; ++i)
			CHECK(out_thrust[i] == out_custom[i]);
	}
	
	hipFree(d_arr);
	hipFree(out_thrust);
	hipFree(out_custom);
}