#include "hip/hip_runtime.h"
#include <chrono>
#include <cmath>

#include "function.cuh"
#include "quad/quad.h"
#include "quad/util/cudaUtil.h"
#include <iomanip>

#include "quad/GPUquad/Cuhre.cuh"
#include "quad/util/Volume.cuh"

using namespace quad;
using std::chrono::high_resolution_clock;
using std::chrono::duration;

constexpr double EPSABS = 1e-12;

class GENZ_1_8d{
	
	public:
	double normalization;
	double integral;
	__device__ __host__
	GENZ_1_8d(){
		integral = (1./315.) * sin(1.) * sin(3./2.) * sin(2.) * sin (5./2.) * sin(3.) *
                        sin(7./2.) * sin(4.) * (sin(37./2.) - sin(35./2.));
		normalization = 1./integral;
						
	}
	__device__ __host__ double
	operator()(double s, double t, double u, double v,
                 double w, double x, double y, double z){
		return normalization * cos(s + 2.*t + 3.*u + 4.*v + 5.*w + 6.*x + 7.*y + 8.*z);			 
	}
};



int
main(int argc, char** argv)
{
  // Initialize command line
  CommandLineArgs args(argc, argv);
  bool g_verbose = args.CheckCmdLineFlag("v");

  // Print usage
  if (args.CheckCmdLineFlag("help")) {
    printf("%s "
           "[--e=<relative-error>] "
           "[--verbose=<0/1>] "
           "\n",
           argv[0]);
    exit(0);
  }
	
  TYPE epsrel = 4e-5;
  if (args.CheckCmdLineFlag("e")) {
    args.GetCmdLineArgument("e", epsrel);
  }
  // Verbose output
  int verbose = 0;
  if (args.CheckCmdLineFlag("verbose")) {
    args.GetCmdLineArgument("verbose", verbose);
  }
	
  // Num Devices
  int numDevices = 1;
  if (args.CheckCmdLineFlag("N")) {
    args.GetCmdLineArgument("N", numDevices);
  }

 // Initialize device
  QuadDebugExit(args.DeviceInit());
  
  constexpr int ndim = 5;
  
  Cuhre<TYPE, ndim> cuhre(argc, argv, 0, verbose, numDevices);
		
	//Test integrand;
	absCosSum5D integrand;
	
	//double highs[ndim] = {1, 1, 1, 1, 1, 1, 1, 1};
    //double lows[ndim] =  {0, 0, 0, 0, 0, 0, 0, 0};
	double highs[ndim] = {1, 1, 1, 1, 1};
    double lows[ndim] =  {0, 0, 0, 0, 0};
    Volume<double, ndim> vol(lows, highs);
	using MilliSeconds = std::chrono::duration<double, std::chrono::milliseconds::period>;
	
    /*cuhre.integrate<GENZ_1_8d>(&integrand, epsrel, EPSABS, integral, error, nregions, neval, &vol);*/
    auto t0 = std::chrono::high_resolution_clock::now();
	cuhreResult result = cuhre.integrate<absCosSum5D>(integrand, epsrel, EPSABS, &vol);
	MilliSeconds dt = std::chrono::high_resolution_clock::now() - t0;
	std::cout<< result.value <<"\t"<< result.error <<"\t"<<result.nregions<<std::endl;
	std::cout<<"Time in ms:"<< dt.count()<<std::endl;
  return 0;
}
