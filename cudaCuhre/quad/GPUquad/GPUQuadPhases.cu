#include "hip/hip_runtime.h"
#include "GPUQuadSample.cu"
#include <hip/hip_cooperative_groups.h>

namespace quad {

  template <typename T>
  __device__ void
  INIT_REGION_POOL(T* dRegions,
                   T* dRegionsLength,
                   size_t numRegions,
                   Structures<T>* constMem,
                   int FEVAL,
                   int NSETS)
  {

    size_t index = blockIdx.x;

    if (threadIdx.x == 0) {
      for (int dim = 0; dim < DIM; ++dim) {
        T lower = dRegions[dim * numRegions + index];

        sRegionPool[threadIdx.x].bounds[dim].lower = 0;
        sRegionPool[threadIdx.x].bounds[dim].upper = 1;

        sBound[dim].unScaledLower = lower;
        sBound[dim].unScaledUpper =
          lower + dRegionsLength[dim * numRegions + index];
        // printf("[%i]%.12f - %.12f\n", blockIdx.x, lower,
        // sBound[dim].unScaledUpper); printf("Region %i dim[%i] from
        // dRegionsLength[%lu]\n", blockIdx.x, dim, dim * numRegions + index);
        sRegionPool[threadIdx.x].div = 0;
      }
      // printf("\n");
    }

    __syncthreads();
    SampleRegionBlock<T>(0, constMem, FEVAL, NSETS);
    __syncthreads();
  }

  template <typename T>
  __global__ void
  RefineError(T* dRegionsIntegral,
              T* dRegionsError,
              T* dParentsIntegral,
              T* dParentsError,
              T* newErrs,
              int* activeRegions,
              int numRegions,
              T epsrel,
              T epsabs)
  {

    if (threadIdx.x == 0 && blockIdx.x < numRegions) {
      int fail = 0;

      T selfErr = dRegionsError[blockIdx.x + numRegions];
      T selfRes = dRegionsIntegral[blockIdx.x + numRegions];

      // that's how indices to the right to find the sibling
      // but we want the sibling to be found at the second half of the array
      // only, to avoid race conditions
      int siblingIndex = (numRegions / 2) + blockIdx.x;
      // printf("[%i] si:%i\n", blockIdx.x, siblingIndex);
      if (siblingIndex < numRegions)
        siblingIndex += numRegions;
      // T siblErr 	= dRegionsError[blockIdx.x + (numRegions/2)];
      // T siblRes 	= dRegionsIntegral[blockIdx.x + (numRegions/2)];

      T siblErr = dRegionsError[siblingIndex];
      T siblRes = dRegionsIntegral[siblingIndex];

      T parRes = dParentsIntegral[blockIdx.x];

      T diff = siblRes + selfRes - parRes;
      diff = fabs(.25 * diff);

      T err = selfErr + siblErr;

      /*if(blockIdx.x<10)
              printf("[%i] Refining %.12f +- %.12f (sibling:%.12f parent:%.12f
         +- %.12f) sibIndex:%i err:%.12f diff:%.12f numRegions:%i\n",
         blockIdx.x, selfRes, selfErr, siblErr, parRes, parErr, siblingIndex ,
                                                                                                                                                                                                                                              err,
                                                                                                                                                                                                                                              diff,
                                                                                                                                                                                                                                              numRegions);*/

      if (err > 0.0) {
        T c = 1 + 2 * diff / err;
        selfErr *= c;
        // dRegionsError[gridDim.x + blockIdx.x] 	*=c;
      }

      selfErr += diff;

      if ((selfErr / MaxErr(selfRes, epsrel, epsabs)) > 1) {
        fail = 1;
        newErrs[blockIdx.x] = 0;
        dRegionsIntegral[blockIdx.x] = 0;
        // if(blockIdx.x<10)

        // printf("[%i]Bad %.12f  +- %.12f\n", blockIdx.x, selfRes, selfErr);
        // selfErr = 0;
        // printf("[%i] Bad  %.12f ---Refinement---> %.12f\n", blockIdx.x,
        // selfErr);
      } else {
        newErrs[blockIdx.x] = selfErr;
        // if(blockIdx.x<10)
        // printf("[%i]Good %.18f  +- %.18f\n", blockIdx.x, selfRes, selfErr);
      }

      activeRegions[blockIdx.x] = fail;
      newErrs[blockIdx.x + numRegions] = selfErr;
    }
  }

  template <typename T>
  __global__ void
  INTEGRATE_GPU_PHASE1(T* dRegions,
                       T* dRegionsLength,
                       size_t numRegions,
                       T* dRegionsIntegral,
                       T* dRegionsError,
                       T* dParentsIntegral,
                       T* dParentsError,
                       int* activeRegions,
                       int* subDividingDimension,
                       T epsrel,
                       T epsabs,
                       Structures<T> constMem,
                       int FEVAL,
                       int NSETS)
  {

    T ERR = 0, RESULT = 0;
    int fail = 0;

    INIT_REGION_POOL(
      dRegions, dRegionsLength, numRegions, &constMem, FEVAL, NSETS);

    if (threadIdx.x == 0) {
      ERR = sRegionPool[threadIdx.x].result.err;
      RESULT = sRegionPool[threadIdx.x].result.avg;
      T ratio = ERR / MaxErr(RESULT, epsrel, epsabs);
      int fourthDiffDim = sRegionPool[threadIdx.x].result.bisectdim;

      dRegionsIntegral[gridDim.x + blockIdx.x] = RESULT;
      dRegionsError[gridDim.x + blockIdx.x] = ERR;

      if (ratio > 1) {
        fail = 1;
        ERR = 0;
        RESULT = 0;
      }

      activeRegions[blockIdx.x] = fail;
      subDividingDimension[blockIdx.x] = fourthDiffDim;
      dRegionsIntegral[blockIdx.x] = RESULT;
      dRegionsError[blockIdx.x] = ERR;

      __syncthreads();

      /*if(ratio>1 && numRegions == 1){
            dRegionsIntegral[blockIdx.x] 			= 0;
            dRegionsError[blockIdx.x]				= 0;
      }*/
    }
  }

  template <typename T>
  __global__ void
  INTEGRATE_GPU_PHASE12(T* dRegions,
                        T* dRegionsLength,
                        size_t numRegions,
                        T* dRegionsIntegral,
                        T* dRegionsError,
                        T* dParentsIntegral,
                        T* dParentsError,
                        int* activeRegions,
                        int* subDividingDimension,
                        T epsrel,
                        T epsabs,
                        Structures<T> constMem,
                        int FEVAL,
                        int NSETS)
  {

    T ERR = 0, RESULT = 0;
    int fail = 0;

    INIT_REGION_POOL(
      dRegions, dRegionsLength, numRegions, &constMem, FEVAL, NSETS);

    if (threadIdx.x == 0) {
      ERR = sRegionPool[threadIdx.x].result.err;
      RESULT = sRegionPool[threadIdx.x].result.avg;
      T ratio = ERR / MaxErr(RESULT, epsrel, epsabs);
      int fourthDiffDim = sRegionPool[threadIdx.x].result.bisectdim;
      // printf("[%i] bisectDim:%i\n", blockIdx.x, fourthDiffDim);
      dRegionsIntegral[gridDim.x + blockIdx.x] = RESULT;
      dRegionsError[gridDim.x + blockIdx.x] = ERR;
      // printf("Unrefined %.18f +- %.18f\n", RESULT, ERR);
      if (ratio > 1) {
        fail = 1;
      }

      activeRegions[blockIdx.x] = fail;
      subDividingDimension[blockIdx.x] = fourthDiffDim;
      dRegionsIntegral[blockIdx.x] = RESULT;
      dRegionsError[blockIdx.x] = ERR;

      __syncthreads();

      if (ratio > 1 && numRegions == 1) {
        dRegionsIntegral[blockIdx.x] = 0;
        dRegionsError[blockIdx.x] = 0;
      }
    }
  }

  ////PHASE 2 Procedures Starts
  template <typename T>
  __device__ void
  ComputeErrResult(T& ERR, T& RESULT)
  {
    /*sdata[threadIdx.x] = sRegionPool[threadIdx.x].result.err;
    sdata[blockDim.x + threadIdx.x] = sRegionPool[threadIdx.x].result.avg;
    __syncthreads();

    // contiguous range pattern
    for(size_t offset = size / 2; offset > 0; offset >>= 1){
      if(threadIdx.x < offset){
        sdata[threadIdx.x] += sdata[threadIdx.x + offset];
        sdata[blockDim.x + threadIdx.x] += sdata[blockDim.x + threadIdx.x +
    offset];
      }
      __syncthreads();
    }
    */
    if (threadIdx.x == 0) {
      ERR = sRegionPool[threadIdx.x].result.err;
      RESULT = sRegionPool[threadIdx.x].result.avg;
    }
    __syncthreads();
  }

  template <typename T>
  __device__ int
  INIT_REGION_POOL(T* dRegions,
                   T* dRegionsLength,
                   int* subDividingDimension,
                   size_t numRegions,
                   Structures<T>* constMem,
                   int FEVAL,
                   int NSETS)
  {

    size_t intervalIndex = blockIdx.x;
    int idx = 0;

    // idx<0 always? SM_R = 128 (quad.h) BLOCK_SIZE=256
    for (; idx < SM_REGION_POOL_SIZE / BLOCK_SIZE; ++idx) {

      int index = idx * BLOCK_SIZE + threadIdx.x;
      sRegionPool[index].div = 0;
      sRegionPool[index].result.err = 0;
      sRegionPool[index].result.avg = 0;
      sRegionPool[index].result.bisectdim = 0;

      for (int dim = 0; dim < DIM; ++dim) {
        sRegionPool[index].bounds[dim].lower = 0;
        sRegionPool[index].bounds[dim].upper = 0;
      }
    }

    int index = idx * BLOCK_SIZE + threadIdx.x; // essentially threadIdx.x
    if (index < SM_REGION_POOL_SIZE) {

      sRegionPool[index].div = 0;
      sRegionPool[index].result.err = 0;
      sRegionPool[index].result.avg = 0;
      sRegionPool[index].result.bisectdim = 0;

      for (int dim = 0; dim < DIM; ++dim) {
        sRegionPool[index].bounds[dim].lower = 0;
        sRegionPool[index].bounds[dim].upper = 0;
      }
    }

    // gets unscaled lower and upper bounds for region
    if (threadIdx.x == 0) {
      for (int dim = 0; dim < DIM; ++dim) {

        sRegionPool[threadIdx.x].bounds[dim].lower = 0;
        sRegionPool[threadIdx.x].bounds[dim].upper = 1;
        T lower = dRegions[dim * numRegions + intervalIndex];
        sBound[dim].unScaledLower = lower;
        sBound[dim].unScaledUpper =
          lower + dRegionsLength[dim * numRegions + intervalIndex];
      }
    }

    __syncthreads();

    SampleRegionBlock<T>(0, constMem, FEVAL, NSETS);

    if (threadIdx.x == 0) {
      gPool = (Region*)malloc(sizeof(Region) * (SM_REGION_POOL_SIZE / 2));
      gRegionPoolSize = (SM_REGION_POOL_SIZE / 2); // BLOCK_SIZE;
    }

    __syncthreads();

    for (idx = 0; idx < (SM_REGION_POOL_SIZE / 2) / BLOCK_SIZE; ++idx) {
      int index = idx * BLOCK_SIZE + threadIdx.x;
      gRegionPos[index] = index;
      gPool[index] = sRegionPool[index];
    }

    index = idx * BLOCK_SIZE + threadIdx.x;
    if (index < (SM_REGION_POOL_SIZE / 2)) {
      gRegionPos[index] = index;
      gPool[index] = sRegionPool[index];
    }
    return 1;
  }

  template <typename T>
  __device__ int
  INIT_REGION_POOL2(T* dRegions,
                    T* dRegionsLength,
                    int* subDividingDimension,
                    size_t numRegions,
                    Structures<T>* constMem,
                    int FEVAL,
                    int NSETS)
  {

    size_t intervalIndex = blockIdx.x;
    int idx = 0;

    // idx<0 always? SM_R = 128 (quad.h) BLOCK_SIZE=256
    for (; idx < SM_REGION_POOL_SIZE / BLOCK_SIZE; ++idx) {

      int index = idx * BLOCK_SIZE + threadIdx.x;
      sRegionPool[index].div = 0;
      sRegionPool[index].result.err = 0;
      sRegionPool[index].result.avg = 0;
      sRegionPool[index].result.bisectdim = 0;

      for (int dim = 0; dim < DIM; ++dim) {
        sRegionPool[index].bounds[dim].lower = 0;
        sRegionPool[index].bounds[dim].upper = 0;
      }
    }

    int index = idx * BLOCK_SIZE + threadIdx.x; // essentially threadIdx.x
    if (index < SM_REGION_POOL_SIZE) {

      sRegionPool[index].div = 0;
      sRegionPool[index].result.err = 0;
      sRegionPool[index].result.avg = 0;
      sRegionPool[index].result.bisectdim = 0;

      for (int dim = 0; dim < DIM; ++dim) {
        sRegionPool[index].bounds[dim].lower = 0;
        sRegionPool[index].bounds[dim].upper = 0;
      }
    }

    // gets unscaled lower and upper bounds for region
    if (threadIdx.x == 0) {
      for (int dim = 0; dim < DIM; ++dim) {

        sRegionPool[threadIdx.x].bounds[dim].lower = 0;
        sRegionPool[threadIdx.x].bounds[dim].upper = 1;
        T lower = dRegions[dim * numRegions + intervalIndex];
        sBound[dim].unScaledLower = lower;
        sBound[dim].unScaledUpper =
          lower + dRegionsLength[dim * numRegions + intervalIndex];
      }
    }

    __syncthreads();

    SampleRegionBlock<T>(0, constMem, FEVAL, NSETS);

    if (threadIdx.x == 0) {
      Region* ptr = (Region*)malloc(sizeof(Region) * (SM_REGION_POOL_SIZE / 2));
      //  gRegionPoolSize = (SM_REGION_POOL_SIZE/2);//BLOCK_SIZE;
    }

    __syncthreads();

    for (idx = 0; idx < (SM_REGION_POOL_SIZE / 2) / BLOCK_SIZE; ++idx) {
      int index = idx * BLOCK_SIZE + threadIdx.x;
      gRegionPos[index] = index;
      gPool[index] = sRegionPool[index];
    }

    index = idx * BLOCK_SIZE + threadIdx.x;
    if (index < (SM_REGION_POOL_SIZE / 2)) {
      gRegionPos[index] = index;
      gPool[index] = sRegionPool[index];
    }
    return 1;
  }

  template <class T>
  __device__ void
  swap(T& a, T& b)
  {
    T c(a);
    a = b;
    b = c;
  }

  template <typename T>
  __device__ void
  INSERT_GLOBAL_STORE(Region* sRegionPool, Region* gRegionPool, int gpuId)
  {

    if (threadIdx.x == 0) {
      // if(blockIdx.x == 0)
      //		printf("Block 0 allocated:%i\n", gRegionPoolSize*2);
      gPool = (Region*)malloc(sizeof(Region) *
                              (gRegionPoolSize + (SM_REGION_POOL_SIZE / 2)));
      if (gPool == NULL) {
        printf("Failed to malloc at block:%i threadIndex:%i gpu:%i "
               "currentSize:%lu requestedSize:%lu\n",
               blockIdx.x,
               threadIdx.x,
               gpuId,
               gRegionPoolSize,
               gRegionPoolSize + ((size_t)SM_REGION_POOL_SIZE / 2));
      }
    }
    __syncthreads();

    // Copy existing global regions into newly allocated spaced
    // This loop activates when gRegionPoolSize is at least 256, must be
    // expanded three times
    int iterationsPerThread = 0;
    for (iterationsPerThread = 0;
         iterationsPerThread < gRegionPoolSize / BLOCK_SIZE;
         ++iterationsPerThread) {
      size_t dataIndex = iterationsPerThread * BLOCK_SIZE + threadIdx.x;

      gPool[dataIndex] = gRegionPool[dataIndex];
      __syncthreads();
    }

    // if above loop didnt' activate, we enter this stament with dataIndex =
    // threadIdx.x else we enter this statement to finish last batch of copies
    // with dataIndex = multiple of threadIdx.x
    size_t dataIndex = iterationsPerThread * BLOCK_SIZE + threadIdx.x;
    if (dataIndex < gRegionPoolSize) {
      gPool[dataIndex] = gRegionPool[dataIndex];
    }

    // the loop and if statement above, copied from global memory to global
    // memory, AKA took care of the extension

    // Fill the previous occupied postion in global memory by half of shared
    // memory regions THIS IS ONLY EXECUTED WHEN BLOCK SIZE IS MUCH SMALLER THAN
    // SM_REGION_POOL_SIZE otherwise we never enter the loop
    for (iterationsPerThread = 0;
         iterationsPerThread < (SM_REGION_POOL_SIZE / 2) / BLOCK_SIZE;
         ++iterationsPerThread) {
      int index = iterationsPerThread * BLOCK_SIZE + threadIdx.x;
      gPool[gRegionPos[index]] = sRegionPool[index];
      gPool[gRegionPoolSize + index] =
        sRegionPool[(SM_REGION_POOL_SIZE / 2) + index];
    }

    // if above loop was not entered
    // we do the copies here with index = threadIdx.x
    // otherwise, index = multiple of threadIdx.x
    int index = iterationsPerThread * BLOCK_SIZE + threadIdx.x;
    if (index < (SM_REGION_POOL_SIZE / 2)) {
      int index = iterationsPerThread * BLOCK_SIZE + threadIdx.x;
      gPool[gRegionPos[index]] = sRegionPool[index];
      gPool[gRegionPoolSize + index] =
        sRegionPool[(SM_REGION_POOL_SIZE / 2) + index];
    }

    __syncthreads();
    if (threadIdx.x == 0) {
      gRegionPoolSize = gRegionPoolSize + (SM_REGION_POOL_SIZE / 2);
      free(gRegionPool);
    }
    __syncthreads();

    gRegionPool = gPool;
    // gSize += BLOCK_SIZE;

    // return gSize;
  }

  template <typename T>
  __device__ void
  EXTRACT_MAX(T* serror, size_t* serrorPos, size_t gSize)
  {

    for (size_t offset = gSize / 2; offset > 0; offset >>= 1) {
      int idx = 0;
      for (idx = 0; idx < offset / BLOCK_SIZE; ++idx) {
        size_t index = idx * BLOCK_SIZE + threadIdx.x;
        if (index < offset) {
          if (serror[index] < serror[index + offset]) {
            swap(serror[index], serror[index + offset]);
            swap(serrorPos[index], serrorPos[index + offset]);
          }
          // printf("%ld %ld\n",index, index+offset);
        }
      }
      size_t index = idx * BLOCK_SIZE + threadIdx.x;
      if (index < offset) {
        if (serror[index] < serror[index + offset]) {
          swap(serror[index], serror[index + offset]);
          swap(serrorPos[index], serrorPos[index + offset]);
        }
      }
      __syncthreads();
    }
  }

  template <typename T>
  __device__ void
  EXTRACT_TOPK(Region* sRegionPool, Region* gRegionPool)
  {

    // Comment 3 instructions these section if you are directly using new shared
    // memory instead of reusing shared memory

    T* sarray = (T*)&sRegionPool[0];

    if (threadIdx.x == 0) {
      // T *sarray = (T *)&sRegionPool[0];

      if ((gRegionPoolSize * sizeof(T) + gRegionPoolSize * sizeof(size_t)) <
          sizeof(Region) * SM_REGION_POOL_SIZE) {
        serror = &sarray[0];
        // TODO:Size of sRegionPool vs sarray constrain
        serrorPos = (size_t*)&sarray[gRegionPoolSize];
      } else {
        // if(blockIdx.x == 0)
        //			printf("Block 0 error allocated:%i\n",
        // gRegionPoolSize);
        serror = (T*)malloc(sizeof(T) * gRegionPoolSize);
        serrorPos = (size_t*)malloc(sizeof(size_t) * gRegionPoolSize);
      }
    }
    __syncthreads();

    int offset = 0;
    for (offset = 0; (offset < MAX_GLOBALPOOL_SIZE / BLOCK_SIZE) &&
                     (offset < gRegionPoolSize / BLOCK_SIZE);
         offset++) {
      size_t regionIndex = offset * BLOCK_SIZE + threadIdx.x;
      serror[regionIndex] = gRegionPool[regionIndex].result.err;
      serrorPos[regionIndex] = regionIndex;
    }
    size_t regionIndex = offset * BLOCK_SIZE + threadIdx.x;
    if (regionIndex < gRegionPoolSize) {
      serror[regionIndex] = gRegionPool[regionIndex].result.err;
      serrorPos[regionIndex] = regionIndex;
    }

    __syncthreads();
    for (int k = 0; k < (SM_REGION_POOL_SIZE / 2); ++k) {
      EXTRACT_MAX<T>(&serror[k], &serrorPos[k], gRegionPoolSize - k);
    }

    int iterationsPerThread = 0;
    for (iterationsPerThread = 0;
         iterationsPerThread < (SM_REGION_POOL_SIZE / 2) / BLOCK_SIZE;
         ++iterationsPerThread) {
      int index = iterationsPerThread * BLOCK_SIZE + threadIdx.x;
      size_t pos = serrorPos[index];
      gRegionPos[index] = pos;
    }
    int index = iterationsPerThread * BLOCK_SIZE + threadIdx.x;
    if (index < (SM_REGION_POOL_SIZE / 2)) {
      size_t pos = serrorPos[index];
      gRegionPos[index] = pos;
    }

    // Old
    // size_t pos = serrorPos[threadIdx.x];
    // gRegionPos[threadIdx.x] = pos;
    __syncthreads();

    if (threadIdx.x == 0) {
      // sRegionPool = (Region *)sarray;
      if (2 * gRegionPoolSize * sizeof(T) >=
          sizeof(Region) * SM_REGION_POOL_SIZE) {
        free(serror);
        free(serrorPos);
      }
    }
    __syncthreads();

    /*if((2*gRegionPoolSize*sizeof(T) >= sizeof(Region) * SM_REGION_POOL_SIZE)
      && threadIdx.x == 0){ free(serror); free(serrorPos);
      }*/

    // Copy top K into SM and reset the remaining
    for (iterationsPerThread = 0;
         iterationsPerThread < (SM_REGION_POOL_SIZE / 2) / BLOCK_SIZE;
         ++iterationsPerThread) {
      int index = iterationsPerThread * BLOCK_SIZE + threadIdx.x;
      sRegionPool[index] = gPool[gRegionPos[index]];
      sRegionPool[(SM_REGION_POOL_SIZE / 2) + index].result.err = -INFTY;
      sRegionPool[(SM_REGION_POOL_SIZE / 2) + index].result.avg = 0;
      sRegionPool[(SM_REGION_POOL_SIZE / 2) + index].div = 0;
    }

    index = iterationsPerThread * BLOCK_SIZE + threadIdx.x;
    if (index < (SM_REGION_POOL_SIZE / 2)) {
      sRegionPool[index] = gPool[gRegionPos[index]];
      sRegionPool[(SM_REGION_POOL_SIZE / 2) + index].result.err = -INFTY;
      sRegionPool[(SM_REGION_POOL_SIZE / 2) + index].result.avg = 0;
      sRegionPool[(SM_REGION_POOL_SIZE / 2) + index].div = 0;
    }

    // Old
    // sRegionPool[threadIdx.x] = gPool[pos];

    // sRegionPool[BLOCK_SIZE+threadIdx.x].result.err = -INFTY;
    // sRegionPool[BLOCK_SIZE+threadIdx.x].result.avg = 0;
    // sRegionPool[BLOCK_SIZE+threadIdx.x].div = 0;
  }

  template <typename T>
  __device__ size_t
  EXTRACT_MAX(Region* sRegionPool, Region* gRegionPool, size_t sSize, int gpuId)
  {
    // If SharedPool is full
    if (sSize == SM_REGION_POOL_SIZE) {

      INSERT_GLOBAL_STORE<T>(sRegionPool, gRegionPool, gpuId);
      __syncthreads();

      gRegionPool = gPool;
      EXTRACT_TOPK<T>(sRegionPool, gRegionPool);
      sSize = (SM_REGION_POOL_SIZE / 2);
      __syncthreads();
    }

    for (size_t offset = (SM_REGION_POOL_SIZE / 2); offset > 0; offset >>= 1) {
      int idx = 0;
      for (idx = 0; idx < offset / BLOCK_SIZE; ++idx) {
        size_t index = idx * BLOCK_SIZE + threadIdx.x;
        if (index < offset) {
          Region* r1 = &sRegionPool[index];
          Region* r2 = &sRegionPool[index + offset];
          if (r1->result.err < r2->result.err) {
            swap<Region>(sRegionPool[index], sRegionPool[offset + index]);
          }
        }
      }

      size_t index = idx * BLOCK_SIZE + threadIdx.x;
      if (index < offset) {
        Region* r1 = &sRegionPool[index];
        Region* r2 = &sRegionPool[index + offset];
        if (r1->result.err < r2->result.err) {
          swap<Region>(sRegionPool[index], sRegionPool[offset + index]);
        }
      }
      __syncthreads();
    }

    return sSize;
  }

  template <typename T>
  __device__ size_t
  EXTRACT_MAX2(Region* sRegionPool,
               Region* gRegionPool,
               size_t sSize,
               int gpuId)
  {
    // If SharedPool is full
    if (sSize == SM_REGION_POOL_SIZE) {

      INSERT_GLOBAL_STORE<T>(sRegionPool, gRegionPool, gpuId);
      __syncthreads();

      gRegionPool = gPool;
      EXTRACT_TOPK<T>(sRegionPool, gRegionPool);
      sSize = (SM_REGION_POOL_SIZE / 2);
      __syncthreads();
    }

    for (size_t offset = (SM_REGION_POOL_SIZE / 2); offset > 0; offset >>= 1) {
      int idx = 0;
      for (idx = 0; idx < offset / BLOCK_SIZE; ++idx) {
        size_t index = idx * BLOCK_SIZE + threadIdx.x;
        if (index < offset) {
          Region* r1 = &sRegionPool[index];
          Region* r2 = &sRegionPool[index + offset];
          if (r1->result.err < r2->result.err) {
            swap<Region>(sRegionPool[index], sRegionPool[offset + index]);
          }
        }
      }

      size_t index = idx * BLOCK_SIZE + threadIdx.x;
      if (index < offset) {
        Region* r1 = &sRegionPool[index];
        Region* r2 = &sRegionPool[index + offset];
        if (r1->result.err < r2->result.err) {
          swap<Region>(sRegionPool[index], sRegionPool[offset + index]);
        }
      }
      __syncthreads();
    }

    return sSize;
  }

  template <typename T>
  __global__ void
  BLOCK_INTEGRATE_GPU_PHASE2(T* dRegions,
                             T* dRegionsLength,
                             size_t numRegions,
                             T* dRegionsIntegral,
                             T* dRegionsError,
                             int* dRegionsNumRegion,
                             int* activeRegions,
                             int* subDividingDimension,
                             T epsrel,
                             T epsabs,
                             int gpuId,
                             Structures<T> constMem,
                             int FEVAL,
                             int NSETS,
                             double* exitCondition)
  {

    /*if(threadIdx.x == 0){
            printf("[%i] Initial ERR:%.12f\n", blockIdx.x, ERR);
    }*/

    Region* gRegionPool = 0;
    int sRegionPoolSize = INIT_REGION_POOL<T>(dRegions,
                                              dRegionsLength,
                                              subDividingDimension,
                                              numRegions,
                                              &constMem,
                                              FEVAL,
                                              NSETS);

    ComputeErrResult<T>(ERR, RESULT);
    // TODO : May be redundance sync
    __syncthreads();

    int nregions = sRegionPoolSize; // is only 1 at this point

    // commented out by Ioannis
    // max pool size:2048
    // for(; (nregions <= MAX_GLOBALPOOL_SIZE) && (nregions == 1 || ERR >
    // MaxErr(RESULT, epsrel, epsabs)); ++nregions )
    /*if(threadIdx.x == 0){
            printf("[%i]Here %f +- %f (contributing values %f +- %f)\n",
    blockIdx.x, exitCondition[1], exitCondition[0], RESULT, ERR);
    }*/

    if (threadIdx.x == 0 && blockIdx.x < 10)
      printf("[%i] Phase 2 Bad %.12f +- %.12f MaxRegions:%lu\n",
             blockIdx.x,
             RESULT,
             ERR,
             MAX_GLOBALPOOL_SIZE);

    while (nregions <= MAX_GLOBALPOOL_SIZE &&
           ERR > MaxErr(RESULT, epsrel, epsabs)) {

      /*if(threadIdx.x == 0){
              printf("[%i]ERR:%.12f\n", blockIdx.x, ERR);
              printf("[%i]exitCondition[1]:%.12f\n", blockIdx.x,
      exitCondition[1]);
      }*/

      gRegionPool = gPool;
      sRegionPoolSize =
        EXTRACT_MAX<T>(sRegionPool, gRegionPool, sRegionPoolSize, gpuId);
      Region *RegionLeft, *RegionRight;
      Result result;

      if (threadIdx.x == 0) {
        Bounds *bL, *bR;
        Region* R = &sRegionPool[0];
        result.err = R->result.err;
        result.avg = R->result.avg;
        result.bisectdim = R->result.bisectdim;

        int bisectdim = result.bisectdim;

        RegionLeft = R;
        RegionRight = &sRegionPool[sRegionPoolSize];

        bL = &RegionLeft->bounds[bisectdim];
        bR = &RegionRight->bounds[bisectdim];

        // TODO: What does div do!
        RegionRight->div = ++RegionLeft->div;
        for (int dim = 0; dim < DIM; ++dim) {
          RegionRight->bounds[dim].lower = RegionLeft->bounds[dim].lower;
          RegionRight->bounds[dim].upper = RegionLeft->bounds[dim].upper;
        }
        // Subdivide the chosen axis
        bL->upper = bR->lower = 0.5 * (bL->lower + bL->upper);
      }

      sRegionPoolSize++;

      __syncthreads();
      SampleRegionBlock<T>(0, &constMem, FEVAL, NSETS);
      __syncthreads();
      SampleRegionBlock<T>(sRegionPoolSize - 1, &constMem, FEVAL, NSETS);
      __syncthreads();

      // update ERR & RESULT
      if (threadIdx.x == 0) {
        Result* rL = &RegionLeft->result;
        Result* rR = &RegionRight->result;

        T diff = rL->avg + rR->avg - result.avg;
        diff = fabs(.25 * diff);
        T err = rL->err + rR->err;
        if (err > 0) {
          T c = 1 + 2 * diff / err;
          rL->err *= c;
          rR->err *= c;
        }
        rL->err += diff;
        rR->err += diff;

        ERR += rL->err + rR->err - result.err;
        RESULT += rL->avg + rR->avg - result.avg;

        // atomicAdd(&exitCondition[0], ERR);
        // atomicAdd(&exitCondition[1], RESULT);
      }
      __syncthreads();
    }

    if (threadIdx.x == 0) {

      int isActive = ERR > MaxErr(RESULT, epsrel, epsabs);

      if (/*(nregions > MAX_GLOBALPOOL_SIZE) || isActive || */ ERR > (1e+10)) {
        // printf("Bad region at block:%i\n", blockIdx.x);

        RESULT = 0.0;
        ERR = 0.0;
        isActive = 1;
      }

      activeRegions[blockIdx.x] = isActive;
      dRegionsIntegral[blockIdx.x] = RESULT;
      dRegionsError[blockIdx.x] = ERR;
      dRegionsNumRegion[blockIdx.x] = nregions;

      free(gPool);
    }

    // if(threadIdx.x == 0 && blockIdx.x == 0)
    //	printf("exiting max pool size:%i\n", MAX_GLOBALPOOL_SIZE);
  }
}
