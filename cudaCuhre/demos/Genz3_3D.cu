#include "function.cuh"
#include "demo_utils.cuh"
#include <chrono>
#include <cmath>
#include <fstream>
#include <iomanip>
#include <iostream>

using namespace quad;

int
main()
{
  double epsrel = 1.0e-3; // starting error tolerance.
  double const epsrel_min = 1.0e-12;
  double true_value = 0.010846560846560846561;
  GENZ_3_3D integrand;
 
  Config configuration;
  configuration.outfileVerbosity = 0;
  configuration.heuristicID = 4;
  constexpr int ndim = 3;
  
  PrintHeader();
  while (cu_time_and_call<GENZ_3_3D, ndim>("pdc_f1_latest",
                       integrand,
                       epsrel,
                       true_value,
                       "gpucuhre",
                       std::cout,
                       configuration) == true &&
         epsrel >= epsrel_min) {
    epsrel /= 5.0;
  }
}
