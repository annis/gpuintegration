#include "demo_utils.cuh"
#include <cmath>
#include <fstream>
#include <iomanip>
#include <iostream>

using namespace quad;

int
main()
{
  double epsrel = 1.0e-3; // starting error tolerance.
  double const epsrel_min = 1.0e-13;
  double true_value = 23434.04;
  GENZ_2_2D integrand;
  constexpr int ndim = 2;
  
  PrintHeader();
  while (cu_time_and_call<GENZ_2_2D, ndim>("GENZ_2_2D",
                       integrand,
                       epsrel,
                       true_value,
                       std::cout) == true &&
         epsrel >= epsrel_min) {
    epsrel /= 5.0;
  }
}
