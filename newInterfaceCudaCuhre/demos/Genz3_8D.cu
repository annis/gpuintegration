#include "hip/hip_runtime.h"
#include "demo_utils.cuh"
#include <cmath>
#include <fstream>
#include <iomanip>
#include <iostream>

using namespace quad;

namespace detail{
    class GENZ_3_8D {
        public:
          __device__ __host__ double
          operator()(double x, double y, double z, double w, double v, double u, double t, double s)
          {
            return pow(1+8*s+7*t+6*u+5*v+4*w+3*x+2*y+z, -9)/*/2.2751965817917756076e-10*/;
          }
    };
}

int
main()
{
  double epsrel = 1e-3;
  double const epsrel_min = 1.024e-10;
  double true_value = 2.2751965817917756076e-10;
  detail::GENZ_3_8D integrand;
  PrintHeader();
  
  constexpr int ndim = 8;
  
  while (cu_time_and_call<detail::GENZ_3_8D, ndim>("Genz3_8D",
                           integrand,
                           epsrel,
                           true_value,
                           std::cout) == true &&
             epsrel > epsrel_min) {
    epsrel /= 5.0;
   }

  
}
