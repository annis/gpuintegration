#include "hip/hip_runtime.h"
/*

nvcc -O2 -DCUSTOM -o vegas1D vegas_mcubes_1D.cu -arch=sm_70
OR
nvcc -O2 -DCURAND -o vegas1D vegas_mcubes_1D.cu -arch=sm_70

example run command

nvprof ./vegas1D 0 6 0.0  10.0  2.0E+09  58, 0, 0

nvprof  ./vegas1D 1 9 -1.0  1.0  1.0E+07 15 10 10

nvprof ./vegas1D 2 2 -1.0 1.0  1.0E+09 1 0 0

Last three arguments are: total iterations, iteration

 */
#include <chrono>
#include <stdio.h>
//#include <malloc.h>
#include <stdlib.h>
#include <math.h>
#include <hiprand/hiprand_kernel.h>
#include <stdint.h>
#include <ctime>
#include "func.cuh"
#include <iostream>

#define WARP_SIZE 32
#define BLOCK_DIM_X 128
#define ALPH 1.5
#define NDMX  500
#define MXDIM 20

#define NDMX1 NDMX+1
#define MXDIM1 MXDIM+1
//#define SCALE 1.0E+10
#define SCALE 1.0E-200

#define IMAX(a,b) \
    ({ __typeof__ (a) _a = (a); \
      __typeof__ (b) _b = (b); \
      _a > _b ? _a : _b; })

#define IMIN(a,b) \
    ({ __typeof__ (a) _a = (a); \
      __typeof__ (b) _b = (b); \
      _a < _b ? _a : _b; })


//Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError() {                                          \
        hipError_t e=hipGetLastError();                                 \
        if(e!=hipSuccess) {                                              \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
            exit(EXIT_FAILURE);                                           \
        }                                                                 \
}

using MilliSeconds = std::chrono::duration<double, std::chrono::milliseconds::period>;
int verbosity = 0;
double GetTrueValue(int fcode){
    if(fcode == 0)
        return -49.165073;
    else if(fcode == 1)
        return 1.0;
    else if(fcode == 4)
        return (1. / 315.) * sin(1.) * sin(3. / 2.) * sin(2.) * sin(5. / 2.) * sin(3.) *
    sin(7. / 2.) * sin(4.) *
    (sin(37. / 2.) - sin(35. / 2.));
    else if(fcode == 9)
        return 2.425217625641885e-06;
    else if(fcode == 10)
        return 8879.851175413485;
    else if(fcode == 11)
        return 1.5477367885091207413e8;
    else if(fcode == 12)
        return 6.383802190004379e-10;
    else if(fcode == 13)
        return  2.2751965817917756076e-10;
    return 0.;
}

__inline__ __device__
double warpReduceSum(double val) {
        val += __shfl_down_sync(0xffffffff, val, 16, WARP_SIZE);
        val += __shfl_down_sync(0xffffffff, val, 8, WARP_SIZE);
        val += __shfl_down_sync(0xffffffff, val, 4, WARP_SIZE);
        val += __shfl_down_sync(0xffffffff, val, 2, WARP_SIZE);
        val += __shfl_down_sync(0xffffffff, val, 1, WARP_SIZE);
        return val;
}

__inline__ __device__
double blockReduceSum(double val) {

        static __shared__ double shared[32]; // Shared mem for 32 partial sums
        int lane = threadIdx.x % warpSize;
        int wid = threadIdx.x / warpSize;

        val = warpReduceSum(val);     // Each warp performs partial reduction

        if (lane == 0) shared[wid] = val; // Write reduced value to shared memory

        __syncthreads();              // Wait for all partial reductions

        //read from shared memory only if that warp existed
        val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;

        if (wid == 0) val = warpReduceSum(val); //Final reduce within first warp

        return val;
}

__inline__ __device__  void get_indx(int ms, int *da, int ND, int NINTV) {
        int dp[MXDIM];
        int j, t0, t1;
        int m = ms;
        dp[0] = 1;
        dp[1] = NINTV;


        for (j = 0; j < ND - 2; j++) {
                dp[j + 2] = dp[j + 1] * NINTV;
        }
        //
        for (j = 0; j < ND; j++) {
                t0 = dp[ND - j - 1];
                t1 = m / t0;
                da[j] = 1 + t1;
                m = m - t1 * t0;

        }
}

__global__ void vegas_kernel(int ng, int ndim, int npg, double xjac, double dxg,
                             double *result_dev, double xnd, double *xi,
                             double *d, double *dx, double *regn, int ncubes,
                             int iter, double sc, double sci, double ing,
                             int chunkSize, uint32_t totalNumThreads,
                             int LastChunk, int fcode) {

#ifdef CUSTOM
        uint64_t temp;
        uint32_t a = 1103515245;
        uint32_t c = 12345;
        uint32_t one, expi;
        one = 1;
        expi = 31;
        uint32_t p = one << expi;
#endif

        uint32_t seed, seed_init;
        seed_init = (iter) * ncubes;

        int m = blockIdx.x * blockDim.x + threadIdx.x;
        int tx = threadIdx.x;

        double fb, f2b, wgt, xn, xo, rc, f, f2, ran00;
        int kg[MXDIM + 1];
        int ia[MXDIM + 1];
        double x[MXDIM + 1];
        int k, j;
        double fbg, f2bg;
        //if(tx == 30 && blockIdx.x == 6771) printf("here m is %d\n", m);

        if (m < totalNumThreads) {
                if (m == totalNumThreads - 1) chunkSize = LastChunk + 1;
                //if(tx == 30 && blockIdx.x == 6771) printf("here m is %d\n", m);
                seed = seed_init + m * chunkSize;
#ifdef CURAND
                hiprandState localState;
                hiprand_init(seed, 0, 0, &localState);
#endif
                fbg = f2bg = 0.0;
                get_indx(m * chunkSize, &kg[1], ndim, ng);
                for (int t = 0; t < chunkSize; t++) {
                        fb = f2b = 0.0;
                        //get_indx(m*chunkSize+t, &kg[1], ndim, ng);
                        for ( k = 1; k <= npg; k++) {
                                wgt = xjac;
                                for ( j = 1; j <= ndim; j++) {
#ifdef CUSTOM
                                        temp =  a * seed + c;
                                        seed = temp & (p - 1);
                                        ran00 = (double) seed / (double) p ;
#endif
#ifdef CURAND
                                        ran00 = hiprand_uniform(&localState);
#endif

                                        xn = (kg[j] - ran00) * dxg + 1.0;
                                        ia[j] = IMAX(IMIN((int)(xn), NDMX), 1);

                                        if (ia[j] > 1) {
                                                xo = xi[j * NDMX1 + ia[j]] - xi[j * NDMX1 + ia[j] - 1];
                                                rc = xi[j * NDMX1 + ia[j] - 1] + (xn - ia[j]) * xo;
                                        } else {
                                                xo = xi[j * NDMX1 + ia[j]];
                                                rc = (xn - ia[j]) * xo;
                                        }

                                        x[j] = regn[j] + rc * dx[j];
                                        wgt *= xo * xnd;

                                }

                                double tmp;

                                switch (fcode) {
                                    case 0:
                                        tmp = (*func1)(x, ndim);
                                        break;
                                    case 1:
                                        tmp = (*func2)(x, ndim);
                                        break;
                                    case 2:
                                        tmp = (*func3)(x, ndim);
                                        break;			
                                    case 3:
                                        tmp = (*BoxIntegral8_22)(x, ndim);
                                        break;
                                    case 4:
                                        tmp = (*GENZ1_8D)(x, ndim);
                                        break;
                                    case 5:
                                        tmp = (*GENZ2_2D)(x, ndim);
                                        break;
                                    case 6:
                                        tmp = (*GENZ2_6D)(x, ndim);
                                        break;
                                    case 7:
                                        tmp = (*GENZ3_3D)(x, ndim);
                                        break;
                                    case 8:
                                        tmp = (*GENZ4_5D)(x, ndim);
                                        break;
                                    case 9:
                                        tmp = (*GENZ5_8D)(x, ndim);
                                        break;
                                    case 10:
                                        tmp = (*BoxIntegral8_15)(x, ndim);
                                        break;
                                    case 11:
                                        tmp = (*GENZ6_6D)(x, ndim);
                                        break;
                                    case 12:
                                        tmp = (*GENZ4_8D)(x, ndim);
                                        break;
                                    case 13:
                                        tmp = (*GENZ3_8D)(x, ndim);
                                        break;
                                    case 14:
                                        tmp = sqsum(x, ndim);
                                        break;
                                    case 15:
                                        tmp = sumsqroot(x, ndim);
                                        break;
                                    case 16:
                                        tmp = prodones(x, ndim);
                                        break;
                                    case 17:
                                        tmp = prodexp(x, ndim);
                                        break;
                                    case 18:
                                        tmp = prodcub(x, ndim);
                                        break;
                                    case 19:
                                        tmp = prodx(x, ndim);
                                        break;
                                    case 20:
                                        tmp = sumfifj(x, ndim);
                                        break;
                                    case 21:
                                        tmp = sumfonefj(x, ndim);
                                        break;
                                    case 22:
                                        tmp = hellekalek(x, ndim);
                                        break;
                                    case 23:
                                        tmp = roosarnoldone(x, ndim);
                                        break;
                                    case 24:
                                        tmp = roosarnoldtwo(x, ndim);
                                        break;
                                    case 25:
                                        tmp = roosarnoldthree(x, ndim);
                                        break;
                                    case 26:
                                        tmp = rst(x, ndim);
                                        break;
                                    case 27:
                                        tmp = sobolprod(x, ndim);
                                        break;
                                    case 28:
                                        tmp = oscill(x, ndim);
                                        break;
                                    case 29:
                                        tmp = prpeak(x, ndim);
                                        break;
                                    case 30:
                                        tmp = sum(x, ndim);
                                        break;
                                    default:
                                        tmp = (*func2)(x, ndim);
                                        break;
                                    }

                                f = wgt * tmp;
                                f2 = f * f;

                                fb += f;
                                f2b += f2;


                                atomicAdd(&d[ia[1]*MXDIM1 + 1], fabs(f));

                        }  // end of npg loop

                        f2b = sqrt(f2b * npg);
                        f2b = (f2b - fb) * (f2b + fb);

                        fbg += fb;
                        f2bg += f2b;

                        for (int k = ndim; k >= 1; k--) {
                                kg[k] %= ng;
                                if (++kg[k] != 1) break;
                        }

                } //end of chunk for loop

                fbg  = blockReduceSum(fbg);
                f2bg = blockReduceSum(f2bg);


                if (tx == 0) {
                        atomicAdd(&result_dev[0], fbg);
                        atomicAdd(&result_dev[1], f2bg);
                        //if(iter==10) printf("ti is %e, tsi is %e\n",result_dev[0], result_dev[1]);

                }


        } // end of subcube if

}

__global__ void vegas_kernelF(int ng, int ndim, int npg, double xjac, double dxg,
                              double *result_dev, double xnd, double *xi,
                              double *d, double *dx, double *regn, int ncubes,
                              int iter, double sc, double sci, double ing,
                              int chunkSize, uint32_t totalNumThreads,
                              int LastChunk, int fcode) {

#ifdef CUSTOM
        uint64_t temp;
        uint32_t a = 1103515245;
        uint32_t c = 12345;
        uint32_t one, expi;
        one = 1;
        expi = 31;
        uint32_t p = one << expi;
#endif

        uint32_t seed, seed_init;
        seed_init = (iter) * ncubes;

        int m = blockIdx.x * blockDim.x + threadIdx.x;
        int tx = threadIdx.x;

        double fb, f2b, wgt, xn, xo, rc, f, f2, ran00;
        int kg[MXDIM + 1];
        //int ia[MXDIM + 1];
        int iaj;
        double x[MXDIM + 1];
        int k, j;
        double fbg, f2bg;
        //if(tx == 30 && blockIdx.x == 6771) printf("here m is %d\n", m);

        if (m < totalNumThreads) {
                if (m == totalNumThreads - 1) chunkSize = LastChunk + 1;
                //if(tx == 30 && blockIdx.x == 6771) printf("here m is %d\n", m);
                seed = seed_init + m * chunkSize;
#ifdef CURAND
                hiprandState localState;
                hiprand_init(seed, 0, 0, &localState);
#endif
                fbg = f2bg = 0.0;
                get_indx(m * chunkSize, &kg[1], ndim, ng);
                for (int t = 0; t < chunkSize; t++) {
                        fb = f2b = 0.0;
                        //get_indx(m*chunkSize+t, &kg[1], ndim, ng);
                        for ( k = 1; k <= npg; k++) {
                                wgt = xjac;
                                for ( j = 1; j <= ndim; j++) {
#ifdef CUSTOM
                                        temp =  a * seed + c;
                                        seed = temp & (p - 1);
                                        ran00 = (double) seed / (double) p ;
#endif
#ifdef CURAND
                                        ran00 = hiprand_uniform(&localState);
#endif

                                        xn = (kg[j] - ran00) * dxg + 1.0;
                                        iaj   = IMAX(IMIN((int)(xn), NDMX), 1);

                                        if (iaj > 1) {
                                                xo = xi[j * NDMX1 + iaj] - xi[j * NDMX1 + iaj - 1];
                                                rc = xi[j * NDMX1 + iaj - 1] + (xn - iaj) * xo;
                                        } else {
                                                xo = xi[j * NDMX1 + iaj];
                                                rc = (xn - iaj) * xo;
                                        }

                                        x[j] = regn[j] + rc * dx[j];
                                        wgt *= xo * xnd;

                                }

                                double tmp;

                                switch (fcode) {
                                    case 0:
                                        tmp = (*func1)(x, ndim);
                                        break;
                                    case 1:
                                        tmp = (*func2)(x, ndim);
                                        break;
                                    case 2:
                                        tmp = (*func3)(x, ndim);
                                        break;			
                                    case 3:
                                        
                                        tmp = (*BoxIntegral8_22)(x, ndim);
                                        break;
                                    case 4:
                                        tmp = (*GENZ1_8D)(x, ndim);
                                        break;
                                    case 5:
                                        tmp = (*GENZ2_2D)(x, ndim);
                                        break;
                                    case 6:
                                        tmp = (*GENZ2_6D)(x, ndim);
                                        break;
                                    case 7:
                                        tmp = (*GENZ3_3D)(x, ndim);
                                        break;
                                    case 8:
                                        tmp = (*GENZ4_5D)(x, ndim);
                                        break;
                                    case 9:
                                        tmp = (*GENZ5_8D)(x, ndim);
                                        break;
                                    case 10:
                                        tmp = (*BoxIntegral8_15)(x, ndim);
                                        break;
                                    case 11:
                                        tmp = (*GENZ6_6D)(x, ndim);
                                        break;
                                    case 12:
                                        tmp = (*GENZ4_8D)(x, ndim);
                                        break;
                                    case 13:
                                        tmp = (*GENZ3_8D)(x, ndim);
                                        break;
                                    case 14:
                                        tmp = wgt * sqsum(x, ndim);
                                        break;
                                    case 15:
                                        tmp = wgt * sumsqroot(x, ndim);
                                        break;
                                    case 16:
                                        tmp = wgt * prodones(x, ndim);
                                        break;
                                    case 17:
                                        tmp = wgt * prodexp(x, ndim);
                                        break;
                                    case 18:
                                        tmp = wgt * prodcub(x, ndim);
                                        break;
                                    case 19:
                                        tmp = wgt * prodx(x, ndim);
                                        break;
                                    case 20:
                                        tmp = wgt * sumfifj(x, ndim);
                                        break;
                                    case 21:
                                        tmp = wgt * sumfonefj(x, ndim);
                                        break;
                                    case 22:
                                        tmp = wgt * hellekalek(x, ndim);
                                        break;
                                    case 23:
                                        tmp = wgt * roosarnoldone(x, ndim);
                                        break;
                                    case 24:
                                        tmp = wgt * roosarnoldtwo(x, ndim);
                                        break;
                                    case 25:
                                        tmp = wgt * roosarnoldthree(x, ndim);
                                        break;
                                    case 26:
                                        tmp = wgt * rst(x, ndim);
                                        break;
                                    case 27:
                                        tmp = wgt * sobolprod(x, ndim);
                                        break;
                                    case 28:
                                        tmp = wgt * oscill(x, ndim);
                                        break;
                                    case 29:
                                        tmp = wgt * prpeak(x, ndim);
                                        break;
                                    case 30:
                                        tmp = wgt * sum(x, ndim);
                                        break;
                                    default:
                                        tmp = (*func2)(x, ndim);
                                        break;
                                    }

                                f = wgt * tmp;
                                f2 = f * f;

                                fb += f;
                                f2b += f2;

                        }  // end of npg loop

                        f2b = sqrt(f2b * npg);
                        f2b = (f2b - fb) * (f2b + fb);

                        fbg += fb;
                        f2bg += f2b;

                        for (int k = ndim; k >= 1; k--) {
                                kg[k] %= ng;
                                if (++kg[k] != 1) break;
                        }

                } //end of chunk for loop

                fbg  = blockReduceSum(fbg);
                f2bg = blockReduceSum(f2bg);


                if (tx == 0) {
                        atomicAdd(&result_dev[0], fbg);
                        atomicAdd(&result_dev[1], f2bg);
                        //if(iter==10) printf("ti is %e, tsi is %e\n",result_dev[0], result_dev[1]);

                }


        } // end of subcube if

}

void rebin(double rc, int nd, double r[], double xin[], double xi[])

{
        int i, k = 0;
        double dr = 0.0, xn = 0.0, xo = 0.0;
        for (i = 1; i < nd; i++) {
                while (rc > dr)
                        dr += r[++k];
                if (k > 1) xo = xi[k - 1];
                xn = xi[k];
                dr -= rc;
                xin[i] = xn - (xn - xo) * dr / r[k];
        }

        for (i = 1; i < nd; i++) xi[i] = xin[i];
        xi[nd] = 1.0;
        // for (i=1;i<=nd;i++) printf("bins edges: %.10f\n", xi[i]);
        // printf("---------------------\n");
}


void vegas(double regn[], int ndim, int fcode,
           double ncall, double *tgral, double *sd,
           double *chi2a, int titer, int itmax, int skip)

{

        int i, it, j, k, nd, ndo, ng, npg, ncubes;
        //int ia[MXDIM + 1];
        double calls, dv2g, dxg, rc, ti, tsi, wgt, xjac, xn, xnd, xo;
        /* double d[(NDMX + 1)*(MXDIM + 1)], dt[MXDIM + 1],
                dx[MXDIM + 1], r[NDMX + 1], x[MXDIM + 1], xi[(MXDIM + 1)*(NDMX + 1)], xin[NDMX + 1];*/

        double schi, si, swgt;
        double result[2];
        double *d, *dt, *dx, *r, *x, *xi, *xin;
        int *ia;

        d = (double*)malloc(sizeof(double) * (NDMX + 1) * (MXDIM + 1)) ;
        dt = (double*)malloc(sizeof(double) * (MXDIM + 1)) ;
        dx = (double*)malloc(sizeof(double) * (MXDIM + 1)) ;
        r = (double*)malloc(sizeof(double) * (NDMX + 1)) ;
        x = (double*)malloc(sizeof(double) * (MXDIM + 1)) ;
        xi = (double*)malloc(sizeof(double) * (MXDIM + 1) * (NDMX + 1)) ;
        xin = (double*)malloc(sizeof(double) * (NDMX + 1)) ;
        ia = (int*)malloc(sizeof(int) * (MXDIM + 1)) ;


// code works only  for (2 * ng - NDMX) >= 0)

        ndo = 1;
        for (j = 1; j <= ndim; j++) xi[j * NDMX1 + 1] = 1.0;
        si = swgt = schi = 0.0;
        nd = NDMX;
        ng = 1;
        ng = (int)pow(ncall / 2.0 + 0.25, 1.0 / ndim);
        for (k = 1, i = 1; i < ndim; i++) k *= ng;
        double sci = 1.0 / k;
        double sc = k;
        k *= ng;
        ncubes = k;
        npg = IMAX(ncall / k, 2);
        calls = (double)npg * (double)k;
        dxg = 1.0 / ng;
        double ing = dxg;
        for (dv2g = 1, i = 1; i <= ndim; i++) dv2g *= dxg;
        dv2g = (calls * dv2g * calls * dv2g) / npg / npg / (npg - 1.0);
        xnd = nd;
        dxg *= xnd;
        xjac = 1.0 / calls;
        for (j = 1; j <= ndim; j++) {
                dx[j] = regn[j + ndim] - regn[j];
                //printf("%e, %e\n", dx[j], xjac);
                xjac *= dx[j];
        }



        for (i = 1; i <= IMAX(nd, ndo); i++) r[i] = 1.0;
        for (j = 1; j <= ndim; j++) rebin(ndo / xnd, nd, r, xin, &xi[j * NDMX1]);
        ndo = nd;



        //printf("ng, npg, ncubes, xjac, %d, %d, %12d, %e\n", ng, npg, ncubes, xjac);





        double *d_dev, *dx_dev, *x_dev, *xi_dev, *regn_dev,  *result_dev;
        int *ia_dev;

        hipMalloc((void**)&result_dev, sizeof(double) * 2); cudaCheckError();
        hipMalloc((void**)&d_dev, sizeof(double) * (NDMX + 1) * (MXDIM + 1)); cudaCheckError();
        hipMalloc((void**)&dx_dev, sizeof(double) * (MXDIM + 1)); cudaCheckError();
        hipMalloc((void**)&x_dev, sizeof(double) * (MXDIM + 1)); cudaCheckError();
        hipMalloc((void**)&xi_dev, sizeof(double) * (MXDIM + 1) * (NDMX + 1)); cudaCheckError();
        hipMalloc((void**)&regn_dev, sizeof(double) * ((ndim * 2) + 1)); cudaCheckError();
        hipMalloc((void**)&ia_dev, sizeof(int) * (MXDIM + 1)); cudaCheckError();




        hipMemcpy( dx_dev, dx, sizeof(double) * (MXDIM + 1), hipMemcpyHostToDevice) ; cudaCheckError();
        hipMemcpy( x_dev, x, sizeof(double) * (MXDIM + 1), hipMemcpyHostToDevice) ; cudaCheckError();
        hipMemcpy( regn_dev, regn, sizeof(double) * ((ndim * 2) + 1), hipMemcpyHostToDevice) ; cudaCheckError();

        hipMemset(ia_dev, 0, sizeof(int) * (MXDIM + 1));

        int chunkSize;

        switch (fcode) {
        case 0:
                chunkSize = 2048;
                break;
        case 1:
                chunkSize = 32;
                break;
        case 2:
                chunkSize = 2048;
                break;
        default:
                chunkSize = 32;
                break;
        }


        uint32_t totalNumThreads = (uint32_t) ((ncubes + chunkSize - 1) / chunkSize);
        uint32_t totalCubes = totalNumThreads * chunkSize;
        int extra = totalCubes - ncubes;
        int LastChunk = chunkSize - extra;
        uint32_t nBlocks = ((uint32_t) (((ncubes + BLOCK_DIM_X - 1) / BLOCK_DIM_X)) / chunkSize) + 1;
        uint32_t nThreads = BLOCK_DIM_X;
        //printf("ncubes %d nBlocks %d nThreads %d totalNumThreads %d totalCubes %d extra  %d LastChunk %d\n", ncubes, nBlocks, nThreads, totalNumThreads, totalCubes, extra, LastChunk);

        //printf("the number of evaluation will be %e\n", calls);



        for (it = 1; it <= itmax; it++) {

                ti = tsi = 0.0;
                for (j = 1; j <= ndim; j++) {
                        for (i = 1; i <= nd; i++) d[i * MXDIM1 + j] = 0.0;
                }


                hipMemcpy( xi_dev, xi, sizeof(double) * (MXDIM + 1) * (NDMX + 1), hipMemcpyHostToDevice) ; cudaCheckError();
                hipMemset(d_dev, 0, sizeof(double) * (NDMX + 1) * (MXDIM + 1));
                hipMemset(result_dev, 0, 2 * sizeof(double));

                vegas_kernel <<< nBlocks, nThreads>>>(ng, ndim, npg, xjac, dxg, result_dev, xnd,
                                                      xi_dev, d_dev, dx_dev, regn_dev, ncubes,
                                                      it, sc, sci,  ing, chunkSize,
                                                      totalNumThreads, LastChunk, fcode);

                hipMemcpy(xi, xi_dev, sizeof(double) * (MXDIM + 1) * (NDMX + 1), hipMemcpyDeviceToHost); cudaCheckError();
                hipMemcpy( d, d_dev,  sizeof(double) * (NDMX + 1) * (MXDIM + 1), hipMemcpyDeviceToHost) ; cudaCheckError();

                hipMemcpy(result, result_dev, sizeof(double) * 2, hipMemcpyDeviceToHost);

                //printf("ti is %f", ti);
                ti  = result[0];
                tsi = result[1];
                tsi *= dv2g;
                //printf("iter = %d  integ = %e   std = %e\n", it, ti, sqrt(tsi));

                if (it > skip) {
                        wgt = 1.0 / tsi;
                        si += wgt * ti;
                        schi += wgt * ti * ti;
                        swgt += wgt;
                        *tgral = si / swgt;
                        *chi2a = (schi - si * (*tgral)) / (it - 0.9999);
                        if (*chi2a < 0.0) *chi2a = 0.0;
                        *sd = sqrt(1.0 / swgt);
                        tsi = sqrt(tsi);
                        //printf("it %d\n", it);
                        if(verbosity)
                            printf("%5d,%14.7g,%9.2g,%9.2g\n", it, *tgral, *sd, *chi2a);
                }
                //printf("%3d   %e  %e\n", it, ti, tsi);



                for (j = 1; j <= 1; j++) {
                        xo = d[1 * MXDIM1 + j];
                        xn = d[2 * MXDIM1 + j];
                        d[1 * MXDIM1 + j] = (xo + xn) / 2.0;
                        dt[j] = d[1 * MXDIM1 + j];
                        for (i = 2; i < nd; i++) {
                                rc = xo + xn;
                                xo = xn;
                                xn = d[(i + 1) * MXDIM1 + j];
                                d[i * MXDIM1 + j] = (rc + xn) / 3.0;
                                dt[j] += d[i * MXDIM1 + j];
                        }
                        d[nd * MXDIM1 + j] = (xo + xn) / 2.0;
                        dt[j] += d[nd * MXDIM1 + j];
                        //printf("iter, j, dtj:    %d    %d      %e\n", it, j, dt[j]);
                }

                for (j = 1; j <= 1; j++) {
                        if (dt[j] > 0.0) {
                                rc = 0.0;
                                for (i = 1; i <= nd; i++) {
                                        //if (d[i * MXDIM1 + j] < TINY) d[i * MXDIM1 + j] = TINY;
                                        r[i] = pow((1.0 - d[i * MXDIM1 + j] / dt[j]) /
                                                   (log(dt[j]) - log(d[i * MXDIM1 + j])), ALPH);
                                        rc += r[i];
                                }

                                rebin(rc / xnd, nd, r, xin, &xi[j * NDMX1]);
                        }

                }
                for (j = 2; j <= ndim; j++) {
                        for (i = 1; i <= nd; i++) {
                                xi[j * NDMX1 + i] = xi[NDMX1 + i];
                        }
                }

        }  // end of iteration loop


        hipMemcpy( xi_dev, xi, sizeof(double) * (MXDIM + 1) * (NDMX + 1), hipMemcpyHostToDevice) ; cudaCheckError();


  for (it = itmax+1; it <= titer; it++) {

                ti = tsi = 0.0;

                hipMemset(result_dev, 0, 2 * sizeof(double));

                vegas_kernelF <<< nBlocks, nThreads>>>(ng, ndim, npg, xjac, dxg, result_dev, xnd,
                                                       xi_dev, d_dev, dx_dev, regn_dev, ncubes, it, sc,
                                                       sci,  ing, chunkSize, totalNumThreads,
                                                       LastChunk, fcode);
                hipMemcpy(result, result_dev, sizeof(double) * 2, hipMemcpyDeviceToHost);

                //printf("ti is %f", ti);
                ti  = result[0];
                tsi = result[1];
                tsi *= dv2g;
                //printf("iter = %d  integ = %e   std = %e\n", it, ti, sqrt(tsi));

                wgt = 1.0 / tsi;
                si += wgt * ti;
                schi += wgt * ti * ti;
                swgt += wgt;
                *tgral = si / swgt;
                *chi2a = (schi - si * (*tgral)) / (it - 0.9999);
                if (*chi2a < 0.0) *chi2a = 0.0;
                *sd = sqrt(1.0 / swgt);
                tsi = sqrt(tsi);
                //printf("it %d\n", it);
                if(verbosity)
                    printf("%5d,%14.7g,%9.4g,%9.2g\n", it, *tgral, *sd, *chi2a);
                //printf("%3d   %e  %e\n", it, ti, tsi);

  }  // end of iteration

        free(d);
        free(dt);
        free(dx);
        free(ia);
        free(x);
        free(xi);

        hipFree(d_dev);
        hipFree(dx_dev);
        hipFree(ia_dev);
        hipFree(x_dev);
        hipFree(xi_dev);
        hipFree(regn_dev);



}


int main(int argc, char **argv)

{

        if (argc < 9) {
                printf( "****************************************\n"
                        "Usage (6 arguments):\n"
                        "./vegas_mcubes FCODE  DIM LL  UL  NCALLS  SKIP\n"
                        "FCODE = 0 to MAX_NUMBER_OF_FUNCTIONS-1\n"
                        "NCALLS in scientific notation, e.g. 1.0E+07 \n"
                        "****************************************\n");
                exit(-1);
        }
        
        int j;
        double avgi, chi2a, sd;
        double regn[2 * MXDIM + 1];

        int fcode = atoi(argv[1]);
        int ndim = atoi(argv[2]);
        float LL = atof(argv[3]);
        float UL = atof(argv[4]);
        double ncall = atof(argv[5]);
        int titer = atoi(argv[6]);
        int itmax = atoi(argv[7]);
        int skip = atoi(argv[8]);
        verbosity = atoi(argv[9]);
        auto t0 = std::chrono::high_resolution_clock::now();

        avgi = sd = chi2a = 0.0;
        for (j = 1; j <= ndim; j++) {
                regn[j] = LL;
                regn[j + ndim] = UL;
        }


        //std::cout<<"True value:"<<GetTrueValue(fcode)<<"\n";
        
        vegas(regn, ndim, fcode, ncall, &avgi, &sd, &chi2a, titer, itmax, skip);
        MilliSeconds dt = std::chrono::high_resolution_clock::now() - t0;
        
        double absolute_error = abs(GetTrueValue(fcode) - avgi);
        double relative_error = abs(absolute_error/GetTrueValue(fcode));
        //std::cout<<abs(GetTrueValue(fcode) - avgi)<<"\t"<<abs(absolute_error/GetTrueValue(fcode))<<"\n";
       // std::cout.precision(15); 
        std::cout << fcode << ","
            << std::scientific << avgi << "," 
            << std::scientific << sd << "," 
            << chi2a << ","
            << titer << "," 
            << itmax << "," 
            << skip << "," 
            << ncall << ","
            << dt.count() << "\t,";
        std::cout<<absolute_error<<",\t"<<relative_error<<"\n";



        return 0;

}

